#include "hip/hip_runtime.h"
/*
  Implements the Landau kernel
*/
#include <petscconf.h>

#include <petsclandau.h>
#if defined(PETSC_HAVE_CUDA_CLANG)
  #define LANDAU_NOT_IMPLEMENTED SETERRQ(PETSC_COMM_SELF, PETSC_ERR_SUP, "Not supported with CLANG")
PetscErrorCode LandauCUDAJacobian(DM[], const PetscInt, const PetscInt, const PetscInt, const PetscInt[], PetscReal[], PetscScalar[], const PetscScalar[], const LandauStaticData *, const PetscReal, const PetscLogEvent[], const PetscInt[], const PetscInt[], Mat[], Mat)
{
  LANDAU_NOT_IMPLEMENTED;
}
PetscErrorCode LandauCUDACreateMatMaps(P4estVertexMaps *, pointInterpolationP4est (*)[LANDAU_MAX_Q_FACE], PetscInt[], PetscInt, PetscInt)
{
  LANDAU_NOT_IMPLEMENTED;
}
PetscErrorCode LandauCUDADestroyMatMaps(P4estVertexMaps *, PetscInt)
{
  LANDAU_NOT_IMPLEMENTED;
}
PetscErrorCode LandauCUDAStaticDataSet(DM, const PetscInt, const PetscInt, const PetscInt, PetscInt[], PetscInt[], PetscInt[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], PetscReal[], LandauStaticData *)
{
  LANDAU_NOT_IMPLEMENTED;
}
PetscErrorCode LandauCUDAStaticDataClear(LandauStaticData *)
{
  LANDAU_NOT_IMPLEMENTED;
}
#else
  #include <petsc/private/dmpleximpl.h> /*I  "dmpleximpl.h"   I*/
  #define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1
  #include <../src/mat/impls/aij/seq/aij.h>
  #include <petscmat.h>
  #include <petscdevice_cuda.h>

  #include "../land_tensors.h"
  #include <petscaijdevice.h>

PETSC_EXTERN PetscErrorCode LandauCUDACreateMatMaps(P4estVertexMaps maps[], pointInterpolationP4est (*pointMaps)[LANDAU_MAX_Q_FACE], PetscInt Nf[], PetscInt, PetscInt grid)
{
  P4estVertexMaps h_maps;
  PetscFunctionBegin;
  h_maps.num_elements = maps[grid].num_elements;
  h_maps.num_face     = maps[grid].num_face;
  h_maps.num_reduced  = maps[grid].num_reduced;
  h_maps.deviceType   = maps[grid].deviceType;
  h_maps.Nf           = Nf[grid];
  h_maps.numgrids     = maps[grid].numgrids;
  PetscCallCUDA(hipMalloc((void **)&h_maps.c_maps, maps[grid].num_reduced * sizeof *pointMaps));
  PetscCallCUDA(hipMemcpy(h_maps.c_maps, maps[grid].c_maps, maps[grid].num_reduced * sizeof *pointMaps, hipMemcpyHostToDevice));
  PetscCallCUDA(hipMalloc((void **)&h_maps.gIdx, maps[grid].num_elements * sizeof *maps[grid].gIdx));
  PetscCallCUDA(hipMemcpy(h_maps.gIdx, maps[grid].gIdx, maps[grid].num_elements * sizeof *maps[grid].gIdx, hipMemcpyHostToDevice));
  PetscCallCUDA(hipMalloc((void **)&maps[grid].d_self, sizeof(P4estVertexMaps)));
  PetscCallCUDA(hipMemcpy(maps[grid].d_self, &h_maps, sizeof(P4estVertexMaps), hipMemcpyHostToDevice));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_EXTERN PetscErrorCode LandauCUDADestroyMatMaps(P4estVertexMaps maps[], PetscInt num_grids)
{
  PetscFunctionBegin;
  for (PetscInt grid = 0; grid < num_grids; grid++) {
    P4estVertexMaps *d_maps = maps[grid].d_self, h_maps;
    PetscCallCUDA(hipMemcpy(&h_maps, d_maps, sizeof(P4estVertexMaps), hipMemcpyDeviceToHost));
    PetscCallCUDA(hipFree(h_maps.c_maps));
    PetscCallCUDA(hipFree(h_maps.gIdx));
    PetscCallCUDA(hipFree(d_maps));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode LandauCUDAStaticDataSet(DM plex, const PetscInt Nq, const PetscInt batch_sz, const PetscInt num_grids, PetscInt a_numCells[], PetscInt a_species_offset[], PetscInt a_mat_offset[], PetscReal nu_alpha[], PetscReal nu_beta[], PetscReal a_invMass[], PetscReal[], PetscReal a_invJ[], PetscReal a_x[], PetscReal a_y[], PetscReal a_z[], PetscReal a_w[], LandauStaticData *SData_d)
{
  PetscTabulation *Tf;
  PetscReal       *BB, *DD;
  PetscInt         dim, Nb = Nq, szf = sizeof(PetscReal), szs = sizeof(PetscScalar), szi = sizeof(PetscInt);
  PetscInt         h_ip_offset[LANDAU_MAX_GRIDS + 1], h_ipf_offset[LANDAU_MAX_GRIDS + 1], h_elem_offset[LANDAU_MAX_GRIDS + 1], nip, IPfdf_sz, Nf;
  PetscDS          prob;

  PetscFunctionBegin;
  PetscCall(DMGetDimension(plex, &dim));
  PetscCall(DMGetDS(plex, &prob));
  PetscCheck(LANDAU_DIM == dim, PETSC_COMM_WORLD, PETSC_ERR_PLIB, "dim %" PetscInt_FMT " != LANDAU_DIM %d", dim, LANDAU_DIM);
  PetscCall(PetscDSGetTabulation(prob, &Tf));
  BB = Tf[0]->T[0];
  DD = Tf[0]->T[1];
  Nf = h_ip_offset[0] = h_ipf_offset[0] = h_elem_offset[0] = 0;
  nip                                                      = 0;
  IPfdf_sz                                                 = 0;
  for (PetscInt grid = 0; grid < num_grids; grid++) {
    PetscInt nfloc          = a_species_offset[grid + 1] - a_species_offset[grid];
    h_elem_offset[grid + 1] = h_elem_offset[grid] + a_numCells[grid];
    nip += a_numCells[grid] * Nq;
    h_ip_offset[grid + 1] = nip;
    IPfdf_sz += Nq * nfloc * a_numCells[grid];
    h_ipf_offset[grid + 1] = IPfdf_sz;
  }
  Nf = a_species_offset[num_grids];
  {
    PetscCallCUDA(hipMalloc((void **)&SData_d->B, Nq * Nb * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->B, BB, Nq * Nb * szf, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->D, Nq * Nb * dim * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->D, DD, Nq * Nb * dim * szf, hipMemcpyHostToDevice));

    PetscCallCUDA(hipMalloc((void **)&SData_d->alpha, Nf * szf));   // kernel input
    PetscCallCUDA(hipMalloc((void **)&SData_d->beta, Nf * szf));    // kernel input
    PetscCallCUDA(hipMalloc((void **)&SData_d->invMass, Nf * szf)); // kernel input

    PetscCallCUDA(hipMemcpy(SData_d->alpha, nu_alpha, Nf * szf, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(SData_d->beta, nu_beta, Nf * szf, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(SData_d->invMass, a_invMass, Nf * szf, hipMemcpyHostToDevice));

    // collect geometry
    PetscCallCUDA(hipMalloc((void **)&SData_d->invJ, nip * dim * dim * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->invJ, a_invJ, nip * dim * dim * szf, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->x, nip * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->x, a_x, nip * szf, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->y, nip * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->y, a_y, nip * szf, hipMemcpyHostToDevice));
  #if LANDAU_DIM == 3
    PetscCallCUDA(hipMalloc((void **)&SData_d->z, nip * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->z, a_z, nip * szf, hipMemcpyHostToDevice));
  #else
    (void)a_z;
  #endif
    PetscCallCUDA(hipMalloc((void **)&SData_d->w, nip * szf)); // kernel input
    PetscCallCUDA(hipMemcpy(SData_d->w, a_w, nip * szf, hipMemcpyHostToDevice));

    PetscCallCUDA(hipMalloc((void **)&SData_d->NCells, num_grids * szi));
    PetscCallCUDA(hipMemcpy(SData_d->NCells, a_numCells, num_grids * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->species_offset, (num_grids + 1) * szi));
    PetscCallCUDA(hipMemcpy(SData_d->species_offset, a_species_offset, (num_grids + 1) * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->mat_offset, (num_grids + 1) * szi));
    PetscCallCUDA(hipMemcpy(SData_d->mat_offset, a_mat_offset, (num_grids + 1) * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->ip_offset, (num_grids + 1) * szi));
    PetscCallCUDA(hipMemcpy(SData_d->ip_offset, h_ip_offset, (num_grids + 1) * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->ipf_offset, (num_grids + 1) * szi));
    PetscCallCUDA(hipMemcpy(SData_d->ipf_offset, h_ipf_offset, (num_grids + 1) * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->elem_offset, (num_grids + 1) * szi));
    PetscCallCUDA(hipMemcpy(SData_d->elem_offset, h_elem_offset, (num_grids + 1) * szi, hipMemcpyHostToDevice));
    PetscCallCUDA(hipMalloc((void **)&SData_d->maps, num_grids * sizeof(P4estVertexMaps *)));
    // allocate space for dynamic data once
    PetscCallCUDA(hipMalloc((void **)&SData_d->Eq_m, Nf * szf));               // this could be for each vertex (todo?)
    PetscCallCUDA(hipMalloc((void **)&SData_d->f, nip * Nf * szs * batch_sz)); // for each vertex in batch
    PetscCallCUDA(hipMalloc((void **)&SData_d->dfdx, nip * Nf * szs * batch_sz));
    PetscCallCUDA(hipMalloc((void **)&SData_d->dfdy, nip * Nf * szs * batch_sz));
  #if LANDAU_DIM == 3
    PetscCallCUDA(hipMalloc((void **)&SData_d->dfdz, nip * Nf * szs * batch_sz));
  #endif
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode LandauCUDAStaticDataClear(LandauStaticData *SData_d)
{
  PetscFunctionBegin;
  if (SData_d->alpha) {
    PetscCallCUDA(hipFree(SData_d->alpha));
    SData_d->alpha = NULL;
    PetscCallCUDA(hipFree(SData_d->beta));
    PetscCallCUDA(hipFree(SData_d->invMass));
    PetscCallCUDA(hipFree(SData_d->B));
    PetscCallCUDA(hipFree(SData_d->D));
    PetscCallCUDA(hipFree(SData_d->invJ));
  #if LANDAU_DIM == 3
    PetscCallCUDA(hipFree(SData_d->z));
  #endif
    PetscCallCUDA(hipFree(SData_d->x));
    PetscCallCUDA(hipFree(SData_d->y));
    PetscCallCUDA(hipFree(SData_d->w));
    // dynamic data
    PetscCallCUDA(hipFree(SData_d->Eq_m));
    PetscCallCUDA(hipFree(SData_d->f));
    PetscCallCUDA(hipFree(SData_d->dfdx));
    PetscCallCUDA(hipFree(SData_d->dfdy));
  #if LANDAU_DIM == 3
    PetscCallCUDA(hipFree(SData_d->dfdz));
  #endif
    PetscCallCUDA(hipFree(SData_d->NCells));
    PetscCallCUDA(hipFree(SData_d->species_offset));
    PetscCallCUDA(hipFree(SData_d->mat_offset));
    PetscCallCUDA(hipFree(SData_d->ip_offset));
    PetscCallCUDA(hipFree(SData_d->ipf_offset));
    PetscCallCUDA(hipFree(SData_d->elem_offset));
    PetscCallCUDA(hipFree(SData_d->maps));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}
//
// The GPU Landau kernel
//
__global__ void landau_form_fdf(const PetscInt dim, const PetscInt Nb, const PetscInt num_grids, const PetscReal d_invJ[], const PetscReal *const BB, const PetscReal *const DD, PetscScalar *d_vertex_f, P4estVertexMaps *d_maps[], PetscReal d_f[], PetscReal d_dfdx[], PetscReal d_dfdy[],
  #if LANDAU_DIM == 3
                                PetscReal      d_dfdz[],
  #endif
                                const PetscInt d_numCells[], const PetscInt d_species_offset[], const PetscInt d_mat_offset[], const PetscInt d_ip_offset[], const PetscInt d_ipf_offset[], const PetscInt d_elem_offset[]) // output
{
  const PetscInt   Nq = blockDim.y, myQi = threadIdx.y;
  const PetscInt   b_elem_idx = blockIdx.y, b_id = blockIdx.x, IPf_sz_glb = d_ipf_offset[num_grids];
  const PetscReal *Bq = &BB[myQi * Nb], *Dq = &DD[myQi * Nb * dim];
  PetscInt         grid = 0, f, d, b, e, q;
  while (b_elem_idx >= d_elem_offset[grid + 1]) grid++;
  {
    const PetscInt     loc_nip = d_numCells[grid] * Nq, loc_Nf = d_species_offset[grid + 1] - d_species_offset[grid], loc_elem = b_elem_idx - d_elem_offset[grid];
    const PetscInt     moffset = LAND_MOFFSET(b_id, grid, gridDim.x, num_grids, d_mat_offset);
    const PetscScalar *coef;
    PetscReal          u_x[LANDAU_DIM];
    const PetscReal   *invJ = &d_invJ[(d_ip_offset[grid] + loc_elem * Nq + myQi) * dim * dim];
    PetscScalar        coef_buff[LANDAU_MAX_SPECIES * LANDAU_MAX_NQ];
    if (!d_maps) {
      coef = &d_vertex_f[b_id * IPf_sz_glb + d_ipf_offset[grid] + loc_elem * Nb * loc_Nf]; // closure and IP indexing are the same
    } else {
      coef = coef_buff;
      for (f = 0; f < loc_Nf; ++f) {
        LandauIdx *const Idxs = &d_maps[grid]->gIdx[loc_elem][f][0];
        for (b = 0; b < Nb; ++b) {
          PetscInt idx = Idxs[b];
          if (idx >= 0) {
            coef_buff[f * Nb + b] = d_vertex_f[idx + moffset];
          } else {
            idx                   = -idx - 1;
            coef_buff[f * Nb + b] = 0;
            for (q = 0; q < d_maps[grid]->num_face; q++) {
              PetscInt  id    = d_maps[grid]->c_maps[idx][q].gid;
              PetscReal scale = d_maps[grid]->c_maps[idx][q].scale;
              if (id >= 0) coef_buff[f * Nb + b] += scale * d_vertex_f[id + moffset];
            }
          }
        }
      }
    }

    /* get f and df */
    for (f = threadIdx.x; f < loc_Nf; f += blockDim.x) {
      PetscReal      refSpaceDer[LANDAU_DIM];
      const PetscInt idx = b_id * IPf_sz_glb + d_ipf_offset[grid] + f * loc_nip + loc_elem * Nq + myQi;
      d_f[idx]           = 0.0;
      for (d = 0; d < LANDAU_DIM; ++d) refSpaceDer[d] = 0.0;
      for (b = 0; b < Nb; ++b) {
        const PetscInt cidx = b;
        d_f[idx] += Bq[cidx] * PetscRealPart(coef[f * Nb + cidx]);
        for (d = 0; d < dim; ++d) refSpaceDer[d] += Dq[cidx * dim + d] * PetscRealPart(coef[f * Nb + cidx]);
      }
      for (d = 0; d < dim; ++d) {
        for (e = 0, u_x[d] = 0.0; e < dim; ++e) u_x[d] += invJ[e * dim + d] * refSpaceDer[e];
      }
      d_dfdx[idx] = u_x[0];
      d_dfdy[idx] = u_x[1];
  #if LANDAU_DIM == 3
      d_dfdz[idx] = u_x[2];
  #endif
    }
  }
}

__device__ void landau_jac_kernel(const PetscInt num_grids, const PetscInt jpidx, PetscInt nip_global, const PetscInt grid, const PetscReal xx[], const PetscReal yy[], const PetscReal ww[], const PetscReal invJj[], const PetscInt Nftot, const PetscReal nu_alpha[], const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[], const PetscReal *const BB, const PetscReal *const DD, PetscScalar *elemMat, P4estVertexMaps *d_maps[], PetscSplitCSRDataStructure d_mat, // output
                                  PetscScalar s_fieldMats[][LANDAU_MAX_NQ], // all these arrays are in shared memory
                                  PetscReal s_scale[][LANDAU_MAX_Q_FACE], PetscInt s_idx[][LANDAU_MAX_Q_FACE], PetscReal s_g2[][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES], PetscReal s_g3[][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES], PetscReal s_gg2[][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES], PetscReal s_gg3[][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES], PetscReal s_nu_alpha[], PetscReal s_nu_beta[], PetscReal s_invMass[], PetscReal s_f[], PetscReal s_dfx[], PetscReal s_dfy[], PetscReal d_f[], PetscReal d_dfdx[], PetscReal d_dfdy[], // global memory
  #if LANDAU_DIM == 3
                                  const PetscReal zz[], PetscReal s_dfz[], PetscReal d_dfdz[],
  #endif
                                  const PetscInt d_numCells[], const PetscInt d_species_offset[], const PetscInt d_mat_offset[], const PetscInt d_ip_offset[], const PetscInt d_ipf_offset[], const PetscInt d_elem_offset[])
{
  const PetscInt  Nq = blockDim.y, myQi = threadIdx.y;
  const PetscInt  b_elem_idx = blockIdx.y, b_id = blockIdx.x, IPf_sz_glb = d_ipf_offset[num_grids];
  const PetscInt  loc_Nf = d_species_offset[grid + 1] - d_species_offset[grid], loc_elem = b_elem_idx - d_elem_offset[grid];
  const PetscInt  moffset = LAND_MOFFSET(b_id, grid, gridDim.x, num_grids, d_mat_offset);
  int             delta, d, f, g, d2, dp, d3, fieldA, ipidx_b;
  PetscReal       gg2_temp[LANDAU_DIM], gg3_temp[LANDAU_DIM][LANDAU_DIM];
  #if LANDAU_DIM == 2
  const PetscReal vj[3] = {xx[jpidx], yy[jpidx]};
  constexpr int   dim   = 2;
  #else
  const PetscReal vj[3] = {xx[jpidx], yy[jpidx], zz[jpidx]};
  constexpr int   dim   = 3;
  #endif
  const PetscInt  f_off = d_species_offset[grid], Nb = Nq;
  // create g2 & g3
  for (f = threadIdx.x; f < loc_Nf; f += blockDim.x) {
    for (d = 0; d < dim; d++) { // clear accumulation data D & K
      s_gg2[d][myQi][f] = 0;
      for (d2 = 0; d2 < dim; d2++) s_gg3[d][d2][myQi][f] = 0;
    }
  }
  #pragma unroll
  for (d2 = 0; d2 < dim; d2++) {
    gg2_temp[d2] = 0;
  #pragma unroll
    for (d3 = 0; d3 < dim; d3++) gg3_temp[d2][d3] = 0;
  }
  if (threadIdx.y == 0) {
    // copy species into shared memory
    for (fieldA = threadIdx.x; fieldA < Nftot; fieldA += blockDim.x) {
      s_nu_alpha[fieldA] = nu_alpha[fieldA];
      s_nu_beta[fieldA]  = nu_beta[fieldA];
      s_invMass[fieldA]  = invMass[fieldA];
    }
  }
  __syncthreads();
  // inner integral, collect gg2/3
  for (ipidx_b = 0; ipidx_b < nip_global; ipidx_b += blockDim.x) {
    const PetscInt ipidx = ipidx_b + threadIdx.x;
    PetscInt       f_off_r, grid_r, loc_Nf_r, nip_loc_r, ipidx_g, fieldB, IPf_idx_r;
    __syncthreads();
    if (ipidx < nip_global) {
      grid_r = 0;
      while (ipidx >= d_ip_offset[grid_r + 1]) grid_r++;
      f_off_r   = d_species_offset[grid_r];
      ipidx_g   = ipidx - d_ip_offset[grid_r];
      nip_loc_r = d_numCells[grid_r] * Nq;
      loc_Nf_r  = d_species_offset[grid_r + 1] - d_species_offset[grid_r];
      IPf_idx_r = b_id * IPf_sz_glb + d_ipf_offset[grid_r] + ipidx_g;
      for (fieldB = threadIdx.y; fieldB < loc_Nf_r; fieldB += blockDim.y) {
        const PetscInt idx                       = IPf_idx_r + fieldB * nip_loc_r;
        s_f[fieldB * blockDim.x + threadIdx.x]   = d_f[idx]; // all vector threads get copy of data
        s_dfx[fieldB * blockDim.x + threadIdx.x] = d_dfdx[idx];
        s_dfy[fieldB * blockDim.x + threadIdx.x] = d_dfdy[idx];
  #if LANDAU_DIM == 3
        s_dfz[fieldB * blockDim.x + threadIdx.x] = d_dfdz[idx];
  #endif
      }
    }
    __syncthreads();
    if (ipidx < nip_global) {
      const PetscReal wi = ww[ipidx], x = xx[ipidx], y = yy[ipidx];
      PetscReal       temp1[3] = {0, 0, 0}, temp2 = 0;
  #if LANDAU_DIM == 2
      PetscReal       Ud[2][2], Uk[2][2], mask = (PetscAbs(vj[0] - x) < 100 * PETSC_SQRT_MACHINE_EPSILON && PetscAbs(vj[1] - y) < 100 * PETSC_SQRT_MACHINE_EPSILON) ? 0. : 1.;
      LandauTensor2D(vj, x, y, Ud, Uk, mask);
  #else
      PetscReal U[3][3], z = zz[ipidx], mask = (PetscAbs(vj[0] - x) < 100 * PETSC_SQRT_MACHINE_EPSILON && PetscAbs(vj[1] - y) < 100 * PETSC_SQRT_MACHINE_EPSILON && PetscAbs(vj[2] - z) < 100 * PETSC_SQRT_MACHINE_EPSILON) ? 0. : 1.;
      LandauTensor3D(vj, x, y, z, U, mask);
  #endif
      for (int fieldB = 0; fieldB < loc_Nf_r; fieldB++) {
        temp1[0] += s_dfx[fieldB * blockDim.x + threadIdx.x] * s_nu_beta[fieldB + f_off_r] * s_invMass[fieldB + f_off_r] * 7; // todo : bring lambdas in
        temp1[1] += s_dfy[fieldB * blockDim.x + threadIdx.x] * s_nu_beta[fieldB + f_off_r] * s_invMass[fieldB + f_off_r] * 7;
  #if LANDAU_DIM == 3
        temp1[2] += s_dfz[fieldB * blockDim.x + threadIdx.x] * s_nu_beta[fieldB + f_off_r] * s_invMass[fieldB + f_off_r] * 7;
  #endif
        temp2 += s_f[fieldB * blockDim.x + threadIdx.x] * s_nu_beta[fieldB + f_off_r] * 7;
      }
      temp1[0] *= wi;
      temp1[1] *= wi;
  #if LANDAU_DIM == 3
      temp1[2] *= wi;
  #endif
      temp2 *= wi;
  #if LANDAU_DIM == 2
    #pragma unroll
      for (d2 = 0; d2 < 2; d2++) {
    #pragma unroll
        for (d3 = 0; d3 < 2; ++d3) {
          /* K = U * grad(f): g2=e: i,A */
          gg2_temp[d2] += Uk[d2][d3] * temp1[d3];
          /* D = -U * (I \kron (fx)): g3=f: i,j,A */
          gg3_temp[d2][d3] += Ud[d2][d3] * temp2;
        }
      }
  #else
    #pragma unroll
      for (d2 = 0; d2 < 3; ++d2) {
    #pragma unroll
        for (d3 = 0; d3 < 3; ++d3) {
          /* K = U * grad(f): g2 = e: i,A */
          gg2_temp[d2] += U[d2][d3] * temp1[d3];
          /* D = -U * (I \kron (fx)): g3 = f: i,j,A */
          gg3_temp[d2][d3] += U[d2][d3] * temp2;
        }
      }
  #endif
    }
  } /* IPs */

  /* reduce gg temp sums across threads */
  for (delta = blockDim.x / 2; delta > 0; delta /= 2) {
  #pragma unroll
    for (d2 = 0; d2 < dim; d2++) {
      gg2_temp[d2] += __shfl_xor_sync(0xffffffff, gg2_temp[d2], delta, blockDim.x);
  #pragma unroll
      for (d3 = 0; d3 < dim; d3++) gg3_temp[d2][d3] += __shfl_xor_sync(0xffffffff, gg3_temp[d2][d3], delta, blockDim.x);
    }
  }
  // add alpha and put in gg2/3
  for (fieldA = threadIdx.x; fieldA < loc_Nf; fieldA += blockDim.x) {
  #pragma unroll
    for (d2 = 0; d2 < dim; d2++) {
      s_gg2[d2][myQi][fieldA] += gg2_temp[d2] * s_nu_alpha[fieldA + f_off];
  #pragma unroll
      for (d3 = 0; d3 < dim; d3++) s_gg3[d2][d3][myQi][fieldA] -= gg3_temp[d2][d3] * s_nu_alpha[fieldA + f_off] * s_invMass[fieldA + f_off];
    }
  }
  __syncthreads();
  /* add electric field term once per IP */
  for (fieldA = threadIdx.x; fieldA < loc_Nf; fieldA += blockDim.x) s_gg2[dim - 1][myQi][fieldA] += Eq_m[fieldA + f_off];
  __syncthreads();
  /* Jacobian transform - g2 */
  for (fieldA = threadIdx.x; fieldA < loc_Nf; fieldA += blockDim.x) {
    PetscReal wj = ww[jpidx];
    for (d = 0; d < dim; ++d) {
      s_g2[d][myQi][fieldA] = 0.0;
      for (d2 = 0; d2 < dim; ++d2) {
        s_g2[d][myQi][fieldA] += invJj[d * dim + d2] * s_gg2[d2][myQi][fieldA];
        s_g3[d][d2][myQi][fieldA] = 0.0;
        for (d3 = 0; d3 < dim; ++d3) {
          for (dp = 0; dp < dim; ++dp) s_g3[d][d2][myQi][fieldA] += invJj[d * dim + d3] * s_gg3[d3][dp][myQi][fieldA] * invJj[d2 * dim + dp];
        }
        s_g3[d][d2][myQi][fieldA] *= wj;
      }
      s_g2[d][myQi][fieldA] *= wj;
    }
  }
  __syncthreads(); // Synchronize (ensure all the data is available) and sum IP matrices
  /* FE matrix construction */
  {
    int fieldA, d, qj, d2, q, idx, totDim = Nb * loc_Nf;
    /* assemble */
    for (fieldA = 0; fieldA < loc_Nf; fieldA++) {
      for (f = threadIdx.y; f < Nb; f += blockDim.y) {
        for (g = threadIdx.x; g < Nb; g += blockDim.x) {
          PetscScalar t = 0;
          for (qj = 0; qj < Nq; qj++) {
            const PetscReal *BJq = &BB[qj * Nb], *DIq = &DD[qj * Nb * dim];
            for (d = 0; d < dim; ++d) {
              t += DIq[f * dim + d] * s_g2[d][qj][fieldA] * BJq[g];
              for (d2 = 0; d2 < dim; ++d2) t += DIq[f * dim + d] * s_g3[d][d2][qj][fieldA] * DIq[g * dim + d2];
            }
          }
          if (elemMat) {
            const PetscInt fOff = (fieldA * Nb + f) * totDim + fieldA * Nb + g;
            elemMat[fOff] += t; // ????
          } else s_fieldMats[f][g] = t;
        }
      }
      if (s_fieldMats) {
        PetscScalar            vals[LANDAU_MAX_Q_FACE * LANDAU_MAX_Q_FACE];
        PetscInt               nr, nc;
        const LandauIdx *const Idxs = &d_maps[grid]->gIdx[loc_elem][fieldA][0];
        __syncthreads();
        if (threadIdx.y == 0) {
          for (f = threadIdx.x; f < Nb; f += blockDim.x) {
            idx = Idxs[f];
            if (idx >= 0) {
              s_idx[f][0]   = idx + moffset;
              s_scale[f][0] = 1.;
            } else {
              idx = -idx - 1;
              for (q = 0; q < d_maps[grid]->num_face; q++) {
                if (d_maps[grid]->c_maps[idx][q].gid >= 0) s_idx[f][q] = d_maps[grid]->c_maps[idx][q].gid + moffset;
                else s_idx[f][q] = -1;
                s_scale[f][q] = d_maps[grid]->c_maps[idx][q].scale;
              }
            }
          }
        }
        __syncthreads();
        for (f = threadIdx.y; f < Nb; f += blockDim.y) {
          idx = Idxs[f];
          if (idx >= 0) {
            nr = 1;
          } else {
            nr = d_maps[grid]->num_face;
          }
          for (g = threadIdx.x; g < Nb; g += blockDim.x) {
            idx = Idxs[g];
            if (idx >= 0) {
              nc = 1;
            } else {
              nc = d_maps[grid]->num_face;
            }
            for (q = 0; q < nr; q++) {
              for (d = 0; d < nc; d++) vals[q * nc + d] = s_scale[f][q] * s_scale[g][d] * s_fieldMats[f][g];
            }
            static_cast<void>(MatSetValuesDevice(d_mat, nr, s_idx[f], nc, s_idx[g], vals, ADD_VALUES));
          }
        }
        __syncthreads();
      }
    }
  }
}

//
// The CUDA Landau kernel
//
__global__ void __launch_bounds__(256, 2) landau_jacobian(const PetscInt nip_global, const PetscInt dim, const PetscInt Nb, const PetscInt num_grids, const PetscReal invJj[], const PetscInt Nftot, const PetscReal nu_alpha[], const PetscReal nu_beta[], const PetscReal invMass[], const PetscReal Eq_m[], const PetscReal *const BB, const PetscReal *const DD, const PetscReal xx[], const PetscReal yy[], const PetscReal ww[], PetscScalar d_elem_mats[], P4estVertexMaps *d_maps[], PetscSplitCSRDataStructure d_mat, PetscReal d_f[], PetscReal d_dfdx[], PetscReal d_dfdy[],
  #if LANDAU_DIM == 3
                                                          const PetscReal zz[], PetscReal d_dfdz[],
  #endif
                                                          const PetscInt d_numCells[], const PetscInt d_species_offset[], const PetscInt d_mat_offset[], const PetscInt d_ip_offset[], const PetscInt d_ipf_offset[], const PetscInt d_elem_offset[])
{
  extern __shared__ PetscReal smem[];
  int                         size                                = 0;
  PetscReal(*s_g2)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES] = (PetscReal(*)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) & smem[size];
  size += LANDAU_MAX_NQ * LANDAU_MAX_SPECIES * LANDAU_DIM;
  PetscReal(*s_g3)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES] = (PetscReal(*)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) & smem[size];
  size += LANDAU_DIM * LANDAU_DIM * LANDAU_MAX_NQ * LANDAU_MAX_SPECIES;
  PetscReal(*s_gg2)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES] = (PetscReal(*)[LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) & smem[size];
  size += LANDAU_MAX_NQ * LANDAU_MAX_SPECIES * LANDAU_DIM;
  PetscReal(*s_gg3)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES] = (PetscReal(*)[LANDAU_DIM][LANDAU_DIM][LANDAU_MAX_NQ][LANDAU_MAX_SPECIES]) & smem[size];
  size += LANDAU_DIM * LANDAU_DIM * LANDAU_MAX_NQ * LANDAU_MAX_SPECIES;
  PetscReal *s_nu_alpha = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_nu_beta = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_invMass = &smem[size];
  size += LANDAU_MAX_SPECIES;
  PetscReal *s_f = &smem[size];
  size += blockDim.x * LANDAU_MAX_SPECIES;
  PetscReal *s_dfx = &smem[size];
  size += blockDim.x * LANDAU_MAX_SPECIES;
  PetscReal *s_dfy = &smem[size];
  size += blockDim.x * LANDAU_MAX_SPECIES;
  #if LANDAU_DIM == 3
  PetscReal *s_dfz = &smem[size];
  size += blockDim.x * LANDAU_MAX_SPECIES;
  #endif
  PetscScalar(*s_fieldMats)[LANDAU_MAX_NQ][LANDAU_MAX_NQ];
  PetscReal(*s_scale)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE] = nullptr;
  PetscInt(*s_idx)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE]    = nullptr;
  const PetscInt b_elem_idx = blockIdx.y, b_id = blockIdx.x;
  PetscInt       Nq = blockDim.y, grid = 0; // Nq == Nb
  PetscScalar   *elemMat = NULL;            /* my output */
  while (b_elem_idx >= d_elem_offset[grid + 1]) grid++;
  {
    const PetscInt   loc_Nf = d_species_offset[grid + 1] - d_species_offset[grid], loc_elem = b_elem_idx - d_elem_offset[grid];
    const PetscInt   myQi  = threadIdx.y;
    const PetscInt   jpidx = d_ip_offset[grid] + myQi + loc_elem * Nq;
    const PetscReal *invJ  = &invJj[jpidx * dim * dim];
    if (d_elem_mats) {
      PetscInt totDim = loc_Nf * Nb;
      elemMat         = d_elem_mats; // start a beginning and get to my element matrix
      for (PetscInt b_id2 = 0; b_id2 < b_id; b_id2++) {
        for (PetscInt grid2 = 0; grid2 < num_grids; grid2++) {
          PetscInt Nfloc2 = d_species_offset[grid2 + 1] - d_species_offset[grid2], totDim2 = Nfloc2 * Nb;
          elemMat += d_numCells[grid2] * totDim2 * totDim2; // jump past grids,could be in an offset
        }
      }
      for (PetscInt grid2 = 0; grid2 < grid; grid2++) {
        PetscInt Nfloc2 = d_species_offset[grid2 + 1] - d_species_offset[grid2], totDim2 = Nfloc2 * Nb;
        elemMat += d_numCells[grid2] * totDim2 * totDim2; // jump past grids, could be in an offset
      }
      elemMat += loc_elem * totDim * totDim; // index into local matrix & zero out
      for (int i = threadIdx.x + threadIdx.y * blockDim.x; i < totDim * totDim; i += blockDim.x * blockDim.y) elemMat[i] = 0;
    }
    __syncthreads();
    if (d_maps) {
      // reuse the space for fieldMats
      s_fieldMats = (PetscScalar(*)[LANDAU_MAX_NQ][LANDAU_MAX_NQ]) & smem[size];
      size += LANDAU_MAX_NQ * LANDAU_MAX_NQ;
      s_scale = (PetscReal(*)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE]) & smem[size];
      size += LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE;
      s_idx = (PetscInt(*)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE]) & smem[size];
      size += LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE; // this is too big, idx is an integer
    } else {
      s_fieldMats = NULL;
    }
    __syncthreads();
    landau_jac_kernel(num_grids, jpidx, nip_global, grid, xx, yy, ww, invJ, Nftot, nu_alpha, nu_beta, invMass, Eq_m, BB, DD, elemMat, d_maps, d_mat, *s_fieldMats, *s_scale, *s_idx, *s_g2, *s_g3, *s_gg2, *s_gg3, s_nu_alpha, s_nu_beta, s_invMass, s_f, s_dfx, s_dfy, d_f, d_dfdx, d_dfdy,
  #if LANDAU_DIM == 3
                      zz, s_dfz, d_dfdz,
  #endif
                      d_numCells, d_species_offset, d_mat_offset, d_ip_offset, d_ipf_offset, d_elem_offset);
  }
}

__global__ void __launch_bounds__(256, 4) landau_mass(const PetscInt dim, const PetscInt Nb, const PetscInt num_grids, const PetscReal d_w[], const PetscReal *const BB, const PetscReal *const DD, PetscScalar d_elem_mats[], P4estVertexMaps *d_maps[], PetscSplitCSRDataStructure d_mat, PetscReal shift, const PetscInt d_numCells[], const PetscInt d_species_offset[], const PetscInt d_mat_offset[], const PetscInt d_ip_offset[], const PetscInt d_elem_offset[])
{
  extern __shared__ PetscReal smem[];
  const PetscInt              Nq = blockDim.y, b_elem_idx = blockIdx.y, b_id = blockIdx.x;
  PetscScalar                *elemMat = NULL; /* my output */
  PetscInt                    fieldA, d, qj, q, idx, f, g, grid = 0, size = 0;
  PetscScalar(*s_fieldMats)[LANDAU_MAX_NQ][LANDAU_MAX_NQ];
  PetscReal(*s_scale)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE];
  PetscInt(*s_idx)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE];
  if (d_maps) {
    // reuse the space for fieldMats
    s_fieldMats = (PetscScalar(*)[LANDAU_MAX_NQ][LANDAU_MAX_NQ]) & smem[size];
    size += LANDAU_MAX_NQ * LANDAU_MAX_NQ;
    s_scale = (PetscReal(*)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE]) & smem[size];
    size += LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE;
    s_idx = (PetscInt(*)[LANDAU_MAX_NQ][LANDAU_MAX_Q_FACE]) & smem[size];
    size += LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE; // this is too big, idx is an integer
  } else {
    s_fieldMats = NULL;
  }
  while (b_elem_idx >= d_elem_offset[grid + 1]) grid++;
  {
    const PetscInt loc_Nf = d_species_offset[grid + 1] - d_species_offset[grid], loc_elem = b_elem_idx - d_elem_offset[grid];
    const PetscInt moffset = LAND_MOFFSET(b_id, grid, gridDim.x, num_grids, d_mat_offset), totDim = loc_Nf * Nq;
    if (d_elem_mats) {
      elemMat = d_elem_mats; // start a beginning
      for (PetscInt b_id2 = 0; b_id2 < b_id; b_id2++) {
        for (PetscInt grid2 = 0; grid2 < num_grids; grid2++) {
          PetscInt Nfloc2 = d_species_offset[grid2 + 1] - d_species_offset[grid2], totDim2 = Nfloc2 * Nb;
          elemMat += d_numCells[grid2] * totDim2 * totDim2; // jump past grids,could be in an offset
        }
      }
      for (PetscInt grid2 = 0; grid2 < grid; grid2++) {
        PetscInt Nfloc2 = d_species_offset[grid2 + 1] - d_species_offset[grid2], totDim2 = Nfloc2 * Nb;
        elemMat += d_numCells[grid2] * totDim2 * totDim2; // jump past grids,could be in an offset
      }
      elemMat += loc_elem * totDim * totDim;
      for (int i = threadIdx.x + threadIdx.y * blockDim.x; i < totDim * totDim; i += blockDim.x * blockDim.y) elemMat[i] = 0;
    }
    __syncthreads();
    /* FE mass matrix construction */
    for (fieldA = 0; fieldA < loc_Nf; fieldA++) {
      PetscScalar vals[LANDAU_MAX_Q_FACE * LANDAU_MAX_Q_FACE];
      PetscInt    nr, nc;
      for (f = threadIdx.y; f < Nb; f += blockDim.y) {
        for (g = threadIdx.x; g < Nb; g += blockDim.x) {
          PetscScalar t = 0;
          for (qj = 0; qj < Nq; qj++) {
            const PetscReal *BJq   = &BB[qj * Nb];
            const PetscInt   jpidx = d_ip_offset[grid] + qj + loc_elem * Nq;
            if (dim == 2) {
              t += BJq[f] * d_w[jpidx] * shift * BJq[g] * 2. * PETSC_PI;
            } else {
              t += BJq[f] * d_w[jpidx] * shift * BJq[g];
            }
          }
          if (elemMat) {
            const PetscInt fOff = (fieldA * Nb + f) * totDim + fieldA * Nb + g;
            elemMat[fOff] += t; // ????
          } else (*s_fieldMats)[f][g] = t;
        }
      }
      if (!elemMat) {
        const LandauIdx *const Idxs = &d_maps[grid]->gIdx[loc_elem][fieldA][0];
        __syncthreads();
        if (threadIdx.y == 0) {
          for (f = threadIdx.x; f < Nb; f += blockDim.x) {
            idx = Idxs[f];
            if (idx >= 0) {
              (*s_idx)[f][0]   = idx + moffset;
              (*s_scale)[f][0] = 1.;
            } else {
              idx = -idx - 1;
              for (q = 0; q < d_maps[grid]->num_face; q++) {
                if (d_maps[grid]->c_maps[idx][q].gid >= 0) (*s_idx)[f][q] = d_maps[grid]->c_maps[idx][q].gid + moffset;
                else (*s_idx)[f][q] = -1;
                (*s_scale)[f][q] = d_maps[grid]->c_maps[idx][q].scale;
              }
            }
          }
        }
        __syncthreads();
        for (f = threadIdx.y; f < Nb; f += blockDim.y) {
          idx = Idxs[f];
          if (idx >= 0) {
            nr = 1;
          } else {
            nr = d_maps[grid]->num_face;
          }
          for (g = threadIdx.x; g < Nb; g += blockDim.x) {
            idx = Idxs[g];
            if (idx >= 0) {
              nc = 1;
            } else {
              nc = d_maps[grid]->num_face;
            }
            for (q = 0; q < nr; q++) {
              for (d = 0; d < nc; d++) vals[q * nc + d] = (*s_scale)[f][q] * (*s_scale)[g][d] * (*s_fieldMats)[f][g];
            }
            static_cast<void>(MatSetValuesDevice(d_mat, nr, (*s_idx)[f], nc, (*s_idx)[g], vals, ADD_VALUES));
          }
        }
      }
      __syncthreads();
    }
  }
}

PetscErrorCode LandauCUDAJacobian(DM plex[], const PetscInt Nq, const PetscInt batch_sz, const PetscInt num_grids, const PetscInt a_numCells[], PetscReal a_Eq_m[], PetscScalar a_elem_closure[], const PetscScalar a_xarray[], const LandauStaticData *SData_d, const PetscReal shift, const PetscLogEvent events[], const PetscInt a_mat_offset[], const PetscInt a_species_offset[], Mat subJ[], Mat JacP)
{
  hipError_t                cerr;
  PetscInt                   Nb = Nq, dim, nip_global, num_cells_batch, elem_mat_size_tot;
  PetscInt                  *d_numCells, *d_species_offset, *d_mat_offset, *d_ip_offset, *d_ipf_offset, *d_elem_offset;
  PetscInt                   szf = sizeof(PetscReal), szs = sizeof(PetscScalar), Nftot = a_species_offset[num_grids];
  PetscReal                 *d_BB = NULL, *d_DD = NULL, *d_invJj = NULL, *d_nu_alpha = NULL, *d_nu_beta = NULL, *d_invMass = NULL, *d_Eq_m = NULL, *d_x = NULL, *d_y = NULL, *d_w = NULL;
  PetscScalar               *d_elem_mats = NULL, *d_vertex_f = NULL;
  PetscReal                 *d_f = NULL, *d_dfdx = NULL, *d_dfdy = NULL;
  #if LANDAU_DIM == 3
  PetscReal                 *d_dfdz = NULL, *d_z = NULL;
  #endif
  LandauCtx                 *ctx;
  PetscSplitCSRDataStructure d_mat = NULL;
  P4estVertexMaps          **d_maps, *maps[LANDAU_MAX_GRIDS];
  int                        nnn = 256 / Nq; // machine dependent
  PetscContainer             container;

  PetscFunctionBegin;
  PetscCall(PetscLogEventBegin(events[3], 0, 0, 0, 0));
  while (nnn & nnn - 1) nnn = nnn & nnn - 1;
  if (nnn > 16) nnn = 16;
  PetscCall(DMGetApplicationContext(plex[0], &ctx));
  PetscCheck(ctx, PETSC_COMM_SELF, PETSC_ERR_PLIB, "no context");
  PetscCall(DMGetDimension(plex[0], &dim));
  PetscCheck(dim == LANDAU_DIM, PETSC_COMM_SELF, PETSC_ERR_PLIB, "LANDAU_DIM %d != dim %" PetscInt_FMT, LANDAU_DIM, dim);
  if (ctx->gpu_assembly) {
    PetscCall(PetscObjectQuery((PetscObject)JacP, "assembly_maps", (PetscObject *)&container));
    if (container) {       // not here first call
      static int init = 0; // hack. just do every time, or put in setup (but that is in base class code), or add init_maps flag
      if (!init++) {
        P4estVertexMaps *h_maps = NULL;
        PetscCall(PetscContainerGetPointer(container, (void **)&h_maps));
        for (PetscInt grid = 0; grid < num_grids; grid++) {
          if (h_maps[grid].d_self) {
            maps[grid] = h_maps[grid].d_self;
          } else {
            SETERRQ(PETSC_COMM_SELF, PETSC_ERR_PLIB, "GPU assembly but no metadata in container");
          }
        }
        PetscCallCUDA(hipMemcpy(SData_d->maps, maps, num_grids * sizeof(P4estVertexMaps *), hipMemcpyHostToDevice));
      }
      d_maps = (P4estVertexMaps **)SData_d->maps;
      // this does the setup the first time called
      PetscCall(MatCUSPARSEGetDeviceMatWrite(JacP, &d_mat));
    } else {
      d_maps = NULL;
    }
  } else {
    container = NULL;
    d_maps    = NULL;
  }
  PetscCall(PetscLogEventEnd(events[3], 0, 0, 0, 0));
  {
    PetscInt elem_mat_size = 0;
    nip_global = num_cells_batch = 0;
    for (PetscInt grid = 0; grid < num_grids; grid++) {
      PetscInt Nfloc = a_species_offset[grid + 1] - a_species_offset[grid], totDim = Nfloc * Nb;
      nip_global += a_numCells[grid] * Nq;
      num_cells_batch += a_numCells[grid];                 // is in d_elem_offset, but not on host
      elem_mat_size += a_numCells[grid] * totDim * totDim; // could save in an offset here -- batch major ordering
    }
    elem_mat_size_tot = d_maps ? 0 : elem_mat_size;
  }
  dim3 dimGrid(batch_sz, num_cells_batch);
  if (elem_mat_size_tot) {
    PetscCallCUDA(hipMalloc((void **)&d_elem_mats, batch_sz * elem_mat_size_tot * szs)); // kernel output - first call is on CPU
  } else d_elem_mats = NULL;
  // create data
  d_BB = (PetscReal *)SData_d->B;
  d_DD = (PetscReal *)SData_d->D;
  if (a_elem_closure || a_xarray) { // form f and df
    PetscCall(PetscLogEventBegin(events[1], 0, 0, 0, 0));
    PetscCallCUDA(hipMemcpy(SData_d->Eq_m, a_Eq_m, Nftot * szf, hipMemcpyHostToDevice));
    d_invJj    = (PetscReal *)SData_d->invJ;
    d_nu_alpha = (PetscReal *)SData_d->alpha;
    d_nu_beta  = (PetscReal *)SData_d->beta;
    d_invMass  = (PetscReal *)SData_d->invMass;
    d_x        = (PetscReal *)SData_d->x;
    d_y        = (PetscReal *)SData_d->y;
    d_w        = (PetscReal *)SData_d->w;
    d_Eq_m     = (PetscReal *)SData_d->Eq_m;
    d_dfdx     = (PetscReal *)SData_d->dfdx;
    d_dfdy     = (PetscReal *)SData_d->dfdy;
  #if LANDAU_DIM == 3
    d_dfdz     = (PetscReal *)SData_d->dfdz;
    d_z        = (PetscReal *)SData_d->z;
  #endif
    d_f        = (PetscReal *)SData_d->f;
    // get a d_vertex_f
    if (a_elem_closure) {
      PetscInt closure_sz = 0; // argh, don't have this on the host!!!
      for (PetscInt grid = 0; grid < num_grids; grid++) {
        PetscInt nfloc = a_species_offset[grid + 1] - a_species_offset[grid];
        closure_sz += Nq * nfloc * a_numCells[grid];
      }
      closure_sz *= batch_sz;
      PetscCallCUDA(hipMalloc((void **)&d_vertex_f, closure_sz * sizeof(*a_elem_closure)));
      PetscCallCUDA(hipMemcpy(d_vertex_f, a_elem_closure, closure_sz * sizeof(*a_elem_closure), hipMemcpyHostToDevice));
    } else {
      d_vertex_f = (PetscScalar *)a_xarray;
    }
    PetscCall(PetscLogEventEnd(events[1], 0, 0, 0, 0));
  } else {
    d_w = (PetscReal *)SData_d->w; // mass just needs the weights
  }
  //
  d_numCells       = (PetscInt *)SData_d->NCells; // redundant -- remove
  d_species_offset = (PetscInt *)SData_d->species_offset;
  d_mat_offset     = (PetscInt *)SData_d->mat_offset;
  d_ip_offset      = (PetscInt *)SData_d->ip_offset;
  d_ipf_offset     = (PetscInt *)SData_d->ipf_offset;
  d_elem_offset    = (PetscInt *)SData_d->elem_offset;
  if (a_elem_closure || a_xarray) { // form f and df
    dim3 dimBlockFDF(nnn > Nftot ? Nftot : nnn, Nq), dimBlock((nip_global > nnn) ? nnn : nip_global, Nq);
    PetscCall(PetscLogEventBegin(events[8], 0, 0, 0, 0));
    PetscCall(PetscLogGpuTimeBegin());
    PetscCall(PetscInfo(plex[0], "Form F and dF/dx vectors: nip_global=%" PetscInt_FMT " num_grids=%" PetscInt_FMT "\n", nip_global, num_grids));
    landau_form_fdf<<<dimGrid, dimBlockFDF>>>(dim, Nb, num_grids, d_invJj, d_BB, d_DD, d_vertex_f, d_maps, d_f, d_dfdx, d_dfdy,
  #if LANDAU_DIM == 3
                                              d_dfdz,
  #endif
                                              d_numCells, d_species_offset, d_mat_offset, d_ip_offset, d_ipf_offset, d_elem_offset);
    PetscCUDACheckLaunch;
    PetscCall(PetscLogGpuFlops(batch_sz * nip_global * (PetscLogDouble)(2 * Nb * (1 + dim))));
    if (a_elem_closure) {
      PetscCallCUDA(hipFree(d_vertex_f));
      d_vertex_f = NULL;
    }
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogEventEnd(events[8], 0, 0, 0, 0));
    // Jacobian
    PetscCall(PetscLogEventBegin(events[4], 0, 0, 0, 0));
    PetscCall(PetscLogGpuTimeBegin());
    PetscCall(PetscLogGpuFlops(batch_sz * nip_global * (PetscLogDouble)(a_elem_closure ? (nip_global * (11 * Nftot + 4 * dim * dim) + 6 * Nftot * dim * dim * dim + 10 * Nftot * dim * dim + 4 * Nftot * dim + Nb * Nftot * Nb * Nq * dim * dim * 5) : Nb * Nftot * Nb * Nq * 4)));
    PetscInt ii = 2 * LANDAU_MAX_NQ * LANDAU_MAX_SPECIES * LANDAU_DIM * (1 + LANDAU_DIM) + 3 * LANDAU_MAX_SPECIES + (1 + LANDAU_DIM) * dimBlock.x * LANDAU_MAX_SPECIES + LANDAU_MAX_NQ * LANDAU_MAX_NQ + 2 * LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE;
    if (ii * szf >= 49152) {
      cerr = hipFuncSetAttribute(reinterpret_cast<const void*>(landau_jacobian), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
      PetscCallCUDA(cerr);
    }
    PetscCall(PetscInfo(plex[0], "Jacobian shared memory size: %" PetscInt_FMT " bytes, d_elem_mats=%p d_maps=%p\n", ii, d_elem_mats, d_maps));
    landau_jacobian<<<dimGrid, dimBlock, ii * szf>>>(nip_global, dim, Nb, num_grids, d_invJj, Nftot, d_nu_alpha, d_nu_beta, d_invMass, d_Eq_m, d_BB, d_DD, d_x, d_y, d_w, d_elem_mats, d_maps, d_mat, d_f, d_dfdx, d_dfdy,
  #if LANDAU_DIM == 3
                                                     d_z, d_dfdz,
  #endif
                                                     d_numCells, d_species_offset, d_mat_offset, d_ip_offset, d_ipf_offset, d_elem_offset);
    PetscCUDACheckLaunch; // has sync
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogEventEnd(events[4], 0, 0, 0, 0));
  } else { // mass
    dim3     dimBlock(nnn, Nq);
    PetscInt ii = LANDAU_MAX_NQ * LANDAU_MAX_NQ + 2 * LANDAU_MAX_NQ * LANDAU_MAX_Q_FACE;
    if (ii * szf >= 49152) {
      cerr = hipFuncSetAttribute(reinterpret_cast<const void*>(landau_mass), hipFuncAttributeMaxDynamicSharedMemorySize, 98304);
      PetscCallCUDA(cerr);
    }
    PetscCall(PetscInfo(plex[0], "Mass d_maps = %p. Nq=%" PetscInt_FMT ", vector size %d num_cells_batch=%" PetscInt_FMT ", %" PetscInt_FMT " shared memory words\n", d_maps, Nq, nnn, num_cells_batch, ii));
    PetscCall(PetscLogEventBegin(events[16], 0, 0, 0, 0));
    PetscCall(PetscLogGpuTimeBegin());
    landau_mass<<<dimGrid, dimBlock, ii * szf>>>(dim, Nb, num_grids, d_w, d_BB, d_DD, d_elem_mats, d_maps, d_mat, shift, d_numCells, d_species_offset, d_mat_offset, d_ip_offset, d_elem_offset);
    PetscCUDACheckLaunch; // has sync
    PetscCall(PetscLogGpuTimeEnd());
    PetscCall(PetscLogEventEnd(events[16], 0, 0, 0, 0));
  }
  // First time assembly with or without GPU assembly
  if (d_elem_mats) {
    PetscInt elem_mats_idx = 0;
    for (PetscInt b_id = 0; b_id < batch_sz; b_id++) {    // OpenMP (once)
      for (PetscInt grid = 0; grid < num_grids; grid++) { // elem_mats_idx += totDim*totDim*a_numCells[grid];
        const PetscInt     Nfloc = a_species_offset[grid + 1] - a_species_offset[grid], totDim = Nfloc * Nq;
        PetscScalar       *elemMats = NULL, *elMat;
        PetscSection       section, globalSection;
        PetscInt           cStart, cEnd, ej;
        PetscInt           moffset = LAND_MOFFSET(b_id, grid, batch_sz, num_grids, a_mat_offset), nloc, nzl, colbuf[1024], row;
        const PetscInt    *cols;
        const PetscScalar *vals;
        Mat                B = subJ[LAND_PACK_IDX(b_id, grid)];
        PetscCall(PetscLogEventBegin(events[5], 0, 0, 0, 0));
        PetscCall(DMPlexGetHeightStratum(plex[grid], 0, &cStart, &cEnd));
        PetscCall(DMGetLocalSection(plex[grid], &section));
        PetscCall(DMGetGlobalSection(plex[grid], &globalSection));
        PetscCall(PetscMalloc1(totDim * totDim * a_numCells[grid], &elemMats));
        PetscCallCUDA(hipMemcpy(elemMats, &d_elem_mats[elem_mats_idx], totDim * totDim * a_numCells[grid] * sizeof(*elemMats), hipMemcpyDeviceToHost));
        PetscCall(PetscLogEventEnd(events[5], 0, 0, 0, 0));
        PetscCall(PetscLogEventBegin(events[6], 0, 0, 0, 0));
        for (ej = cStart, elMat = elemMats; ej < cEnd; ++ej, elMat += totDim * totDim) {
          PetscCall(DMPlexMatSetClosure(plex[grid], section, globalSection, B, ej, elMat, ADD_VALUES));
          if (ej == -1) {
            int d, f;
            PetscCall(PetscPrintf(PETSC_COMM_SELF, "GPU Element matrix\n"));
            for (d = 0; d < totDim; ++d) {
              for (f = 0; f < totDim; ++f) PetscCall(PetscPrintf(PETSC_COMM_SELF, " %12.5e", PetscRealPart(elMat[d * totDim + f])));
              PetscCall(PetscPrintf(PETSC_COMM_SELF, "\n"));
            }
          }
        }
        PetscCall(PetscFree(elemMats));
        PetscCall(MatAssemblyBegin(B, MAT_FINAL_ASSEMBLY));
        PetscCall(MatAssemblyEnd(B, MAT_FINAL_ASSEMBLY));
        // move nest matrix to global JacP
        PetscCall(MatGetSize(B, &nloc, NULL));
        for (int i = 0; i < nloc; i++) {
          PetscCall(MatGetRow(B, i, &nzl, &cols, &vals));
          PetscCheck(nzl <= 1024, PetscObjectComm((PetscObject)B), PETSC_ERR_PLIB, "Row too big: %" PetscInt_FMT, nzl);
          for (int j = 0; j < nzl; j++) colbuf[j] = cols[j] + moffset;
          row = i + moffset;
          PetscCall(MatSetValues(JacP, 1, &row, nzl, colbuf, vals, ADD_VALUES));
          PetscCall(MatRestoreRow(B, i, &nzl, &cols, &vals));
        }
        PetscCall(MatDestroy(&B));
        PetscCall(PetscLogEventEnd(events[6], 0, 0, 0, 0));
        elem_mats_idx += totDim * totDim * a_numCells[grid]; // this can be a stored offset?
      }                                                      // grids
    }
    PetscCheck(elem_mats_idx == batch_sz * elem_mat_size_tot, PetscObjectComm((PetscObject)JacP), PETSC_ERR_PLIB, "elem_mats_idx != batch_sz*elem_mat_size_tot: %" PetscInt_FMT " %" PetscInt_FMT, elem_mats_idx, batch_sz * elem_mat_size_tot);
    PetscCallCUDA(hipFree(d_elem_mats));
  }

  PetscFunctionReturn(PETSC_SUCCESS);
}
#endif
