#include "hip/hip_runtime.h"
#include <../src/vec/is/sf/impls/basic/sfpack.h>

/* Map a thread id to an index in root/leaf space through a series of 3D subdomains. See PetscSFPackOpt. */
__device__ static inline PetscInt MapTidToIndex(const PetscInt *opt, PetscInt tid)
{
  PetscInt        i, j, k, m, n, r;
  const PetscInt *offset, *start, *dx, *dy, *X, *Y;

  n      = opt[0];
  offset = opt + 1;
  start  = opt + n + 2;
  dx     = opt + 2 * n + 2;
  dy     = opt + 3 * n + 2;
  X      = opt + 5 * n + 2;
  Y      = opt + 6 * n + 2;
  for (r = 0; r < n; r++) {
    if (tid < offset[r + 1]) break;
  }
  m = (tid - offset[r]);
  k = m / (dx[r] * dy[r]);
  j = (m - k * dx[r] * dy[r]) / dx[r];
  i = m - k * dx[r] * dy[r] - j * dx[r];

  return (start[r] + k * X[r] * Y[r] + j * X[r] + i);
}

/*====================================================================================*/
/*  Templated CUDA kernels for pack/unpack. The Op can be regular or atomic           */
/*====================================================================================*/

/* Suppose user calls PetscSFReduce(sf,unit,...) and <unit> is an MPI data type made of 16 PetscReals, then
   <Type> is PetscReal, which is the primitive type we operate on.
   <bs>   is 16, which says <unit> contains 16 primitive types.
   <BS>   is 8, which is the maximal SIMD width we will try to vectorize operations on <unit>.
   <EQ>   is 0, which is (bs == BS ? 1 : 0)

  If instead, <unit> has 8 PetscReals, then bs=8, BS=8, EQ=1, rendering MBS below to a compile time constant.
  For the common case in VecScatter, bs=1, BS=1, EQ=1, MBS=1, the inner for-loops below will be totally unrolled.
*/
template <class Type, PetscInt BS, PetscInt EQ>
__global__ static void d_Pack(PetscInt bs, PetscInt count, PetscInt start, const PetscInt *opt, const PetscInt *idx, const Type *data, Type *buf)
{
  PetscInt       i, s, t, tid = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscInt grid_size = gridDim.x * blockDim.x;
  const PetscInt M         = (EQ) ? 1 : bs / BS; /* If EQ, then M=1 enables compiler's const-propagation */
  const PetscInt MBS       = M * BS;             /* MBS=bs. We turn MBS into a compile-time const when EQ=1. */

  for (; tid < count; tid += grid_size) {
    /* opt != NULL ==> idx == NULL, i.e., the indices have patterns but not contiguous;
       opt == NULL && idx == NULL ==> the indices are contiguous;
     */
    t = (opt ? MapTidToIndex(opt, tid) : (idx ? idx[tid] : start + tid)) * MBS;
    s = tid * MBS;
    for (i = 0; i < MBS; i++) buf[s + i] = data[t + i];
  }
}

template <class Type, class Op, PetscInt BS, PetscInt EQ>
__global__ static void d_UnpackAndOp(PetscInt bs, PetscInt count, PetscInt start, const PetscInt *opt, const PetscInt *idx, Type *data, const Type *buf)
{
  PetscInt       i, s, t, tid = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscInt grid_size = gridDim.x * blockDim.x;
  const PetscInt M = (EQ) ? 1 : bs / BS, MBS = M * BS;
  Op             op;

  for (; tid < count; tid += grid_size) {
    t = (opt ? MapTidToIndex(opt, tid) : (idx ? idx[tid] : start + tid)) * MBS;
    s = tid * MBS;
    for (i = 0; i < MBS; i++) op(data[t + i], buf[s + i]);
  }
}

template <class Type, class Op, PetscInt BS, PetscInt EQ>
__global__ static void d_FetchAndOp(PetscInt bs, PetscInt count, PetscInt rootstart, const PetscInt *rootopt, const PetscInt *rootidx, Type *rootdata, Type *leafbuf)
{
  PetscInt       i, l, r, tid = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscInt grid_size = gridDim.x * blockDim.x;
  const PetscInt M = (EQ) ? 1 : bs / BS, MBS = M * BS;
  Op             op;

  for (; tid < count; tid += grid_size) {
    r = (rootopt ? MapTidToIndex(rootopt, tid) : (rootidx ? rootidx[tid] : rootstart + tid)) * MBS;
    l = tid * MBS;
    for (i = 0; i < MBS; i++) leafbuf[l + i] = op(rootdata[r + i], leafbuf[l + i]);
  }
}

template <class Type, class Op, PetscInt BS, PetscInt EQ>
__global__ static void d_ScatterAndOp(PetscInt bs, PetscInt count, PetscInt srcx, PetscInt srcy, PetscInt srcX, PetscInt srcY, PetscInt srcStart, const PetscInt *srcIdx, const Type *src, PetscInt dstx, PetscInt dsty, PetscInt dstX, PetscInt dstY, PetscInt dstStart, const PetscInt *dstIdx, Type *dst)
{
  PetscInt       i, j, k, s, t, tid = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscInt grid_size = gridDim.x * blockDim.x;
  const PetscInt M = (EQ) ? 1 : bs / BS, MBS = M * BS;
  Op             op;

  for (; tid < count; tid += grid_size) {
    if (!srcIdx) { /* src is either contiguous or 3D */
      k = tid / (srcx * srcy);
      j = (tid - k * srcx * srcy) / srcx;
      i = tid - k * srcx * srcy - j * srcx;
      s = srcStart + k * srcX * srcY + j * srcX + i;
    } else {
      s = srcIdx[tid];
    }

    if (!dstIdx) { /* dst is either contiguous or 3D */
      k = tid / (dstx * dsty);
      j = (tid - k * dstx * dsty) / dstx;
      i = tid - k * dstx * dsty - j * dstx;
      t = dstStart + k * dstX * dstY + j * dstX + i;
    } else {
      t = dstIdx[tid];
    }

    s *= MBS;
    t *= MBS;
    for (i = 0; i < MBS; i++) op(dst[t + i], src[s + i]);
  }
}

template <class Type, class Op, PetscInt BS, PetscInt EQ>
__global__ static void d_FetchAndOpLocal(PetscInt bs, PetscInt count, PetscInt rootstart, const PetscInt *rootopt, const PetscInt *rootidx, Type *rootdata, PetscInt leafstart, const PetscInt *leafopt, const PetscInt *leafidx, const Type *leafdata, Type *leafupdate)
{
  PetscInt       i, l, r, tid = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscInt grid_size = gridDim.x * blockDim.x;
  const PetscInt M = (EQ) ? 1 : bs / BS, MBS = M * BS;
  Op             op;

  for (; tid < count; tid += grid_size) {
    r = (rootopt ? MapTidToIndex(rootopt, tid) : (rootidx ? rootidx[tid] : rootstart + tid)) * MBS;
    l = (leafopt ? MapTidToIndex(leafopt, tid) : (leafidx ? leafidx[tid] : leafstart + tid)) * MBS;
    for (i = 0; i < MBS; i++) leafupdate[l + i] = op(rootdata[r + i], leafdata[l + i]);
  }
}

/*====================================================================================*/
/*                             Regular operations on device                           */
/*====================================================================================*/
template <typename Type>
struct Insert {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = y;
    return old;
  }
};
template <typename Type>
struct Add {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x += y;
    return old;
  }
};
template <typename Type>
struct Mult {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x *= y;
    return old;
  }
};
template <typename Type>
struct Min {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = PetscMin(x, y);
    return old;
  }
};
template <typename Type>
struct Max {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = PetscMax(x, y);
    return old;
  }
};
template <typename Type>
struct LAND {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = x && y;
    return old;
  }
};
template <typename Type>
struct LOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = x || y;
    return old;
  }
};
template <typename Type>
struct LXOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = !x != !y;
    return old;
  }
};
template <typename Type>
struct BAND {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = x & y;
    return old;
  }
};
template <typename Type>
struct BOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = x | y;
    return old;
  }
};
template <typename Type>
struct BXOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    x        = x ^ y;
    return old;
  }
};
template <typename Type>
struct Minloc {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    if (y.a < x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b, y.b);
    return old;
  }
};
template <typename Type>
struct Maxloc {
  __device__ Type operator()(Type &x, Type y) const
  {
    Type old = x;
    if (y.a > x.a) x = y;
    else if (y.a == x.a) x.b = min(x.b, y.b); /* See MPI MAXLOC */
    return old;
  }
};

/*====================================================================================*/
/*                             Atomic operations on device                            */
/*====================================================================================*/

/*
  Atomic Insert (exchange) operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.3:

  int atomicExch(int* address, int val);
  unsigned int atomicExch(unsigned int* address, unsigned int val);
  unsigned long long int atomicExch(unsigned long long int* address, unsigned long long int val);
  float atomicExch(float* address, float val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory and stores val back to memory at the same address. These two operations are
  performed in one atomic transaction. The function returns old.

  PETSc notes:

  It may be useful in PetscSFFetchAndOp with op = MPI_REPLACE.

  VecScatter with multiple entries scattered to the same location using INSERT_VALUES does not need
  atomic insertion, since it does not need the old value. A 32-bit or 64-bit store instruction should
  be atomic itself.

  With bs>1 and a unit > 64 bits, the current element-wise atomic approach can not guarantee the whole
  insertion is atomic. Hope no user codes rely on that.
*/
__device__ static double atomicExch(double *address, double val)
{
  return __longlong_as_double(atomicExch((ullint *)address, __double_as_longlong(val)));
}

__device__ static llint atomicExch(llint *address, llint val)
{
  return (llint)(atomicExch((ullint *)address, (ullint)val));
}

template <typename Type>
struct AtomicInsert {
  __device__ Type operator()(Type &x, Type y) const { return atomicExch(&x, y); }
};

#if defined(PETSC_HAVE_COMPLEX)
  #if defined(PETSC_USE_REAL_DOUBLE)
/* CUDA does not support 128-bit atomics. Users should not insert different 128-bit PetscComplex values to the same location */
template <>
struct AtomicInsert<PetscComplex> {
  __device__ PetscComplex operator()(PetscComplex &x, PetscComplex y) const
  {
    PetscComplex         old, *z = &old;
    double              *xp = (double *)&x, *yp = (double *)&y;
    AtomicInsert<double> op;
    z[0] = op(xp[0], yp[0]);
    z[1] = op(xp[1], yp[1]);
    return old; /* The returned value may not be atomic. It can be mix of two ops. Caller should discard it. */
  }
};
  #elif defined(PETSC_USE_REAL_SINGLE)
template <>
struct AtomicInsert<PetscComplex> {
  __device__ PetscComplex operator()(PetscComplex &x, PetscComplex y) const
  {
    double              *xp = (double *)&x, *yp = (double *)&y;
    AtomicInsert<double> op;
    return op(xp[0], yp[0]);
  }
};
  #endif
#endif

/*
  Atomic add operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.1:

  int atomicAdd(int* address, int val);
  unsigned int atomicAdd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAdd(unsigned long long int* address,unsigned long long int val);
  float atomicAdd(float* address, float val);
  double atomicAdd(double* address, double val);
  __half2 atomicAdd(__half2 *address, __half2 val);
  __half atomicAdd(__half *address, __half val);

  reads the 16-bit, 32-bit or 64-bit word old located at the address address in global or shared memory, computes (old + val),
  and stores the result back to memory at the same address. These three operations are performed in one atomic transaction. The
  function returns old.

  The 32-bit floating-point version of atomicAdd() is only supported by devices of compute capability 2.x and higher.
  The 64-bit floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and higher.
  The 32-bit __half2 floating-point version of atomicAdd() is only supported by devices of compute capability 6.x and
  higher. The atomicity of the __half2 add operation is guaranteed separately for each of the two __half elements;
  the entire __half2 is not guaranteed to be atomic as a single 32-bit access.
  The 16-bit __half floating-point version of atomicAdd() is only supported by devices of compute capability 7.x and higher.
*/
__device__ static llint atomicAdd(llint *address, llint val)
{
  return (llint)atomicAdd((ullint *)address, (ullint)val);
}

template <typename Type>
struct AtomicAdd {
  __device__ Type operator()(Type &x, Type y) const { return atomicAdd(&x, y); }
};

template <>
struct AtomicAdd<double> {
  __device__ double operator()(double &x, double y) const
  {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 600)
    return atomicAdd(&x, y);
#else
    double *address = &x, val = y;
    ullint *address_as_ull = (ullint *)address;
    ullint  old            = *address_as_ull, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __longlong_as_double(old);
#endif
  }
};

template <>
struct AtomicAdd<float> {
  __device__ float operator()(float &x, float y) const
  {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 200)
    return atomicAdd(&x, y);
#else
    float *address = &x, val = y;
    int   *address_as_int = (int *)address;
    int    old            = *address_as_int, assumed;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, __float_as_int(val + __int_as_float(assumed)));
      /* Note: uses integer comparison to avoid hang in case of NaN (since NaN !=NaN) */
    } while (assumed != old);
    return __int_as_float(old);
#endif
  }
};

#if defined(PETSC_HAVE_COMPLEX)
template <>
struct AtomicAdd<PetscComplex> {
  __device__ PetscComplex operator()(PetscComplex &x, PetscComplex y) const
  {
    PetscComplex         old, *z = &old;
    PetscReal           *xp = (PetscReal *)&x, *yp = (PetscReal *)&y;
    AtomicAdd<PetscReal> op;
    z[0] = op(xp[0], yp[0]);
    z[1] = op(xp[1], yp[1]);
    return old; /* The returned value may not be atomic. It can be mix of two ops. Caller should discard it. */
  }
};
#endif

/*
  Atomic Mult operations:

  CUDA has no atomicMult at all, so we build our own with atomicCAS
 */
#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMult(double *address, double val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    /* Other threads can access and modify value of *address_as_ull after the read above and before the write below */
    old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val * __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMult(float *address, float val)
{
  int *address_as_int = (int *)(address);
  int  old            = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(val * __int_as_float(assumed)));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

__device__ static int atomicMult(int *address, int val)
{
  int *address_as_int = (int *)(address);
  int  old            = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, val * assumed);
  } while (assumed != old);
  return (int)old;
}

__device__ static llint atomicMult(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(val * (llint)assumed));
  } while (assumed != old);
  return (llint)old;
}

template <typename Type>
struct AtomicMult {
  __device__ Type operator()(Type &x, Type y) const { return atomicMult(&x, y); }
};

/*
  Atomic Min/Max operations

  CUDA C Programming Guide V10.1 Chapter B.12.1.4~5:

  int atomicMin(int* address, int val);
  unsigned int atomicMin(unsigned int* address,unsigned int val);
  unsigned long long int atomicMin(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes the minimum of old and val, and stores the result back to memory
  at the same address. These three operations are performed in one atomic transaction.
  The function returns old.
  The 64-bit version of atomicMin() is only supported by devices of compute capability 3.5 and higher.

  atomicMax() is similar.
 */

#if defined(PETSC_USE_REAL_DOUBLE)
__device__ static double atomicMin(double *address, double val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMin(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__device__ static double atomicMax(double *address, double val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, __double_as_longlong(PetscMax(val, __longlong_as_double(assumed))));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#elif defined(PETSC_USE_REAL_SINGLE)
__device__ static float atomicMin(float *address, float val)
{
  int *address_as_int = (int *)(address);
  int  old            = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMin(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__device__ static float atomicMax(float *address, float val)
{
  int *address_as_int = (int *)(address);
  int  old            = *address_as_int, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_int, assumed, __float_as_int(PetscMax(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}
#endif

/*
  atomicMin/Max(long long *, long long) are not in Nvidia's documentation. But on OLCF Summit we found
  atomicMin/Max/And/Or/Xor(long long *, long long) in /sw/summit/cuda/10.1.243/include/sm_32_atomic_functions.h.
  This causes compilation errors with pgi compilers and 64-bit indices:
      error: function "atomicMin(long long *, long long)" has already been defined

  So we add extra conditions defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320)
*/
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320)
__device__ static llint atomicMin(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(PetscMin(val, (llint)assumed)));
  } while (assumed != old);
  return (llint)old;
}

__device__ static llint atomicMax(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(PetscMax(val, (llint)assumed)));
  } while (assumed != old);
  return (llint)old;
}
#endif

template <typename Type>
struct AtomicMin {
  __device__ Type operator()(Type &x, Type y) const { return atomicMin(&x, y); }
};
template <typename Type>
struct AtomicMax {
  __device__ Type operator()(Type &x, Type y) const { return atomicMax(&x, y); }
};

/*
  Atomic bitwise operations

  CUDA C Programming Guide V10.1 Chapter B.12.2.1 ~ B.12.2.3:

  int atomicAnd(int* address, int val);
  unsigned int atomicAnd(unsigned int* address,unsigned int val);
  unsigned long long int atomicAnd(unsigned long long int* address,unsigned long long int val);

  reads the 32-bit or 64-bit word old located at the address address in global or shared
  memory, computes (old & val), and stores the result back to memory at the same
  address. These three operations are performed in one atomic transaction.
  The function returns old.

  The 64-bit version of atomicAnd() is only supported by devices of compute capability 3.5 and higher.

  atomicOr() and atomicXor are similar.
*/

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 320) /* Why 320? see comments at atomicMin() above */
__device__ static llint atomicAnd(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(val & (llint)assumed));
  } while (assumed != old);
  return (llint)old;
}
__device__ static llint atomicOr(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(val | (llint)assumed));
  } while (assumed != old);
  return (llint)old;
}

__device__ static llint atomicXor(llint *address, llint val)
{
  ullint *address_as_ull = (ullint *)(address);
  ullint  old            = *address_as_ull, assumed;
  do {
    assumed = old;
    old     = atomicCAS(address_as_ull, assumed, (ullint)(val ^ (llint)assumed));
  } while (assumed != old);
  return (llint)old;
}
#endif

template <typename Type>
struct AtomicBAND {
  __device__ Type operator()(Type &x, Type y) const { return atomicAnd(&x, y); }
};
template <typename Type>
struct AtomicBOR {
  __device__ Type operator()(Type &x, Type y) const { return atomicOr(&x, y); }
};
template <typename Type>
struct AtomicBXOR {
  __device__ Type operator()(Type &x, Type y) const { return atomicXor(&x, y); }
};

/*
  Atomic logical operations:

  CUDA has no atomic logical operations at all. We support them on integer types.
*/

/* A template without definition makes any instantiation not using given specializations erroneous at compile time,
   which is what we want since we only support 32-bit and 64-bit integers.
 */
template <typename Type, class Op, int size /* sizeof(Type) */>
struct AtomicLogical;

template <typename Type, class Op>
struct AtomicLogical<Type, Op, 4> {
  __device__ Type operator()(Type &x, Type y) const
  {
    int *address_as_int = (int *)(&x);
    int  old            = *address_as_int, assumed;
    Op   op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_int, assumed, (int)(op((Type)assumed, y)));
    } while (assumed != old);
    return (Type)old;
  }
};

template <typename Type, class Op>
struct AtomicLogical<Type, Op, 8> {
  __device__ Type operator()(Type &x, Type y) const
  {
    ullint *address_as_ull = (ullint *)(&x);
    ullint  old            = *address_as_ull, assumed;
    Op      op;
    do {
      assumed = old;
      old     = atomicCAS(address_as_ull, assumed, (ullint)(op((Type)assumed, y)));
    } while (assumed != old);
    return (Type)old;
  }
};

/* Note land/lor/lxor below are different from LAND etc above. Here we pass arguments by value and return result of ops (not old value) */
template <typename Type>
struct land {
  __device__ Type operator()(Type x, Type y) { return x && y; }
};
template <typename Type>
struct lor {
  __device__ Type operator()(Type x, Type y) { return x || y; }
};
template <typename Type>
struct lxor {
  __device__ Type operator()(Type x, Type y) { return (!x != !y); }
};

template <typename Type>
struct AtomicLAND {
  __device__ Type operator()(Type &x, Type y) const
  {
    AtomicLogical<Type, land<Type>, sizeof(Type)> op;
    return op(x, y);
  }
};
template <typename Type>
struct AtomicLOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    AtomicLogical<Type, lor<Type>, sizeof(Type)> op;
    return op(x, y);
  }
};
template <typename Type>
struct AtomicLXOR {
  __device__ Type operator()(Type &x, Type y) const
  {
    AtomicLogical<Type, lxor<Type>, sizeof(Type)> op;
    return op(x, y);
  }
};

/*====================================================================================*/
/*  Wrapper functions of cuda kernels. Function pointers are stored in 'link'         */
/*====================================================================================*/
template <typename Type, PetscInt BS, PetscInt EQ>
static PetscErrorCode Pack(PetscSFLink link, PetscInt count, PetscInt start, PetscSFPackOpt opt, const PetscInt *idx, const void *data, void *buf)
{
  PetscInt        nthreads = 256;
  PetscInt        nblocks  = (count + nthreads - 1) / nthreads;
  const PetscInt *iarray   = opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  if (!opt && !idx) { /* It is a 'CUDA data to nvshmem buf' memory copy */
    PetscCallCUDA(hipMemcpyAsync(buf, (char *)data + start * link->unitbytes, count * link->unitbytes, hipMemcpyDeviceToDevice, link->stream));
  } else {
    nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);
    d_Pack<Type, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, start, iarray, idx, (const Type *)data, (Type *)buf);
    PetscCallCUDA(hipGetLastError());
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* To specialize UnpackAndOp for the hipMemcpyAsync() below. Usually if this is a contiguous memcpy, we use root/leafdirect and do
   not need UnpackAndOp. Only with nvshmem, we need this 'nvshmem buf to CUDA data' memory copy
*/
template <typename Type, PetscInt BS, PetscInt EQ>
static PetscErrorCode Unpack(PetscSFLink link, PetscInt count, PetscInt start, PetscSFPackOpt opt, const PetscInt *idx, void *data, const void *buf)
{
  PetscInt        nthreads = 256;
  PetscInt        nblocks  = (count + nthreads - 1) / nthreads;
  const PetscInt *iarray   = opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  if (!opt && !idx) { /* It is a 'nvshmem buf to CUDA data' memory copy */
    PetscCallCUDA(hipMemcpyAsync((char *)data + start * link->unitbytes, buf, count * link->unitbytes, hipMemcpyDeviceToDevice, link->stream));
  } else {
    nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);
    d_UnpackAndOp<Type, Insert<Type>, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, start, iarray, idx, (Type *)data, (const Type *)buf);
    PetscCallCUDA(hipGetLastError());
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

template <typename Type, class Op, PetscInt BS, PetscInt EQ>
static PetscErrorCode UnpackAndOp(PetscSFLink link, PetscInt count, PetscInt start, PetscSFPackOpt opt, const PetscInt *idx, void *data, const void *buf)
{
  PetscInt        nthreads = 256;
  PetscInt        nblocks  = (count + nthreads - 1) / nthreads;
  const PetscInt *iarray   = opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);
  d_UnpackAndOp<Type, Op, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, start, iarray, idx, (Type *)data, (const Type *)buf);
  PetscCallCUDA(hipGetLastError());
  PetscFunctionReturn(PETSC_SUCCESS);
}

template <typename Type, class Op, PetscInt BS, PetscInt EQ>
static PetscErrorCode FetchAndOp(PetscSFLink link, PetscInt count, PetscInt start, PetscSFPackOpt opt, const PetscInt *idx, void *data, void *buf)
{
  PetscInt        nthreads = 256;
  PetscInt        nblocks  = (count + nthreads - 1) / nthreads;
  const PetscInt *iarray   = opt ? opt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);
  d_FetchAndOp<Type, Op, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, start, iarray, idx, (Type *)data, (Type *)buf);
  PetscCallCUDA(hipGetLastError());
  PetscFunctionReturn(PETSC_SUCCESS);
}

template <typename Type, class Op, PetscInt BS, PetscInt EQ>
static PetscErrorCode ScatterAndOp(PetscSFLink link, PetscInt count, PetscInt srcStart, PetscSFPackOpt srcOpt, const PetscInt *srcIdx, const void *src, PetscInt dstStart, PetscSFPackOpt dstOpt, const PetscInt *dstIdx, void *dst)
{
  PetscInt nthreads = 256;
  PetscInt nblocks  = (count + nthreads - 1) / nthreads;
  PetscInt srcx = 0, srcy = 0, srcX = 0, srcY = 0, dstx = 0, dsty = 0, dstX = 0, dstY = 0;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);

  /* The 3D shape of source subdomain may be different than that of the destination, which makes it difficult to use CUDA 3D grid and block */
  if (srcOpt) {
    srcx     = srcOpt->dx[0];
    srcy     = srcOpt->dy[0];
    srcX     = srcOpt->X[0];
    srcY     = srcOpt->Y[0];
    srcStart = srcOpt->start[0];
    srcIdx   = NULL;
  } else if (!srcIdx) {
    srcx = srcX = count;
    srcy = srcY = 1;
  }

  if (dstOpt) {
    dstx     = dstOpt->dx[0];
    dsty     = dstOpt->dy[0];
    dstX     = dstOpt->X[0];
    dstY     = dstOpt->Y[0];
    dstStart = dstOpt->start[0];
    dstIdx   = NULL;
  } else if (!dstIdx) {
    dstx = dstX = count;
    dsty = dstY = 1;
  }

  d_ScatterAndOp<Type, Op, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, srcx, srcy, srcX, srcY, srcStart, srcIdx, (const Type *)src, dstx, dsty, dstX, dstY, dstStart, dstIdx, (Type *)dst);
  PetscCallCUDA(hipGetLastError());
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Specialization for Insert since we may use hipMemcpyAsync */
template <typename Type, PetscInt BS, PetscInt EQ>
static PetscErrorCode ScatterAndInsert(PetscSFLink link, PetscInt count, PetscInt srcStart, PetscSFPackOpt srcOpt, const PetscInt *srcIdx, const void *src, PetscInt dstStart, PetscSFPackOpt dstOpt, const PetscInt *dstIdx, void *dst)
{
  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  /*src and dst are contiguous */
  if ((!srcOpt && !srcIdx) && (!dstOpt && !dstIdx) && src != dst) {
    PetscCallCUDA(hipMemcpyAsync((Type *)dst + dstStart * link->bs, (const Type *)src + srcStart * link->bs, count * link->unitbytes, hipMemcpyDeviceToDevice, link->stream));
  } else {
    PetscCall(ScatterAndOp<Type, Insert<Type>, BS, EQ>(link, count, srcStart, srcOpt, srcIdx, src, dstStart, dstOpt, dstIdx, dst));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

template <typename Type, class Op, PetscInt BS, PetscInt EQ>
static PetscErrorCode FetchAndOpLocal(PetscSFLink link, PetscInt count, PetscInt rootstart, PetscSFPackOpt rootopt, const PetscInt *rootidx, void *rootdata, PetscInt leafstart, PetscSFPackOpt leafopt, const PetscInt *leafidx, const void *leafdata, void *leafupdate)
{
  PetscInt        nthreads = 256;
  PetscInt        nblocks  = (count + nthreads - 1) / nthreads;
  const PetscInt *rarray   = rootopt ? rootopt->array : NULL;
  const PetscInt *larray   = leafopt ? leafopt->array : NULL;

  PetscFunctionBegin;
  if (!count) PetscFunctionReturn(PETSC_SUCCESS);
  nblocks = PetscMin(nblocks, link->maxResidentThreadsPerGPU / nthreads);
  d_FetchAndOpLocal<Type, Op, BS, EQ><<<nblocks, nthreads, 0, link->stream>>>(link->bs, count, rootstart, rarray, rootidx, (Type *)rootdata, leafstart, larray, leafidx, (const Type *)leafdata, (Type *)leafupdate);
  PetscCallCUDA(hipGetLastError());
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*====================================================================================*/
/*  Init various types and instantiate pack/unpack function pointers                  */
/*====================================================================================*/
template <typename Type, PetscInt BS, PetscInt EQ>
static void PackInit_RealType(PetscSFLink link)
{
  /* Pack/unpack for remote communication */
  link->d_Pack            = Pack<Type, BS, EQ>;
  link->d_UnpackAndInsert = Unpack<Type, BS, EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type, Add<Type>, BS, EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_UnpackAndMin    = UnpackAndOp<Type, Min<Type>, BS, EQ>;
  link->d_UnpackAndMax    = UnpackAndOp<Type, Max<Type>, BS, EQ>;
  link->d_FetchAndAdd     = FetchAndOp<Type, Add<Type>, BS, EQ>;

  /* Scatter for local communication */
  link->d_ScatterAndInsert = ScatterAndInsert<Type, BS, EQ>; /* Has special optimizations */
  link->d_ScatterAndAdd    = ScatterAndOp<Type, Add<Type>, BS, EQ>;
  link->d_ScatterAndMult   = ScatterAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_ScatterAndMin    = ScatterAndOp<Type, Min<Type>, BS, EQ>;
  link->d_ScatterAndMax    = ScatterAndOp<Type, Max<Type>, BS, EQ>;
  link->d_FetchAndAddLocal = FetchAndOpLocal<Type, Add<Type>, BS, EQ>;

  /* Atomic versions when there are data-race possibilities */
  link->da_UnpackAndInsert = UnpackAndOp<Type, AtomicInsert<Type>, BS, EQ>;
  link->da_UnpackAndAdd    = UnpackAndOp<Type, AtomicAdd<Type>, BS, EQ>;
  link->da_UnpackAndMult   = UnpackAndOp<Type, AtomicMult<Type>, BS, EQ>;
  link->da_UnpackAndMin    = UnpackAndOp<Type, AtomicMin<Type>, BS, EQ>;
  link->da_UnpackAndMax    = UnpackAndOp<Type, AtomicMax<Type>, BS, EQ>;
  link->da_FetchAndAdd     = FetchAndOp<Type, AtomicAdd<Type>, BS, EQ>;

  link->da_ScatterAndInsert = ScatterAndOp<Type, AtomicInsert<Type>, BS, EQ>;
  link->da_ScatterAndAdd    = ScatterAndOp<Type, AtomicAdd<Type>, BS, EQ>;
  link->da_ScatterAndMult   = ScatterAndOp<Type, AtomicMult<Type>, BS, EQ>;
  link->da_ScatterAndMin    = ScatterAndOp<Type, AtomicMin<Type>, BS, EQ>;
  link->da_ScatterAndMax    = ScatterAndOp<Type, AtomicMax<Type>, BS, EQ>;
  link->da_FetchAndAddLocal = FetchAndOpLocal<Type, AtomicAdd<Type>, BS, EQ>;
}

/* Have this templated class to specialize for char integers */
template <typename Type, PetscInt BS, PetscInt EQ, PetscInt size /*sizeof(Type)*/>
struct PackInit_IntegerType_Atomic {
  static void Init(PetscSFLink link)
  {
    link->da_UnpackAndInsert = UnpackAndOp<Type, AtomicInsert<Type>, BS, EQ>;
    link->da_UnpackAndAdd    = UnpackAndOp<Type, AtomicAdd<Type>, BS, EQ>;
    link->da_UnpackAndMult   = UnpackAndOp<Type, AtomicMult<Type>, BS, EQ>;
    link->da_UnpackAndMin    = UnpackAndOp<Type, AtomicMin<Type>, BS, EQ>;
    link->da_UnpackAndMax    = UnpackAndOp<Type, AtomicMax<Type>, BS, EQ>;
    link->da_UnpackAndLAND   = UnpackAndOp<Type, AtomicLAND<Type>, BS, EQ>;
    link->da_UnpackAndLOR    = UnpackAndOp<Type, AtomicLOR<Type>, BS, EQ>;
    link->da_UnpackAndLXOR   = UnpackAndOp<Type, AtomicLXOR<Type>, BS, EQ>;
    link->da_UnpackAndBAND   = UnpackAndOp<Type, AtomicBAND<Type>, BS, EQ>;
    link->da_UnpackAndBOR    = UnpackAndOp<Type, AtomicBOR<Type>, BS, EQ>;
    link->da_UnpackAndBXOR   = UnpackAndOp<Type, AtomicBXOR<Type>, BS, EQ>;
    link->da_FetchAndAdd     = FetchAndOp<Type, AtomicAdd<Type>, BS, EQ>;

    link->da_ScatterAndInsert = ScatterAndOp<Type, AtomicInsert<Type>, BS, EQ>;
    link->da_ScatterAndAdd    = ScatterAndOp<Type, AtomicAdd<Type>, BS, EQ>;
    link->da_ScatterAndMult   = ScatterAndOp<Type, AtomicMult<Type>, BS, EQ>;
    link->da_ScatterAndMin    = ScatterAndOp<Type, AtomicMin<Type>, BS, EQ>;
    link->da_ScatterAndMax    = ScatterAndOp<Type, AtomicMax<Type>, BS, EQ>;
    link->da_ScatterAndLAND   = ScatterAndOp<Type, AtomicLAND<Type>, BS, EQ>;
    link->da_ScatterAndLOR    = ScatterAndOp<Type, AtomicLOR<Type>, BS, EQ>;
    link->da_ScatterAndLXOR   = ScatterAndOp<Type, AtomicLXOR<Type>, BS, EQ>;
    link->da_ScatterAndBAND   = ScatterAndOp<Type, AtomicBAND<Type>, BS, EQ>;
    link->da_ScatterAndBOR    = ScatterAndOp<Type, AtomicBOR<Type>, BS, EQ>;
    link->da_ScatterAndBXOR   = ScatterAndOp<Type, AtomicBXOR<Type>, BS, EQ>;
    link->da_FetchAndAddLocal = FetchAndOpLocal<Type, AtomicAdd<Type>, BS, EQ>;
  }
};

/* CUDA does not support atomics on chars. It is TBD in PETSc. */
template <typename Type, PetscInt BS, PetscInt EQ>
struct PackInit_IntegerType_Atomic<Type, BS, EQ, 1> {
  static void Init(PetscSFLink)
  { /* Nothing to leave function pointers NULL */
  }
};

template <typename Type, PetscInt BS, PetscInt EQ>
static void PackInit_IntegerType(PetscSFLink link)
{
  link->d_Pack            = Pack<Type, BS, EQ>;
  link->d_UnpackAndInsert = Unpack<Type, BS, EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type, Add<Type>, BS, EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_UnpackAndMin    = UnpackAndOp<Type, Min<Type>, BS, EQ>;
  link->d_UnpackAndMax    = UnpackAndOp<Type, Max<Type>, BS, EQ>;
  link->d_UnpackAndLAND   = UnpackAndOp<Type, LAND<Type>, BS, EQ>;
  link->d_UnpackAndLOR    = UnpackAndOp<Type, LOR<Type>, BS, EQ>;
  link->d_UnpackAndLXOR   = UnpackAndOp<Type, LXOR<Type>, BS, EQ>;
  link->d_UnpackAndBAND   = UnpackAndOp<Type, BAND<Type>, BS, EQ>;
  link->d_UnpackAndBOR    = UnpackAndOp<Type, BOR<Type>, BS, EQ>;
  link->d_UnpackAndBXOR   = UnpackAndOp<Type, BXOR<Type>, BS, EQ>;
  link->d_FetchAndAdd     = FetchAndOp<Type, Add<Type>, BS, EQ>;

  link->d_ScatterAndInsert = ScatterAndInsert<Type, BS, EQ>;
  link->d_ScatterAndAdd    = ScatterAndOp<Type, Add<Type>, BS, EQ>;
  link->d_ScatterAndMult   = ScatterAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_ScatterAndMin    = ScatterAndOp<Type, Min<Type>, BS, EQ>;
  link->d_ScatterAndMax    = ScatterAndOp<Type, Max<Type>, BS, EQ>;
  link->d_ScatterAndLAND   = ScatterAndOp<Type, LAND<Type>, BS, EQ>;
  link->d_ScatterAndLOR    = ScatterAndOp<Type, LOR<Type>, BS, EQ>;
  link->d_ScatterAndLXOR   = ScatterAndOp<Type, LXOR<Type>, BS, EQ>;
  link->d_ScatterAndBAND   = ScatterAndOp<Type, BAND<Type>, BS, EQ>;
  link->d_ScatterAndBOR    = ScatterAndOp<Type, BOR<Type>, BS, EQ>;
  link->d_ScatterAndBXOR   = ScatterAndOp<Type, BXOR<Type>, BS, EQ>;
  link->d_FetchAndAddLocal = FetchAndOpLocal<Type, Add<Type>, BS, EQ>;
  PackInit_IntegerType_Atomic<Type, BS, EQ, sizeof(Type)>::Init(link);
}

#if defined(PETSC_HAVE_COMPLEX)
template <typename Type, PetscInt BS, PetscInt EQ>
static void PackInit_ComplexType(PetscSFLink link)
{
  link->d_Pack            = Pack<Type, BS, EQ>;
  link->d_UnpackAndInsert = Unpack<Type, BS, EQ>;
  link->d_UnpackAndAdd    = UnpackAndOp<Type, Add<Type>, BS, EQ>;
  link->d_UnpackAndMult   = UnpackAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_FetchAndAdd     = FetchAndOp<Type, Add<Type>, BS, EQ>;

  link->d_ScatterAndInsert = ScatterAndInsert<Type, BS, EQ>;
  link->d_ScatterAndAdd    = ScatterAndOp<Type, Add<Type>, BS, EQ>;
  link->d_ScatterAndMult   = ScatterAndOp<Type, Mult<Type>, BS, EQ>;
  link->d_FetchAndAddLocal = FetchAndOpLocal<Type, Add<Type>, BS, EQ>;

  link->da_UnpackAndInsert = UnpackAndOp<Type, AtomicInsert<Type>, BS, EQ>;
  link->da_UnpackAndAdd    = UnpackAndOp<Type, AtomicAdd<Type>, BS, EQ>;
  link->da_UnpackAndMult   = NULL; /* Not implemented yet */
  link->da_FetchAndAdd     = NULL; /* Return value of atomicAdd on complex is not atomic */

  link->da_ScatterAndInsert = ScatterAndOp<Type, AtomicInsert<Type>, BS, EQ>;
  link->da_ScatterAndAdd    = ScatterAndOp<Type, AtomicAdd<Type>, BS, EQ>;
}
#endif

typedef signed char   SignedChar;
typedef unsigned char UnsignedChar;
typedef struct {
  int a;
  int b;
} PairInt;
typedef struct {
  PetscInt a;
  PetscInt b;
} PairPetscInt;

template <typename Type>
static void PackInit_PairType(PetscSFLink link)
{
  link->d_Pack            = Pack<Type, 1, 1>;
  link->d_UnpackAndInsert = Unpack<Type, 1, 1>;
  link->d_UnpackAndMaxloc = UnpackAndOp<Type, Maxloc<Type>, 1, 1>;
  link->d_UnpackAndMinloc = UnpackAndOp<Type, Minloc<Type>, 1, 1>;

  link->d_ScatterAndInsert = ScatterAndOp<Type, Insert<Type>, 1, 1>;
  link->d_ScatterAndMaxloc = ScatterAndOp<Type, Maxloc<Type>, 1, 1>;
  link->d_ScatterAndMinloc = ScatterAndOp<Type, Minloc<Type>, 1, 1>;
  /* Atomics for pair types are not implemented yet */
}

template <typename Type, PetscInt BS, PetscInt EQ>
static void PackInit_DumbType(PetscSFLink link)
{
  link->d_Pack             = Pack<Type, BS, EQ>;
  link->d_UnpackAndInsert  = Unpack<Type, BS, EQ>;
  link->d_ScatterAndInsert = ScatterAndInsert<Type, BS, EQ>;
  /* Atomics for dumb types are not implemented yet */
}

/* Some device-specific utilities */
static PetscErrorCode PetscSFLinkSyncDevice_CUDA(PetscSFLink)
{
  PetscFunctionBegin;
  PetscCallCUDA(hipDeviceSynchronize());
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode PetscSFLinkSyncStream_CUDA(PetscSFLink link)
{
  PetscFunctionBegin;
  PetscCallCUDA(hipStreamSynchronize(link->stream));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode PetscSFLinkMemcpy_CUDA(PetscSFLink link, PetscMemType dstmtype, void *dst, PetscMemType srcmtype, const void *src, size_t n)
{
  PetscFunctionBegin;
  enum hipMemcpyKind kinds[2][2] = {
    {hipMemcpyHostToHost,   hipMemcpyHostToDevice  },
    {hipMemcpyDeviceToHost, hipMemcpyDeviceToDevice}
  };

  if (n) {
    if (PetscMemTypeHost(dstmtype) && PetscMemTypeHost(srcmtype)) { /* Separate HostToHost so that pure-cpu code won't call cuda runtime */
      PetscCall(PetscMemcpy(dst, src, n));
    } else {
      int stype = PetscMemTypeDevice(srcmtype) ? 1 : 0;
      int dtype = PetscMemTypeDevice(dstmtype) ? 1 : 0;
      PetscCallCUDA(hipMemcpyAsync(dst, src, n, kinds[stype][dtype], link->stream));
    }
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscSFMalloc_CUDA(PetscMemType mtype, size_t size, void **ptr)
{
  PetscFunctionBegin;
  if (PetscMemTypeHost(mtype)) PetscCall(PetscMalloc(size, ptr));
  else if (PetscMemTypeDevice(mtype)) {
    PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
    PetscCallCUDA(hipMalloc(ptr, size));
  } else SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_WRONG, "Wrong PetscMemType %d", (int)mtype);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscSFFree_CUDA(PetscMemType mtype, void *ptr)
{
  PetscFunctionBegin;
  if (PetscMemTypeHost(mtype)) PetscCall(PetscFree(ptr));
  else if (PetscMemTypeDevice(mtype)) PetscCallCUDA(hipFree(ptr));
  else SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_WRONG, "Wrong PetscMemType %d", (int)mtype);
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Destructor when the link uses MPI for communication on CUDA device */
static PetscErrorCode PetscSFLinkDestroy_MPI_CUDA(PetscSF, PetscSFLink link)
{
  PetscFunctionBegin;
  for (int i = PETSCSF_LOCAL; i <= PETSCSF_REMOTE; i++) {
    PetscCallCUDA(hipFree(link->rootbuf_alloc[i][PETSC_MEMTYPE_DEVICE]));
    PetscCallCUDA(hipFree(link->leafbuf_alloc[i][PETSC_MEMTYPE_DEVICE]));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Some fields of link are initialized by PetscSFPackSetUp_Host. This routine only does what needed on device */
PetscErrorCode PetscSFLinkSetUp_CUDA(PetscSF sf, PetscSFLink link, MPI_Datatype unit)
{
  PetscInt  nSignedChar = 0, nUnsignedChar = 0, nInt = 0, nPetscInt = 0, nPetscReal = 0;
  PetscBool is2Int, is2PetscInt;
#if defined(PETSC_HAVE_COMPLEX)
  PetscInt nPetscComplex = 0;
#endif

  PetscFunctionBegin;
  if (link->deviceinited) PetscFunctionReturn(PETSC_SUCCESS);
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPI_SIGNED_CHAR, &nSignedChar));
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPI_UNSIGNED_CHAR, &nUnsignedChar));
  /* MPI_CHAR is treated below as a dumb type that does not support reduction according to MPI standard */
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPI_INT, &nInt));
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPIU_INT, &nPetscInt));
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPIU_REAL, &nPetscReal));
#if defined(PETSC_HAVE_COMPLEX)
  PetscCall(MPIPetsc_Type_compare_contig(unit, MPIU_COMPLEX, &nPetscComplex));
#endif
  PetscCall(MPIPetsc_Type_compare(unit, MPI_2INT, &is2Int));
  PetscCall(MPIPetsc_Type_compare(unit, MPIU_2INT, &is2PetscInt));

  if (is2Int) {
    PackInit_PairType<PairInt>(link);
  } else if (is2PetscInt) { /* TODO: when is2PetscInt and nPetscInt=2, we don't know which path to take. The two paths support different ops. */
    PackInit_PairType<PairPetscInt>(link);
  } else if (nPetscReal) {
#if !defined(PETSC_HAVE_DEVICE)
    if (nPetscReal == 8) PackInit_RealType<PetscReal, 8, 1>(link);
    else if (nPetscReal % 8 == 0) PackInit_RealType<PetscReal, 8, 0>(link);
    else if (nPetscReal == 4) PackInit_RealType<PetscReal, 4, 1>(link);
    else if (nPetscReal % 4 == 0) PackInit_RealType<PetscReal, 4, 0>(link);
    else if (nPetscReal == 2) PackInit_RealType<PetscReal, 2, 1>(link);
    else if (nPetscReal % 2 == 0) PackInit_RealType<PetscReal, 2, 0>(link);
    else if (nPetscReal == 1) PackInit_RealType<PetscReal, 1, 1>(link);
    else if (nPetscReal % 1 == 0)
#endif
      PackInit_RealType<PetscReal, 1, 0>(link);
  } else if (nPetscInt && sizeof(PetscInt) == sizeof(llint)) {
#if !defined(PETSC_HAVE_DEVICE)
    if (nPetscInt == 8) PackInit_IntegerType<llint, 8, 1>(link);
    else if (nPetscInt % 8 == 0) PackInit_IntegerType<llint, 8, 0>(link);
    else if (nPetscInt == 4) PackInit_IntegerType<llint, 4, 1>(link);
    else if (nPetscInt % 4 == 0) PackInit_IntegerType<llint, 4, 0>(link);
    else if (nPetscInt == 2) PackInit_IntegerType<llint, 2, 1>(link);
    else if (nPetscInt % 2 == 0) PackInit_IntegerType<llint, 2, 0>(link);
    else if (nPetscInt == 1) PackInit_IntegerType<llint, 1, 1>(link);
    else if (nPetscInt % 1 == 0)
#endif
      PackInit_IntegerType<llint, 1, 0>(link);
  } else if (nInt) {
#if !defined(PETSC_HAVE_DEVICE)
    if (nInt == 8) PackInit_IntegerType<int, 8, 1>(link);
    else if (nInt % 8 == 0) PackInit_IntegerType<int, 8, 0>(link);
    else if (nInt == 4) PackInit_IntegerType<int, 4, 1>(link);
    else if (nInt % 4 == 0) PackInit_IntegerType<int, 4, 0>(link);
    else if (nInt == 2) PackInit_IntegerType<int, 2, 1>(link);
    else if (nInt % 2 == 0) PackInit_IntegerType<int, 2, 0>(link);
    else if (nInt == 1) PackInit_IntegerType<int, 1, 1>(link);
    else if (nInt % 1 == 0)
#endif
      PackInit_IntegerType<int, 1, 0>(link);
  } else if (nSignedChar) {
#if !defined(PETSC_HAVE_DEVICE)
    if (nSignedChar == 8) PackInit_IntegerType<SignedChar, 8, 1>(link);
    else if (nSignedChar % 8 == 0) PackInit_IntegerType<SignedChar, 8, 0>(link);
    else if (nSignedChar == 4) PackInit_IntegerType<SignedChar, 4, 1>(link);
    else if (nSignedChar % 4 == 0) PackInit_IntegerType<SignedChar, 4, 0>(link);
    else if (nSignedChar == 2) PackInit_IntegerType<SignedChar, 2, 1>(link);
    else if (nSignedChar % 2 == 0) PackInit_IntegerType<SignedChar, 2, 0>(link);
    else if (nSignedChar == 1) PackInit_IntegerType<SignedChar, 1, 1>(link);
    else if (nSignedChar % 1 == 0)
#endif
      PackInit_IntegerType<SignedChar, 1, 0>(link);
  } else if (nUnsignedChar) {
#if !defined(PETSC_HAVE_DEVICE)
    if (nUnsignedChar == 8) PackInit_IntegerType<UnsignedChar, 8, 1>(link);
    else if (nUnsignedChar % 8 == 0) PackInit_IntegerType<UnsignedChar, 8, 0>(link);
    else if (nUnsignedChar == 4) PackInit_IntegerType<UnsignedChar, 4, 1>(link);
    else if (nUnsignedChar % 4 == 0) PackInit_IntegerType<UnsignedChar, 4, 0>(link);
    else if (nUnsignedChar == 2) PackInit_IntegerType<UnsignedChar, 2, 1>(link);
    else if (nUnsignedChar % 2 == 0) PackInit_IntegerType<UnsignedChar, 2, 0>(link);
    else if (nUnsignedChar == 1) PackInit_IntegerType<UnsignedChar, 1, 1>(link);
    else if (nUnsignedChar % 1 == 0)
#endif
      PackInit_IntegerType<UnsignedChar, 1, 0>(link);
#if defined(PETSC_HAVE_COMPLEX)
  } else if (nPetscComplex) {
  #if !defined(PETSC_HAVE_DEVICE)
    if (nPetscComplex == 8) PackInit_ComplexType<PetscComplex, 8, 1>(link);
    else if (nPetscComplex % 8 == 0) PackInit_ComplexType<PetscComplex, 8, 0>(link);
    else if (nPetscComplex == 4) PackInit_ComplexType<PetscComplex, 4, 1>(link);
    else if (nPetscComplex % 4 == 0) PackInit_ComplexType<PetscComplex, 4, 0>(link);
    else if (nPetscComplex == 2) PackInit_ComplexType<PetscComplex, 2, 1>(link);
    else if (nPetscComplex % 2 == 0) PackInit_ComplexType<PetscComplex, 2, 0>(link);
    else if (nPetscComplex == 1) PackInit_ComplexType<PetscComplex, 1, 1>(link);
    else if (nPetscComplex % 1 == 0)
  #endif
      PackInit_ComplexType<PetscComplex, 1, 0>(link);
#endif
  } else {
    MPI_Aint lb, nbyte;
    PetscCallMPI(MPI_Type_get_extent(unit, &lb, &nbyte));
    PetscCheck(lb == 0, PETSC_COMM_SELF, PETSC_ERR_SUP, "Datatype with nonzero lower bound %ld", (long)lb);
    if (nbyte % sizeof(int)) { /* If the type size is not multiple of int */
#if !defined(PETSC_HAVE_DEVICE)
      if (nbyte == 4) PackInit_DumbType<char, 4, 1>(link);
      else if (nbyte % 4 == 0) PackInit_DumbType<char, 4, 0>(link);
      else if (nbyte == 2) PackInit_DumbType<char, 2, 1>(link);
      else if (nbyte % 2 == 0) PackInit_DumbType<char, 2, 0>(link);
      else if (nbyte == 1) PackInit_DumbType<char, 1, 1>(link);
      else if (nbyte % 1 == 0)
#endif
        PackInit_DumbType<char, 1, 0>(link);
    } else {
      nInt = nbyte / sizeof(int);
#if !defined(PETSC_HAVE_DEVICE)
      if (nInt == 8) PackInit_DumbType<int, 8, 1>(link);
      else if (nInt % 8 == 0) PackInit_DumbType<int, 8, 0>(link);
      else if (nInt == 4) PackInit_DumbType<int, 4, 1>(link);
      else if (nInt % 4 == 0) PackInit_DumbType<int, 4, 0>(link);
      else if (nInt == 2) PackInit_DumbType<int, 2, 1>(link);
      else if (nInt % 2 == 0) PackInit_DumbType<int, 2, 0>(link);
      else if (nInt == 1) PackInit_DumbType<int, 1, 1>(link);
      else if (nInt % 1 == 0)
#endif
        PackInit_DumbType<int, 1, 0>(link);
    }
  }

  if (!sf->maxResidentThreadsPerGPU) { /* Not initialized */
    int                   device;
    struct hipDeviceProp_t props;
    PetscCallCUDA(hipGetDevice(&device));
    PetscCallCUDA(hipGetDeviceProperties(&props, device));
    sf->maxResidentThreadsPerGPU = props.maxThreadsPerMultiProcessor * props.multiProcessorCount;
  }
  link->maxResidentThreadsPerGPU = sf->maxResidentThreadsPerGPU;

  link->stream       = PetscDefaultCudaStream;
  link->Destroy      = PetscSFLinkDestroy_MPI_CUDA;
  link->SyncDevice   = PetscSFLinkSyncDevice_CUDA;
  link->SyncStream   = PetscSFLinkSyncStream_CUDA;
  link->Memcpy       = PetscSFLinkMemcpy_CUDA;
  link->deviceinited = PETSC_TRUE;
  PetscFunctionReturn(PETSC_SUCCESS);
}
