#include "hip/hip_runtime.h"
#include <petsc/private/cudavecimpl.h>
#include <../src/vec/is/sf/impls/basic/sfpack.h>
#include <mpi.h>
#include <nvshmem.h>
#include <nvshmemx.h>

PetscErrorCode PetscNvshmemInitializeCheck(void)
{
  PetscFunctionBegin;
  if (!PetscNvshmemInitialized) { /* Note NVSHMEM does not provide a routine to check whether it is initialized */
    nvshmemx_init_attr_t attr;
    attr.mpi_comm = &PETSC_COMM_WORLD;
    PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
    PetscCall(nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr));
    PetscNvshmemInitialized = PETSC_TRUE;
    PetscBeganNvshmem       = PETSC_TRUE;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemMalloc(size_t size, void **ptr)
{
  PetscFunctionBegin;
  PetscCall(PetscNvshmemInitializeCheck());
  *ptr = nvshmem_malloc(size);
  PetscCheck(*ptr, PETSC_COMM_SELF, PETSC_ERR_ARG_WRONG, "nvshmem_malloc() failed to allocate %zu bytes", size);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemCalloc(size_t size, void **ptr)
{
  PetscFunctionBegin;
  PetscCall(PetscNvshmemInitializeCheck());
  *ptr = nvshmem_calloc(size, 1);
  PetscCheck(*ptr, PETSC_COMM_SELF, PETSC_ERR_ARG_WRONG, "nvshmem_calloc() failed to allocate %zu bytes", size);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemFree_Private(void *ptr)
{
  PetscFunctionBegin;
  nvshmem_free(ptr);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemFinalize(void)
{
  PetscFunctionBegin;
  nvshmem_finalize();
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Free nvshmem related fields in the SF */
PetscErrorCode PetscSFReset_Basic_NVSHMEM(PetscSF sf)
{
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;

  PetscFunctionBegin;
  PetscCall(PetscFree2(bas->leafsigdisp, bas->leafbufdisp));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, bas->leafbufdisp_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, bas->leafsigdisp_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, bas->iranks_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, bas->ioffset_d));

  PetscCall(PetscFree2(sf->rootsigdisp, sf->rootbufdisp));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, sf->rootbufdisp_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, sf->rootsigdisp_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, sf->ranks_d));
  PetscCall(PetscSFFree(sf, PETSC_MEMTYPE_CUDA, sf->roffset_d));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Set up NVSHMEM related fields for an SF of type SFBASIC (only after PetscSFSetup_Basic() already set up dependent fields) */
static PetscErrorCode PetscSFSetUp_Basic_NVSHMEM(PetscSF sf)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  PetscInt       i, nRemoteRootRanks, nRemoteLeafRanks;
  PetscMPIInt    tag;
  MPI_Comm       comm;
  MPI_Request   *rootreqs, *leafreqs;
  PetscInt       tmp, stmp[4], rtmp[4]; /* tmps for send/recv buffers */

  PetscFunctionBegin;
  PetscCall(PetscObjectGetComm((PetscObject)sf, &comm));
  PetscCall(PetscObjectGetNewTag((PetscObject)sf, &tag));

  nRemoteRootRanks      = sf->nranks - sf->ndranks;
  nRemoteLeafRanks      = bas->niranks - bas->ndiranks;
  sf->nRemoteRootRanks  = nRemoteRootRanks;
  bas->nRemoteLeafRanks = nRemoteLeafRanks;

  PetscCall(PetscMalloc2(nRemoteLeafRanks, &rootreqs, nRemoteRootRanks, &leafreqs));

  stmp[0] = nRemoteRootRanks;
  stmp[1] = sf->leafbuflen[PETSCSF_REMOTE];
  stmp[2] = nRemoteLeafRanks;
  stmp[3] = bas->rootbuflen[PETSCSF_REMOTE];

  PetscCall(MPIU_Allreduce(stmp, rtmp, 4, MPIU_INT, MPI_MAX, comm));

  sf->nRemoteRootRanksMax  = rtmp[0];
  sf->leafbuflen_rmax      = rtmp[1];
  bas->nRemoteLeafRanksMax = rtmp[2];
  bas->rootbuflen_rmax     = rtmp[3];

  /* Total four rounds of MPI communications to set up the nvshmem fields */

  /* Root ranks to leaf ranks: send info about rootsigdisp[] and rootbufdisp[] */
  PetscCall(PetscMalloc2(nRemoteRootRanks, &sf->rootsigdisp, nRemoteRootRanks, &sf->rootbufdisp));
  for (i = 0; i < nRemoteRootRanks; i++) PetscCallMPI(MPI_Irecv(&sf->rootsigdisp[i], 1, MPIU_INT, sf->ranks[i + sf->ndranks], tag, comm, &leafreqs[i])); /* Leaves recv */
  for (i = 0; i < nRemoteLeafRanks; i++) PetscCallMPI(MPI_Send(&i, 1, MPIU_INT, bas->iranks[i + bas->ndiranks], tag, comm));                             /* Roots send. Note i changes, so we use MPI_Send. */
  PetscCallMPI(MPI_Waitall(nRemoteRootRanks, leafreqs, MPI_STATUSES_IGNORE));

  for (i = 0; i < nRemoteRootRanks; i++) PetscCallMPI(MPI_Irecv(&sf->rootbufdisp[i], 1, MPIU_INT, sf->ranks[i + sf->ndranks], tag, comm, &leafreqs[i])); /* Leaves recv */
  for (i = 0; i < nRemoteLeafRanks; i++) {
    tmp = bas->ioffset[i + bas->ndiranks] - bas->ioffset[bas->ndiranks];
    PetscCallMPI(MPI_Send(&tmp, 1, MPIU_INT, bas->iranks[i + bas->ndiranks], tag, comm)); /* Roots send. Note tmp changes, so we use MPI_Send. */
  }
  PetscCallMPI(MPI_Waitall(nRemoteRootRanks, leafreqs, MPI_STATUSES_IGNORE));

  PetscCallCUDA(hipMalloc((void **)&sf->rootbufdisp_d, nRemoteRootRanks * sizeof(PetscInt)));
  PetscCallCUDA(hipMalloc((void **)&sf->rootsigdisp_d, nRemoteRootRanks * sizeof(PetscInt)));
  PetscCallCUDA(hipMalloc((void **)&sf->ranks_d, nRemoteRootRanks * sizeof(PetscMPIInt)));
  PetscCallCUDA(hipMalloc((void **)&sf->roffset_d, (nRemoteRootRanks + 1) * sizeof(PetscInt)));

  PetscCallCUDA(hipMemcpyAsync(sf->rootbufdisp_d, sf->rootbufdisp, nRemoteRootRanks * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(sf->rootsigdisp_d, sf->rootsigdisp, nRemoteRootRanks * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(sf->ranks_d, sf->ranks + sf->ndranks, nRemoteRootRanks * sizeof(PetscMPIInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(sf->roffset_d, sf->roffset + sf->ndranks, (nRemoteRootRanks + 1) * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));

  /* Leaf ranks to root ranks: send info about leafsigdisp[] and leafbufdisp[] */
  PetscCall(PetscMalloc2(nRemoteLeafRanks, &bas->leafsigdisp, nRemoteLeafRanks, &bas->leafbufdisp));
  for (i = 0; i < nRemoteLeafRanks; i++) PetscCallMPI(MPI_Irecv(&bas->leafsigdisp[i], 1, MPIU_INT, bas->iranks[i + bas->ndiranks], tag, comm, &rootreqs[i]));
  for (i = 0; i < nRemoteRootRanks; i++) PetscCallMPI(MPI_Send(&i, 1, MPIU_INT, sf->ranks[i + sf->ndranks], tag, comm));
  PetscCallMPI(MPI_Waitall(nRemoteLeafRanks, rootreqs, MPI_STATUSES_IGNORE));

  for (i = 0; i < nRemoteLeafRanks; i++) PetscCallMPI(MPI_Irecv(&bas->leafbufdisp[i], 1, MPIU_INT, bas->iranks[i + bas->ndiranks], tag, comm, &rootreqs[i]));
  for (i = 0; i < nRemoteRootRanks; i++) {
    tmp = sf->roffset[i + sf->ndranks] - sf->roffset[sf->ndranks];
    PetscCallMPI(MPI_Send(&tmp, 1, MPIU_INT, sf->ranks[i + sf->ndranks], tag, comm));
  }
  PetscCallMPI(MPI_Waitall(nRemoteLeafRanks, rootreqs, MPI_STATUSES_IGNORE));

  PetscCallCUDA(hipMalloc((void **)&bas->leafbufdisp_d, nRemoteLeafRanks * sizeof(PetscInt)));
  PetscCallCUDA(hipMalloc((void **)&bas->leafsigdisp_d, nRemoteLeafRanks * sizeof(PetscInt)));
  PetscCallCUDA(hipMalloc((void **)&bas->iranks_d, nRemoteLeafRanks * sizeof(PetscMPIInt)));
  PetscCallCUDA(hipMalloc((void **)&bas->ioffset_d, (nRemoteLeafRanks + 1) * sizeof(PetscInt)));

  PetscCallCUDA(hipMemcpyAsync(bas->leafbufdisp_d, bas->leafbufdisp, nRemoteLeafRanks * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(bas->leafsigdisp_d, bas->leafsigdisp, nRemoteLeafRanks * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(bas->iranks_d, bas->iranks + bas->ndiranks, nRemoteLeafRanks * sizeof(PetscMPIInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));
  PetscCallCUDA(hipMemcpyAsync(bas->ioffset_d, bas->ioffset + bas->ndiranks, (nRemoteLeafRanks + 1) * sizeof(PetscInt), hipMemcpyHostToDevice, PetscDefaultCudaStream));

  PetscCall(PetscFree2(rootreqs, leafreqs));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscSFLinkNvshmemCheck(PetscSF sf, PetscMemType rootmtype, const void *rootdata, PetscMemType leafmtype, const void *leafdata, PetscBool *use_nvshmem)
{
  MPI_Comm    comm;
  PetscBool   isBasic;
  PetscMPIInt result = MPI_UNEQUAL;

  PetscFunctionBegin;
  PetscCall(PetscObjectGetComm((PetscObject)sf, &comm));
  /* Check if the sf is eligible for NVSHMEM, if we have not checked yet.
     Note the check result <use_nvshmem> must be the same over comm, since an SFLink must be collectively either NVSHMEM or MPI.
  */
  sf->checked_nvshmem_eligibility = PETSC_TRUE;
  if (sf->use_nvshmem && !sf->checked_nvshmem_eligibility) {
    /* Only use NVSHMEM for SFBASIC on PETSC_COMM_WORLD  */
    PetscCall(PetscObjectTypeCompare((PetscObject)sf, PETSCSFBASIC, &isBasic));
    if (isBasic) PetscCallMPI(MPI_Comm_compare(PETSC_COMM_WORLD, comm, &result));
    if (!isBasic || (result != MPI_IDENT && result != MPI_CONGRUENT)) sf->use_nvshmem = PETSC_FALSE; /* If not eligible, clear the flag so that we don't try again */

    /* Do further check: If on a rank, both rootdata and leafdata are NULL, we might think they are PETSC_MEMTYPE_CUDA (or HOST)
       and then use NVSHMEM. But if root/leafmtypes on other ranks are PETSC_MEMTYPE_HOST (or DEVICE), this would lead to
       inconsistency on the return value <use_nvshmem>. To be safe, we simply disable nvshmem on these rare SFs.
    */
    if (sf->use_nvshmem) {
      PetscInt hasNullRank = (!rootdata && !leafdata) ? 1 : 0;
      PetscCall(MPIU_Allreduce(MPI_IN_PLACE, &hasNullRank, 1, MPIU_INT, MPI_LOR, comm));
      if (hasNullRank) sf->use_nvshmem = PETSC_FALSE;
    }
    sf->checked_nvshmem_eligibility = PETSC_TRUE; /* If eligible, don't do above check again */
  }

  /* Check if rootmtype and leafmtype collectively are PETSC_MEMTYPE_CUDA */
  if (sf->use_nvshmem) {
    PetscInt oneCuda = (!rootdata || PetscMemTypeCUDA(rootmtype)) && (!leafdata || PetscMemTypeCUDA(leafmtype)) ? 1 : 0; /* Do I use cuda for both root&leafmtype? */
    PetscInt allCuda = oneCuda;                                                                                          /* Assume the same for all ranks. But if not, in opt mode, return value <use_nvshmem> won't be collective! */
#if defined(PETSC_USE_DEBUG)                                                                                             /* Check in debug mode. Note MPI_Allreduce is expensive, so only in debug mode */
    PetscCall(MPIU_Allreduce(&oneCuda, &allCuda, 1, MPIU_INT, MPI_LAND, comm));
    PetscCheck(allCuda == oneCuda, comm, PETSC_ERR_SUP, "root/leaf mtypes are inconsistent among ranks, which may lead to SF nvshmem failure in opt mode. Add -use_nvshmem 0 to disable it.");
#endif
    if (allCuda) {
      PetscCall(PetscNvshmemInitializeCheck());
      if (!sf->setup_nvshmem) { /* Set up nvshmem related fields on this SF on-demand */
        PetscCall(PetscSFSetUp_Basic_NVSHMEM(sf));
        sf->setup_nvshmem = PETSC_TRUE;
      }
      *use_nvshmem = PETSC_TRUE;
    } else {
      *use_nvshmem = PETSC_FALSE;
    }
  } else {
    *use_nvshmem = PETSC_FALSE;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Build dependence between <stream> and <remoteCommStream> at the entry of NVSHMEM communication */
static PetscErrorCode PetscSFLinkBuildDependenceBegin(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas    = (PetscSF_Basic *)sf->data;
  PetscInt       buflen = (direction == PETSCSF_ROOT2LEAF) ? bas->rootbuflen[PETSCSF_REMOTE] : sf->leafbuflen[PETSCSF_REMOTE];

  PetscFunctionBegin;
  if (buflen) {
    PetscCallCUDA(hipEventRecord(link->dataReady, link->stream));
    PetscCallCUDA(hipStreamWaitEvent(link->remoteCommStream, link->dataReady, 0));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Build dependence between <stream> and <remoteCommStream> at the exit of NVSHMEM communication */
static PetscErrorCode PetscSFLinkBuildDependenceEnd(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas    = (PetscSF_Basic *)sf->data;
  PetscInt       buflen = (direction == PETSCSF_ROOT2LEAF) ? sf->leafbuflen[PETSCSF_REMOTE] : bas->rootbuflen[PETSCSF_REMOTE];

  PetscFunctionBegin;
  /* If unpack to non-null device buffer, build the endRemoteComm dependence */
  if (buflen) {
    PetscCallCUDA(hipEventRecord(link->endRemoteComm, link->remoteCommStream));
    PetscCallCUDA(hipStreamWaitEvent(link->stream, link->endRemoteComm, 0));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Send/Put signals to remote ranks

 Input parameters:
  + n        - Number of remote ranks
  . sig      - Signal address in symmetric heap
  . sigdisp  - To i-th rank, use its signal at offset sigdisp[i]
  . ranks    - remote ranks
  - newval   - Set signals to this value
*/
__global__ static void NvshmemSendSignals(PetscInt n, uint64_t *sig, PetscInt *sigdisp, PetscMPIInt *ranks, uint64_t newval)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  /* Each thread puts one remote signal */
  if (i < n) nvshmemx_uint64_signal(sig + sigdisp[i], newval, ranks[i]);
}

/* Wait until local signals equal to the expected value and then set them to a new value

 Input parameters:
  + n        - Number of signals
  . sig      - Local signal address
  . expval   - expected value
  - newval   - Set signals to this new value
*/
__global__ static void NvshmemWaitSignals(PetscInt n, uint64_t *sig, uint64_t expval, uint64_t newval)
{
#if 0
  /* Akhil Langer@NVIDIA said using 1 thread and nvshmem_uint64_wait_until_all is better */
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    nvshmem_signal_wait_until(sig+i,NVSHMEM_CMP_EQ,expval);
    sig[i] = newval;
  }
#else
  nvshmem_uint64_wait_until_all(sig, n, NULL /*no mask*/, NVSHMEM_CMP_EQ, expval);
  for (int i = 0; i < n; i++) sig[i] = newval;
#endif
}

/* ===========================================================================================================

   A set of routines to support receiver initiated communication using the get method

    The getting protocol is:

    Sender has a send buf (sbuf) and a signal variable (ssig);  Receiver has a recv buf (rbuf) and a signal variable (rsig);
    All signal variables have an initial value 0.

    Sender:                                 |  Receiver:
  1.  Wait ssig be 0, then set it to 1
  2.  Pack data into stand alone sbuf       |
  3.  Put 1 to receiver's rsig              |   1. Wait rsig to be 1, then set it 0
                                            |   2. Get data from remote sbuf to local rbuf
                                            |   3. Put 1 to sender's ssig
                                            |   4. Unpack data from local rbuf
   ===========================================================================================================*/
/* PrePack operation -- since sender will overwrite the send buffer which the receiver might be getting data from.
   Sender waits for signals (from receivers) indicating receivers have finished getting data
*/
PetscErrorCode PetscSFLinkWaitSignalsOfCompletionOfGettingData_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  uint64_t      *sig;
  PetscInt       n;

  PetscFunctionBegin;
  if (direction == PETSCSF_ROOT2LEAF) { /* leaf ranks are getting data */
    sig = link->rootSendSig;            /* leaf ranks set my rootSendsig */
    n   = bas->nRemoteLeafRanks;
  } else { /* LEAF2ROOT */
    sig = link->leafSendSig;
    n   = sf->nRemoteRootRanks;
  }

  if (n) {
    NvshmemWaitSignals<<<1, 1, 0, link->remoteCommStream>>>(n, sig, 0, 1); /* wait the signals to be 0, then set them to 1 */
    PetscCallCUDA(hipGetLastError());
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* n thread blocks. Each takes in charge one remote rank */
__global__ static void GetDataFromRemotelyAccessible(PetscInt nsrcranks, PetscMPIInt *srcranks, const char *src, PetscInt *srcdisp, char *dst, PetscInt *dstdisp, PetscInt unitbytes)
{
  int         bid = blockIdx.x;
  PetscMPIInt pe  = srcranks[bid];

  if (!nvshmem_ptr(src, pe)) {
    PetscInt nelems = (dstdisp[bid + 1] - dstdisp[bid]) * unitbytes;
    nvshmem_getmem_nbi(dst + (dstdisp[bid] - dstdisp[0]) * unitbytes, src + srcdisp[bid] * unitbytes, nelems, pe);
  }
}

/* Start communication -- Get data in the given direction */
PetscErrorCode PetscSFLinkGetDataBegin_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;

  PetscInt nsrcranks, ndstranks, nLocallyAccessible = 0;

  char        *src, *dst;
  PetscInt    *srcdisp_h, *dstdisp_h;
  PetscInt    *srcdisp_d, *dstdisp_d;
  PetscMPIInt *srcranks_h;
  PetscMPIInt *srcranks_d, *dstranks_d;
  uint64_t    *dstsig;
  PetscInt    *dstsigdisp_d;

  PetscFunctionBegin;
  PetscCall(PetscSFLinkBuildDependenceBegin(sf, link, direction));
  if (direction == PETSCSF_ROOT2LEAF) { /* src is root, dst is leaf; we will move data from src to dst */
    nsrcranks = sf->nRemoteRootRanks;
    src       = link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]; /* root buf is the send buf; it is in symmetric heap */

    srcdisp_h  = sf->rootbufdisp; /* for my i-th remote root rank, I will access its buf at offset rootbufdisp[i] */
    srcdisp_d  = sf->rootbufdisp_d;
    srcranks_h = sf->ranks + sf->ndranks; /* my (remote) root ranks */
    srcranks_d = sf->ranks_d;

    ndstranks = bas->nRemoteLeafRanks;
    dst       = link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]; /* recv buf is the local leaf buf, also in symmetric heap */

    dstdisp_h  = sf->roffset + sf->ndranks; /* offsets of the local leaf buf. Note dstdisp[0] is not necessarily 0 */
    dstdisp_d  = sf->roffset_d;
    dstranks_d = bas->iranks_d; /* my (remote) leaf ranks */

    dstsig       = link->leafRecvSig;
    dstsigdisp_d = bas->leafsigdisp_d;
  } else { /* src is leaf, dst is root; we will move data from src to dst */
    nsrcranks = bas->nRemoteLeafRanks;
    src       = link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]; /* leaf buf is the send buf */

    srcdisp_h  = bas->leafbufdisp; /* for my i-th remote root rank, I will access its buf at offset rootbufdisp[i] */
    srcdisp_d  = bas->leafbufdisp_d;
    srcranks_h = bas->iranks + bas->ndiranks; /* my (remote) root ranks */
    srcranks_d = bas->iranks_d;

    ndstranks = sf->nRemoteRootRanks;
    dst       = link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]; /* the local root buf is the recv buf */

    dstdisp_h  = bas->ioffset + bas->ndiranks; /* offsets of the local root buf. Note dstdisp[0] is not necessarily 0 */
    dstdisp_d  = bas->ioffset_d;
    dstranks_d = sf->ranks_d; /* my (remote) root ranks */

    dstsig       = link->rootRecvSig;
    dstsigdisp_d = sf->rootsigdisp_d;
  }

  /* After Pack operation -- src tells dst ranks that they are allowed to get data */
  if (ndstranks) {
    NvshmemSendSignals<<<(ndstranks + 255) / 256, 256, 0, link->remoteCommStream>>>(ndstranks, dstsig, dstsigdisp_d, dstranks_d, 1); /* set signals to 1 */
    PetscCallCUDA(hipGetLastError());
  }

  /* dst waits for signals (permissions) from src ranks to start getting data */
  if (nsrcranks) {
    NvshmemWaitSignals<<<1, 1, 0, link->remoteCommStream>>>(nsrcranks, dstsig, 1, 0); /* wait the signals to be 1, then set them to 0 */
    PetscCallCUDA(hipGetLastError());
  }

  /* dst gets data from src ranks using non-blocking nvshmem_gets, which are finished in PetscSFLinkGetDataEnd_NVSHMEM() */

  /* Count number of locally accessible src ranks, which should be a small number */
  for (int i = 0; i < nsrcranks; i++) {
    if (nvshmem_ptr(src, srcranks_h[i])) nLocallyAccessible++;
  }

  /* Get data from remotely accessible PEs */
  if (nLocallyAccessible < nsrcranks) {
    GetDataFromRemotelyAccessible<<<nsrcranks, 1, 0, link->remoteCommStream>>>(nsrcranks, srcranks_d, src, srcdisp_d, dst, dstdisp_d, link->unitbytes);
    PetscCallCUDA(hipGetLastError());
  }

  /* Get data from locally accessible PEs */
  if (nLocallyAccessible) {
    for (int i = 0; i < nsrcranks; i++) {
      int pe = srcranks_h[i];
      if (nvshmem_ptr(src, pe)) {
        size_t nelems = (dstdisp_h[i + 1] - dstdisp_h[i]) * link->unitbytes;
        nvshmemx_getmem_nbi_on_stream(dst + (dstdisp_h[i] - dstdisp_h[0]) * link->unitbytes, src + srcdisp_h[i] * link->unitbytes, nelems, pe, link->remoteCommStream);
      }
    }
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Finish the communication (can be done before Unpack)
   Receiver tells its senders that they are allowed to reuse their send buffer (since receiver has got data from their send buffer)
*/
PetscErrorCode PetscSFLinkGetDataEnd_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  uint64_t      *srcsig;
  PetscInt       nsrcranks, *srcsigdisp;
  PetscMPIInt   *srcranks;

  PetscFunctionBegin;
  if (direction == PETSCSF_ROOT2LEAF) { /* leaf ranks are getting data */
    nsrcranks  = sf->nRemoteRootRanks;
    srcsig     = link->rootSendSig; /* I want to set their root signal */
    srcsigdisp = sf->rootsigdisp_d; /* offset of each root signal */
    srcranks   = sf->ranks_d;       /* ranks of the n root ranks */
  } else {                          /* LEAF2ROOT, root ranks are getting data */
    nsrcranks  = bas->nRemoteLeafRanks;
    srcsig     = link->leafSendSig;
    srcsigdisp = bas->leafsigdisp_d;
    srcranks   = bas->iranks_d;
  }

  if (nsrcranks) {
    nvshmemx_quiet_on_stream(link->remoteCommStream); /* Finish the nonblocking get, so that we can unpack afterwards */
    PetscCallCUDA(hipGetLastError());
    NvshmemSendSignals<<<(nsrcranks + 511) / 512, 512, 0, link->remoteCommStream>>>(nsrcranks, srcsig, srcsigdisp, srcranks, 0); /* set signals to 0 */
    PetscCallCUDA(hipGetLastError());
  }
  PetscCall(PetscSFLinkBuildDependenceEnd(sf, link, direction));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* ===========================================================================================================

   A set of routines to support sender initiated communication using the put-based method (the default)

    The putting protocol is:

    Sender has a send buf (sbuf) and a send signal var (ssig);  Receiver has a stand-alone recv buf (rbuf)
    and a recv signal var (rsig); All signal variables have an initial value 0. rbuf is allocated by SF and
    is in nvshmem space.

    Sender:                                 |  Receiver:
                                            |
  1.  Pack data into sbuf                   |
  2.  Wait ssig be 0, then set it to 1      |
  3.  Put data to remote stand-alone rbuf   |
  4.  Fence // make sure 5 happens after 3  |
  5.  Put 1 to receiver's rsig              |   1. Wait rsig to be 1, then set it 0
                                            |   2. Unpack data from local rbuf
                                            |   3. Put 0 to sender's ssig
   ===========================================================================================================*/

/* n thread blocks. Each takes in charge one remote rank */
__global__ static void WaitAndPutDataToRemotelyAccessible(PetscInt ndstranks, PetscMPIInt *dstranks, char *dst, PetscInt *dstdisp, const char *src, PetscInt *srcdisp, uint64_t *srcsig, PetscInt unitbytes)
{
  int         bid = blockIdx.x;
  PetscMPIInt pe  = dstranks[bid];

  if (!nvshmem_ptr(dst, pe)) {
    PetscInt nelems = (srcdisp[bid + 1] - srcdisp[bid]) * unitbytes;
    nvshmem_uint64_wait_until(srcsig + bid, NVSHMEM_CMP_EQ, 0); /* Wait until the sig = 0 */
    srcsig[bid] = 1;
    nvshmem_putmem_nbi(dst + dstdisp[bid] * unitbytes, src + (srcdisp[bid] - srcdisp[0]) * unitbytes, nelems, pe);
  }
}

/* one-thread kernel, which takes in charge all locally accessible */
__global__ static void WaitSignalsFromLocallyAccessible(PetscInt ndstranks, PetscMPIInt *dstranks, uint64_t *srcsig, const char *dst)
{
  for (int i = 0; i < ndstranks; i++) {
    int pe = dstranks[i];
    if (nvshmem_ptr(dst, pe)) {
      nvshmem_uint64_wait_until(srcsig + i, NVSHMEM_CMP_EQ, 0); /* Wait until the sig = 0 */
      srcsig[i] = 1;
    }
  }
}

/* Put data in the given direction  */
PetscErrorCode PetscSFLinkPutDataBegin_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  PetscInt       ndstranks, nLocallyAccessible = 0;
  char          *src, *dst;
  PetscInt      *srcdisp_h, *dstdisp_h;
  PetscInt      *srcdisp_d, *dstdisp_d;
  PetscMPIInt   *dstranks_h;
  PetscMPIInt   *dstranks_d;
  uint64_t      *srcsig;

  PetscFunctionBegin;
  PetscCall(PetscSFLinkBuildDependenceBegin(sf, link, direction));
  if (direction == PETSCSF_ROOT2LEAF) {                              /* put data in rootbuf to leafbuf  */
    ndstranks = bas->nRemoteLeafRanks;                               /* number of (remote) leaf ranks */
    src       = link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]; /* Both src & dst must be symmetric */
    dst       = link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE];

    srcdisp_h = bas->ioffset + bas->ndiranks; /* offsets of rootbuf. srcdisp[0] is not necessarily zero */
    srcdisp_d = bas->ioffset_d;
    srcsig    = link->rootSendSig;

    dstdisp_h  = bas->leafbufdisp; /* for my i-th remote leaf rank, I will access its leaf buf at offset leafbufdisp[i] */
    dstdisp_d  = bas->leafbufdisp_d;
    dstranks_h = bas->iranks + bas->ndiranks; /* remote leaf ranks */
    dstranks_d = bas->iranks_d;
  } else { /* put data in leafbuf to rootbuf */
    ndstranks = sf->nRemoteRootRanks;
    src       = link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE];
    dst       = link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE];

    srcdisp_h = sf->roffset + sf->ndranks; /* offsets of leafbuf */
    srcdisp_d = sf->roffset_d;
    srcsig    = link->leafSendSig;

    dstdisp_h  = sf->rootbufdisp; /* for my i-th remote root rank, I will access its root buf at offset rootbufdisp[i] */
    dstdisp_d  = sf->rootbufdisp_d;
    dstranks_h = sf->ranks + sf->ndranks; /* remote root ranks */
    dstranks_d = sf->ranks_d;
  }

  /* Wait for signals and then put data to dst ranks using non-blocking nvshmem_put, which are finished in PetscSFLinkPutDataEnd_NVSHMEM */

  /* Count number of locally accessible neighbors, which should be a small number */
  for (int i = 0; i < ndstranks; i++) {
    if (nvshmem_ptr(dst, dstranks_h[i])) nLocallyAccessible++;
  }

  /* For remotely accessible PEs, send data to them in one kernel call */
  if (nLocallyAccessible < ndstranks) {
    WaitAndPutDataToRemotelyAccessible<<<ndstranks, 1, 0, link->remoteCommStream>>>(ndstranks, dstranks_d, dst, dstdisp_d, src, srcdisp_d, srcsig, link->unitbytes);
    PetscCallCUDA(hipGetLastError());
  }

  /* For locally accessible PEs, use host API, which uses CUDA copy-engines and is much faster than device API */
  if (nLocallyAccessible) {
    WaitSignalsFromLocallyAccessible<<<1, 1, 0, link->remoteCommStream>>>(ndstranks, dstranks_d, srcsig, dst);
    for (int i = 0; i < ndstranks; i++) {
      int pe = dstranks_h[i];
      if (nvshmem_ptr(dst, pe)) { /* If return a non-null pointer, then <pe> is locally accessible */
        size_t nelems = (srcdisp_h[i + 1] - srcdisp_h[i]) * link->unitbytes;
        /* Initiate the nonblocking communication */
        nvshmemx_putmem_nbi_on_stream(dst + dstdisp_h[i] * link->unitbytes, src + (srcdisp_h[i] - srcdisp_h[0]) * link->unitbytes, nelems, pe, link->remoteCommStream);
      }
    }
  }

  if (nLocallyAccessible) { nvshmemx_quiet_on_stream(link->remoteCommStream); /* Calling nvshmem_fence/quiet() does not fence the above nvshmemx_putmem_nbi_on_stream! */ }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* A one-thread kernel. The thread takes in charge all remote PEs */
__global__ static void PutDataEnd(PetscInt nsrcranks, PetscInt ndstranks, PetscMPIInt *dstranks, uint64_t *dstsig, PetscInt *dstsigdisp)
{
  /* TODO: Shall we finished the non-blocking remote puts? */

  /* 1. Send a signal to each dst rank */

  /* According to Akhil@NVIDIA, IB is orderred, so no fence is needed for remote PEs.
     For local PEs, we already called nvshmemx_quiet_on_stream(). Therefore, we are good to send signals to all dst ranks now.
  */
  for (int i = 0; i < ndstranks; i++) nvshmemx_uint64_signal(dstsig + dstsigdisp[i], 1, dstranks[i]); /* set sig to 1 */

  /* 2. Wait for signals from src ranks (if any) */
  if (nsrcranks) {
    nvshmem_uint64_wait_until_all(dstsig, nsrcranks, NULL /*no mask*/, NVSHMEM_CMP_EQ, 1); /* wait sigs to be 1, then set them to 0 */
    for (int i = 0; i < nsrcranks; i++) dstsig[i] = 0;
  }
}

/* Finish the communication -- A receiver waits until it can access its receive buffer */
PetscErrorCode PetscSFLinkPutDataEnd_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  PetscMPIInt   *dstranks;
  uint64_t      *dstsig;
  PetscInt       nsrcranks, ndstranks, *dstsigdisp;

  PetscFunctionBegin;
  if (direction == PETSCSF_ROOT2LEAF) { /* put root data to leaf */
    nsrcranks = sf->nRemoteRootRanks;

    ndstranks  = bas->nRemoteLeafRanks;
    dstranks   = bas->iranks_d;      /* leaf ranks */
    dstsig     = link->leafRecvSig;  /* I will set my leaf ranks's RecvSig */
    dstsigdisp = bas->leafsigdisp_d; /* for my i-th remote leaf rank, I will access its signal at offset leafsigdisp[i] */
  } else {                           /* LEAF2ROOT */
    nsrcranks = bas->nRemoteLeafRanks;

    ndstranks  = sf->nRemoteRootRanks;
    dstranks   = sf->ranks_d;
    dstsig     = link->rootRecvSig;
    dstsigdisp = sf->rootsigdisp_d;
  }

  if (nsrcranks || ndstranks) {
    PutDataEnd<<<1, 1, 0, link->remoteCommStream>>>(nsrcranks, ndstranks, dstranks, dstsig, dstsigdisp);
    PetscCallCUDA(hipGetLastError());
  }
  PetscCall(PetscSFLinkBuildDependenceEnd(sf, link, direction));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* PostUnpack operation -- A receiver tells its senders that they are allowed to put data to here (it implies recv buf is free to take new data) */
PetscErrorCode PetscSFLinkSendSignalsToAllowPuttingData_NVSHMEM(PetscSF sf, PetscSFLink link, PetscSFDirection direction)
{
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  uint64_t      *srcsig;
  PetscInt       nsrcranks, *srcsigdisp_d;
  PetscMPIInt   *srcranks_d;

  PetscFunctionBegin;
  if (direction == PETSCSF_ROOT2LEAF) { /* I allow my root ranks to put data to me */
    nsrcranks    = sf->nRemoteRootRanks;
    srcsig       = link->rootSendSig; /* I want to set their send signals */
    srcsigdisp_d = sf->rootsigdisp_d; /* offset of each root signal */
    srcranks_d   = sf->ranks_d;       /* ranks of the n root ranks */
  } else {                            /* LEAF2ROOT */
    nsrcranks    = bas->nRemoteLeafRanks;
    srcsig       = link->leafSendSig;
    srcsigdisp_d = bas->leafsigdisp_d;
    srcranks_d   = bas->iranks_d;
  }

  if (nsrcranks) {
    NvshmemSendSignals<<<(nsrcranks + 255) / 256, 256, 0, link->remoteCommStream>>>(nsrcranks, srcsig, srcsigdisp_d, srcranks_d, 0); /* Set remote signals to 0 */
    PetscCallCUDA(hipGetLastError());
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Destructor when the link uses nvshmem for communication */
static PetscErrorCode PetscSFLinkDestroy_NVSHMEM(PetscSF sf, PetscSFLink link)
{
  hipError_t cerr;

  PetscFunctionBegin;
  PetscCallCUDA(hipEventDestroy(link->dataReady));
  PetscCallCUDA(hipEventDestroy(link->endRemoteComm));
  PetscCallCUDA(hipStreamDestroy(link->remoteCommStream));

  /* nvshmem does not need buffers on host, which should be NULL */
  PetscCall(PetscNvshmemFree(link->leafbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]));
  PetscCall(PetscNvshmemFree(link->leafSendSig));
  PetscCall(PetscNvshmemFree(link->leafRecvSig));
  PetscCall(PetscNvshmemFree(link->rootbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]));
  PetscCall(PetscNvshmemFree(link->rootSendSig));
  PetscCall(PetscNvshmemFree(link->rootRecvSig));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscSFLinkCreate_NVSHMEM(PetscSF sf, MPI_Datatype unit, PetscMemType rootmtype, const void *rootdata, PetscMemType leafmtype, const void *leafdata, MPI_Op op, PetscSFOperation sfop, PetscSFLink *mylink)
{
  hipError_t    cerr;
  PetscSF_Basic *bas = (PetscSF_Basic *)sf->data;
  PetscSFLink   *p, link;
  PetscBool      match, rootdirect[2], leafdirect[2];
  int            greatestPriority;

  PetscFunctionBegin;
  /* Check to see if we can directly send/recv root/leafdata with the given sf, sfop and op.
     We only care root/leafdirect[PETSCSF_REMOTE], since we never need intermediate buffers in local communication with NVSHMEM.
  */
  if (sfop == PETSCSF_BCAST) { /* Move data from rootbuf to leafbuf */
    if (sf->use_nvshmem_get) {
      rootdirect[PETSCSF_REMOTE] = PETSC_FALSE; /* send buffer has to be stand-alone (can't be rootdata) */
      leafdirect[PETSCSF_REMOTE] = (PetscMemTypeNVSHMEM(leafmtype) && sf->leafcontig[PETSCSF_REMOTE] && op == MPI_REPLACE) ? PETSC_TRUE : PETSC_FALSE;
    } else {
      rootdirect[PETSCSF_REMOTE] = (PetscMemTypeNVSHMEM(rootmtype) && bas->rootcontig[PETSCSF_REMOTE]) ? PETSC_TRUE : PETSC_FALSE;
      leafdirect[PETSCSF_REMOTE] = PETSC_FALSE; /* Our put-protocol always needs a nvshmem alloc'ed recv buffer */
    }
  } else if (sfop == PETSCSF_REDUCE) { /* Move data from leafbuf to rootbuf */
    if (sf->use_nvshmem_get) {
      rootdirect[PETSCSF_REMOTE] = (PetscMemTypeNVSHMEM(rootmtype) && bas->rootcontig[PETSCSF_REMOTE] && op == MPI_REPLACE) ? PETSC_TRUE : PETSC_FALSE;
      leafdirect[PETSCSF_REMOTE] = PETSC_FALSE;
    } else {
      rootdirect[PETSCSF_REMOTE] = PETSC_FALSE;
      leafdirect[PETSCSF_REMOTE] = (PetscMemTypeNVSHMEM(leafmtype) && sf->leafcontig[PETSCSF_REMOTE]) ? PETSC_TRUE : PETSC_FALSE;
    }
  } else {                                    /* PETSCSF_FETCH */
    rootdirect[PETSCSF_REMOTE] = PETSC_FALSE; /* FETCH always need a separate rootbuf */
    leafdirect[PETSCSF_REMOTE] = PETSC_FALSE; /* We also force allocating a separate leafbuf so that leafdata and leafupdate can share mpi requests */
  }

  /* Look for free nvshmem links in cache */
  for (p = &bas->avail; (link = *p); p = &link->next) {
    if (link->use_nvshmem) {
      PetscCall(MPIPetsc_Type_compare(unit, link->unit, &match));
      if (match) {
        *p = link->next; /* Remove from available list */
        goto found;
      }
    }
  }
  PetscCall(PetscNew(&link));
  PetscCall(PetscSFLinkSetUp_Host(sf, link, unit));                                          /* Compute link->unitbytes, dup link->unit etc. */
  if (sf->backend == PETSCSF_BACKEND_CUDA) PetscCall(PetscSFLinkSetUp_CUDA(sf, link, unit)); /* Setup pack routines, streams etc */
#if defined(PETSC_HAVE_KOKKOS)
  else if (sf->backend == PETSCSF_BACKEND_KOKKOS) PetscCall(PetscSFLinkSetUp_Kokkos(sf, link, unit));
#endif

  link->rootdirect[PETSCSF_LOCAL] = PETSC_TRUE; /* For the local part we directly use root/leafdata */
  link->leafdirect[PETSCSF_LOCAL] = PETSC_TRUE;

  /* Init signals to zero */
  if (!link->rootSendSig) PetscCall(PetscNvshmemCalloc(bas->nRemoteLeafRanksMax * sizeof(uint64_t), (void **)&link->rootSendSig));
  if (!link->rootRecvSig) PetscCall(PetscNvshmemCalloc(bas->nRemoteLeafRanksMax * sizeof(uint64_t), (void **)&link->rootRecvSig));
  if (!link->leafSendSig) PetscCall(PetscNvshmemCalloc(sf->nRemoteRootRanksMax * sizeof(uint64_t), (void **)&link->leafSendSig));
  if (!link->leafRecvSig) PetscCall(PetscNvshmemCalloc(sf->nRemoteRootRanksMax * sizeof(uint64_t), (void **)&link->leafRecvSig));

  link->use_nvshmem = PETSC_TRUE;
  link->rootmtype   = PETSC_MEMTYPE_DEVICE; /* Only need 0/1-based mtype from now on */
  link->leafmtype   = PETSC_MEMTYPE_DEVICE;
  /* Overwrite some function pointers set by PetscSFLinkSetUp_CUDA */
  link->Destroy = PetscSFLinkDestroy_NVSHMEM;
  if (sf->use_nvshmem_get) { /* get-based protocol */
    link->PrePack             = PetscSFLinkWaitSignalsOfCompletionOfGettingData_NVSHMEM;
    link->StartCommunication  = PetscSFLinkGetDataBegin_NVSHMEM;
    link->FinishCommunication = PetscSFLinkGetDataEnd_NVSHMEM;
  } else { /* put-based protocol */
    link->StartCommunication  = PetscSFLinkPutDataBegin_NVSHMEM;
    link->FinishCommunication = PetscSFLinkPutDataEnd_NVSHMEM;
    link->PostUnpack          = PetscSFLinkSendSignalsToAllowPuttingData_NVSHMEM;
  }

  PetscCallCUDA(hipDeviceGetStreamPriorityRange(NULL, &greatestPriority));
  PetscCallCUDA(hipStreamCreateWithPriority(&link->remoteCommStream, hipStreamNonBlocking, greatestPriority));

  PetscCallCUDA(hipEventCreateWithFlags(&link->dataReady, hipEventDisableTiming));
  PetscCallCUDA(hipEventCreateWithFlags(&link->endRemoteComm, hipEventDisableTiming));

found:
  if (rootdirect[PETSCSF_REMOTE]) {
    link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE] = (char *)rootdata + bas->rootstart[PETSCSF_REMOTE] * link->unitbytes;
  } else {
    if (!link->rootbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]) PetscCall(PetscNvshmemMalloc(bas->rootbuflen_rmax * link->unitbytes, (void **)&link->rootbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]));
    link->rootbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE] = link->rootbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE];
  }

  if (leafdirect[PETSCSF_REMOTE]) {
    link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE] = (char *)leafdata + sf->leafstart[PETSCSF_REMOTE] * link->unitbytes;
  } else {
    if (!link->leafbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]) PetscCall(PetscNvshmemMalloc(sf->leafbuflen_rmax * link->unitbytes, (void **)&link->leafbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE]));
    link->leafbuf[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE] = link->leafbuf_alloc[PETSCSF_REMOTE][PETSC_MEMTYPE_DEVICE];
  }

  link->rootdirect[PETSCSF_REMOTE] = rootdirect[PETSCSF_REMOTE];
  link->leafdirect[PETSCSF_REMOTE] = leafdirect[PETSCSF_REMOTE];
  link->rootdata                   = rootdata; /* root/leafdata are keys to look up links in PetscSFXxxEnd */
  link->leafdata                   = leafdata;
  link->next                       = bas->inuse;
  bas->inuse                       = link;
  *mylink                          = link;
  PetscFunctionReturn(PETSC_SUCCESS);
}

#if defined(PETSC_USE_REAL_SINGLE)
PetscErrorCode PetscNvshmemSum(PetscInt count, float *dst, const float *src)
{
  PetscMPIInt num; /* Assume nvshmem's int is MPI's int */

  PetscFunctionBegin;
  PetscCall(PetscMPIIntCast(count, &num));
  nvshmemx_float_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, dst, src, num, PetscDefaultCudaStream);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemMax(PetscInt count, float *dst, const float *src)
{
  PetscMPIInt num;

  PetscFunctionBegin;
  PetscCall(PetscMPIIntCast(count, &num));
  nvshmemx_float_max_reduce_on_stream(NVSHMEM_TEAM_WORLD, dst, src, num, PetscDefaultCudaStream);
  PetscFunctionReturn(PETSC_SUCCESS);
}
#elif defined(PETSC_USE_REAL_DOUBLE)
PetscErrorCode PetscNvshmemSum(PetscInt count, double *dst, const double *src)
{
  PetscMPIInt num;

  PetscFunctionBegin;
  PetscCall(PetscMPIIntCast(count, &num));
  nvshmemx_double_sum_reduce_on_stream(NVSHMEM_TEAM_WORLD, dst, src, num, PetscDefaultCudaStream);
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscNvshmemMax(PetscInt count, double *dst, const double *src)
{
  PetscMPIInt num;

  PetscFunctionBegin;
  PetscCall(PetscMPIIntCast(count, &num));
  nvshmemx_double_max_reduce_on_stream(NVSHMEM_TEAM_WORLD, dst, src, num, PetscDefaultCudaStream);
  PetscFunctionReturn(PETSC_SUCCESS);
}
#endif
