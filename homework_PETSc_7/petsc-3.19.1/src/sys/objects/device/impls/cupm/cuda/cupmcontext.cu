#include "../cupmcontext.hpp" /*I "petscdevice.h" I*/

using namespace Petsc::device::cupm;

PetscErrorCode PetscDeviceContextCreate_CUDA(PetscDeviceContext dctx)
{
  static constexpr auto cuda_context = CUPMContextCuda();

  PetscFunctionBegin;
  PetscCall(cuda_context.initialize(dctx->device));
  dctx->data = new PetscDeviceContext_(CUDA);
  *dctx->ops = cuda_context.ops;
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* Management of CUBLAS and CUSOLVER handles */
PetscErrorCode PetscCUBLASGetHandle(hipblasHandle_t *handle)
{
  PetscDeviceContext dctx;

  PetscFunctionBegin;
  PetscValidPointer(handle, 1);
  PetscCall(PetscDeviceContextGetCurrentContextAssertType_Internal(&dctx, PETSC_DEVICE_CUDA));
  PetscCall(PetscDeviceContextGetBLASHandle_Internal(dctx, handle));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode PetscCUSOLVERDnGetHandle(hipsolverHandle_t *handle)
{
  PetscDeviceContext dctx;

  PetscFunctionBegin;
  PetscValidPointer(handle, 1);
  PetscCall(PetscDeviceContextGetCurrentContextAssertType_Internal(&dctx, PETSC_DEVICE_CUDA));
  PetscCall(PetscDeviceContextGetSOLVERHandle_Internal(dctx, handle));
  PetscFunctionReturn(PETSC_SUCCESS);
}
