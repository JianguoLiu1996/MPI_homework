static char help[] = "Benchmarking hipPointerGetAttributes() time\n";
/*
  Running example on Summit at OLCF:
  # run with total 1 resource set (RS) (-n1), 1 RS per node (-r1), 1 MPI rank (-a1), 7 cores (-c7) and 1 GPU (-g1) per RS
  $ jsrun -n1 -a1 -c7 -g1 -r1  ./ex2cu
    Average hipPointerGetAttributes() time = 0.31 microseconds
*/
#include <petscsys.h>
#include <petscdevice_cuda.h>

int main(int argc, char **argv)
{
  PetscInt                     i, n = 4000;
  hipError_t                  cerr;
  PetscScalar                **ptrs;
  PetscLogDouble               tstart, tend, time;
  struct hipPointerAttribute_t attr;

  PetscFunctionBeginUser;
  PetscCall(PetscInitialize(&argc, &argv, (char *)0, help));
  PetscCall(PetscOptionsGetInt(NULL, NULL, "-n", &n, NULL));
  PetscCallCUDA(hipStreamSynchronize(NULL)); /* Initialize CUDA runtime to get more accurate timing below */

  PetscCall(PetscMalloc1(n, &ptrs));
  for (i = 0; i < n; i++) {
    if (i % 2) PetscCall(PetscMalloc1(i + 16, &ptrs[i]));
    else PetscCallCUDA(hipMalloc((void **)&ptrs[i], (i + 16) * sizeof(PetscScalar)));
  }

  PetscCall(PetscTime(&tstart));
  for (i = 0; i < n; i++) {
    cerr = hipPointerGetAttributes(&attr, ptrs[i]);
    if (cerr) cerr = hipGetLastError();
  }
  PetscCall(PetscTime(&tend));
  time = (tend - tstart) * 1e6 / n;

  PetscCall(PetscPrintf(PETSC_COMM_WORLD, "Average hipPointerGetAttributes() time = %.2f microseconds\n", time));

  for (i = 0; i < n; i++) {
    if (i % 2) PetscCall(PetscFree(ptrs[i]));
    else PetscCallCUDA(hipFree(ptrs[i]));
  }
  PetscCall(PetscFree(ptrs));

  PetscCall(PetscFinalize());
  return 0;
}

/*TEST
  build:
    requires: cuda

  test:
    requires: cuda
    args: -n 2
    output_file: output/empty.out
    filter: grep "DOES_NOT_EXIST"

TEST*/
