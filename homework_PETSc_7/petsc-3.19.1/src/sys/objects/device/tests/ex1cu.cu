#include "hip/hip_runtime.h"
static char help[] = "Benchmarking CUDA kernel launch time\n";
/*
  Running example on Summit at OLCF:
  # run with total 1 resource set (RS) (-n1), 1 RS per node (-r1), 1 MPI rank (-a1), 7 cores (-c7) and 1 GPU (-g1) per RS
  $ jsrun -n1 -a1 -c7 -g1 -r1  ./ex1cu
  Average asynchronous CUDA kernel launch time = 4.86 microseconds
  Average synchronous  CUDA kernel launch time = 12.83 microseconds
*/
#include <petscsys.h>
#include <petscdevice_cuda.h>

__global__ void NullKernel() { }

int main(int argc, char **argv)
{
  PetscInt       i, n = 100000;
  PetscLogDouble tstart, tend, time;

  PetscFunctionBeginUser;
  PetscCall(PetscInitialize(&argc, &argv, (char *)0, help));
  PetscCall(PetscOptionsGetInt(NULL, NULL, "-n", &n, NULL));
  PetscCallCUDA(hipStreamSynchronize(NULL)); /* Initialize CUDA runtime to get more accurate timing below */

  /* Launch a sequence of kernels asynchronously. Previous launched kernels do not need to be completed before launching a new one */
  PetscCall(PetscTime(&tstart));
  for (i = 0; i < n; i++) NullKernel<<<1, 1, 0, NULL>>>();
  PetscCall(PetscTime(&tend));
  PetscCallCUDA(hipStreamSynchronize(NULL)); /* Sync after tend since we don't want to count kernel execution time */
  time = (tend - tstart) * 1e6 / n;
  PetscCall(PetscPrintf(PETSC_COMM_WORLD, "Average asynchronous CUDA kernel launch time = %.2f microseconds\n", time));

  /* Launch a sequence of kernels synchronously. Only launch a new kernel after the one before it has been completed */
  PetscCall(PetscTime(&tstart));
  for (i = 0; i < n; i++) {
    NullKernel<<<1, 1, 0, NULL>>>();
    PetscCallCUDA(hipStreamSynchronize(NULL));
  }
  PetscCall(PetscTime(&tend));
  time = (tend - tstart) * 1e6 / n;
  PetscCall(PetscPrintf(PETSC_COMM_WORLD, "Average synchronous  CUDA kernel launch time = %.2f microseconds\n", time));

  PetscCall(PetscFinalize());
  return 0;
}

/*TEST
  build:
    requires: cuda

  test:
    requires: cuda
    args: -n 2
    output_file: output/empty.out
    filter: grep "DOES_NOT_EXIST"

TEST*/
