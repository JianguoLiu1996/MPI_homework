#include <petscsys.h>         /*I   "petscsys.h"   I*/
#include <petscdevice_cuda.h> /* Needed to provide PetscCallCUDA() */

static PetscErrorCode PetscCUDAHostMalloc(size_t a, PetscBool, int, const char[], const char[], void **result)
{
  PetscCallCUDA(hipHostMalloc(result, a));
  return PETSC_SUCCESS;
}

static PetscErrorCode PetscCUDAHostFree(void *aa, int, const char[], const char[])
{
  PetscCallCUDA(hipHostFree(aa));
  return PETSC_SUCCESS;
}

static PetscErrorCode PetscCUDAHostRealloc(size_t, int, const char[], const char[], void **)
{
  SETERRQ(PETSC_COMM_SELF, PETSC_ERR_MEM, "CUDA has no Realloc()");
}

static PetscErrorCode (*PetscMallocOld)(size_t, PetscBool, int, const char[], const char[], void **);
static PetscErrorCode (*PetscReallocOld)(size_t, int, const char[], const char[], void **);
static PetscErrorCode (*PetscFreeOld)(void *, int, const char[], const char[]);

/*@C
   PetscMallocSetCUDAHost - Set `PetscMalloc()` to use `CUDAHostMalloc()`
     Switch the current malloc and free routines to the CUDA malloc and free routines

   Not Collective

   Level: developer

   Note:
     This provides a way to use the CUDA malloc and free routines temporarily. One
     can switch back to the previous choice by calling `PetscMallocResetCUDAHost()`.

.seealso: `PetscCUDAHostMalloc()`, `PetscMallocResetCUDAHost()`, `PetscMallocSetHIPHost()`
@*/
PetscErrorCode PetscMallocSetCUDAHost(void)
{
  PetscFunctionBegin;
  /* Save the previous choice */
  PetscMallocOld  = PetscTrMalloc;
  PetscReallocOld = PetscTrRealloc;
  PetscFreeOld    = PetscTrFree;
  PetscTrMalloc   = PetscCUDAHostMalloc;
  PetscTrRealloc  = PetscCUDAHostRealloc;
  PetscTrFree     = PetscCUDAHostFree;
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*@C
   PetscMallocResetCUDAHost - Reset the changes made by `PetscMallocSetCUDAHost()`

   Not Collective

   Level: developer

.seealso: `PetscCUDAHostMalloc()`, `PetscMallocSetCUDAHost()`
@*/
PetscErrorCode PetscMallocResetCUDAHost(void)
{
  PetscFunctionBegin;
  PetscTrMalloc  = PetscMallocOld;
  PetscTrRealloc = PetscReallocOld;
  PetscTrFree    = PetscFreeOld;
  PetscFunctionReturn(PETSC_SUCCESS);
}
