#include "hip/hip_runtime.h"
#define PETSC_SKIP_IMMINTRIN_H_CUDAWORKAROUND 1

#include <petscconf.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h> /*I "petscmat.h" I*/
#include <../src/mat/impls/aij/seq/seqcusparse/cusparsematimpl.h>
#include <../src/mat/impls/aij/mpi/mpicusparse/mpicusparsematimpl.h>
#include <thrust/advance.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <petscsf.h>

struct VecCUDAEquals {
  template <typename Tuple>
  __host__ __device__ void operator()(Tuple t)
  {
    thrust::get<1>(t) = thrust::get<0>(t);
  }
};

static PetscErrorCode MatResetPreallocationCOO_MPIAIJCUSPARSE(Mat mat)
{
  Mat_MPIAIJ         *aij            = (Mat_MPIAIJ *)mat->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)aij->spptr;

  PetscFunctionBegin;
  if (!cusparseStruct) PetscFunctionReturn(PETSC_SUCCESS);
  if (cusparseStruct->use_extended_coo) {
    PetscCallCUDA(hipFree(cusparseStruct->Ajmap1_d));
    PetscCallCUDA(hipFree(cusparseStruct->Aperm1_d));
    PetscCallCUDA(hipFree(cusparseStruct->Bjmap1_d));
    PetscCallCUDA(hipFree(cusparseStruct->Bperm1_d));
    PetscCallCUDA(hipFree(cusparseStruct->Aimap2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Ajmap2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Aperm2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Bimap2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Bjmap2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Bperm2_d));
    PetscCallCUDA(hipFree(cusparseStruct->Cperm1_d));
    PetscCallCUDA(hipFree(cusparseStruct->sendbuf_d));
    PetscCallCUDA(hipFree(cusparseStruct->recvbuf_d));
  }
  cusparseStruct->use_extended_coo = PETSC_FALSE;
  delete cusparseStruct->coo_p;
  delete cusparseStruct->coo_pw;
  cusparseStruct->coo_p  = NULL;
  cusparseStruct->coo_pw = NULL;
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSetValuesCOO_MPIAIJCUSPARSE_Basic(Mat A, const PetscScalar v[], InsertMode imode)
{
  Mat_MPIAIJ         *a    = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusp = (Mat_MPIAIJCUSPARSE *)a->spptr;
  PetscInt            n    = cusp->coo_nd + cusp->coo_no;

  PetscFunctionBegin;
  if (cusp->coo_p && v) {
    thrust::device_ptr<const PetscScalar> d_v;
    THRUSTARRAY                          *w = NULL;

    if (isCudaMem(v)) {
      d_v = thrust::device_pointer_cast(v);
    } else {
      w = new THRUSTARRAY(n);
      w->assign(v, v + n);
      PetscCall(PetscLogCpuToGpu(n * sizeof(PetscScalar)));
      d_v = w->data();
    }

    auto zibit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v, cusp->coo_p->begin()), cusp->coo_pw->begin()));
    auto zieit = thrust::make_zip_iterator(thrust::make_tuple(thrust::make_permutation_iterator(d_v, cusp->coo_p->end()), cusp->coo_pw->end()));
    PetscCall(PetscLogGpuTimeBegin());
    thrust::for_each(zibit, zieit, VecCUDAEquals());
    PetscCall(PetscLogGpuTimeEnd());
    delete w;
    PetscCall(MatSetValuesCOO_SeqAIJCUSPARSE_Basic(a->A, cusp->coo_pw->data().get(), imode));
    PetscCall(MatSetValuesCOO_SeqAIJCUSPARSE_Basic(a->B, cusp->coo_pw->data().get() + cusp->coo_nd, imode));
  } else {
    PetscCall(MatSetValuesCOO_SeqAIJCUSPARSE_Basic(a->A, v, imode));
    PetscCall(MatSetValuesCOO_SeqAIJCUSPARSE_Basic(a->B, v ? v + cusp->coo_nd : NULL, imode));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

template <typename Tuple>
struct IsNotOffDiagT {
  PetscInt _cstart, _cend;

  IsNotOffDiagT(PetscInt cstart, PetscInt cend) : _cstart(cstart), _cend(cend) { }
  __host__ __device__ inline bool operator()(Tuple t) { return !(thrust::get<1>(t) < _cstart || thrust::get<1>(t) >= _cend); }
};

struct IsOffDiag {
  PetscInt _cstart, _cend;

  IsOffDiag(PetscInt cstart, PetscInt cend) : _cstart(cstart), _cend(cend) { }
  __host__ __device__ inline bool operator()(const PetscInt &c) { return c < _cstart || c >= _cend; }
};

struct GlobToLoc {
  PetscInt _start;

  GlobToLoc(PetscInt start) : _start(start) { }
  __host__ __device__ inline PetscInt operator()(const PetscInt &c) { return c - _start; }
};

static PetscErrorCode MatSetPreallocationCOO_MPIAIJCUSPARSE_Basic(Mat B, PetscCount n, PetscInt coo_i[], PetscInt coo_j[])
{
  Mat_MPIAIJ            *b    = (Mat_MPIAIJ *)B->data;
  Mat_MPIAIJCUSPARSE    *cusp = (Mat_MPIAIJCUSPARSE *)b->spptr;
  PetscInt               N, *jj;
  size_t                 noff = 0;
  THRUSTINTARRAY         d_i(n); /* on device, storing partitioned coo_i with diagonal first, and off-diag next */
  THRUSTINTARRAY         d_j(n);
  ISLocalToGlobalMapping l2g;

  PetscFunctionBegin;
  PetscCall(MatDestroy(&b->A));
  PetscCall(MatDestroy(&b->B));

  PetscCall(PetscLogCpuToGpu(2. * n * sizeof(PetscInt)));
  d_i.assign(coo_i, coo_i + n);
  d_j.assign(coo_j, coo_j + n);
  PetscCall(PetscLogGpuTimeBegin());
  auto firstoffd = thrust::find_if(thrust::device, d_j.begin(), d_j.end(), IsOffDiag(B->cmap->rstart, B->cmap->rend));
  auto firstdiag = thrust::find_if_not(thrust::device, firstoffd, d_j.end(), IsOffDiag(B->cmap->rstart, B->cmap->rend));
  if (firstoffd != d_j.end() && firstdiag != d_j.end()) {
    cusp->coo_p  = new THRUSTINTARRAY(n);
    cusp->coo_pw = new THRUSTARRAY(n);
    thrust::sequence(thrust::device, cusp->coo_p->begin(), cusp->coo_p->end(), 0);
    auto fzipp = thrust::make_zip_iterator(thrust::make_tuple(d_i.begin(), d_j.begin(), cusp->coo_p->begin()));
    auto ezipp = thrust::make_zip_iterator(thrust::make_tuple(d_i.end(), d_j.end(), cusp->coo_p->end()));
    auto mzipp = thrust::partition(thrust::device, fzipp, ezipp, IsNotOffDiagT<thrust::tuple<PetscInt, PetscInt, PetscInt>>(B->cmap->rstart, B->cmap->rend));
    firstoffd  = mzipp.get_iterator_tuple().get<1>();
  }
  cusp->coo_nd = thrust::distance(d_j.begin(), firstoffd);
  cusp->coo_no = thrust::distance(firstoffd, d_j.end());

  /* from global to local */
  thrust::transform(thrust::device, d_i.begin(), d_i.end(), d_i.begin(), GlobToLoc(B->rmap->rstart));
  thrust::transform(thrust::device, d_j.begin(), firstoffd, d_j.begin(), GlobToLoc(B->cmap->rstart));
  PetscCall(PetscLogGpuTimeEnd());

  /* copy offdiag column indices to map on the CPU */
  PetscCall(PetscMalloc1(cusp->coo_no, &jj)); /* jj[] will store compacted col ids of the offdiag part */
  PetscCallCUDA(hipMemcpy(jj, d_j.data().get() + cusp->coo_nd, cusp->coo_no * sizeof(PetscInt), hipMemcpyDeviceToHost));
  auto o_j = d_j.begin();
  PetscCall(PetscLogGpuTimeBegin());
  thrust::advance(o_j, cusp->coo_nd); /* sort and unique offdiag col ids */
  thrust::sort(thrust::device, o_j, d_j.end());
  auto wit = thrust::unique(thrust::device, o_j, d_j.end()); /* return end iter of the unique range */
  PetscCall(PetscLogGpuTimeEnd());
  noff = thrust::distance(o_j, wit);
  /* allocate the garray, the columns of B will not be mapped in the multiply setup */
  PetscCall(PetscMalloc1(noff, &b->garray));
  PetscCallCUDA(hipMemcpy(b->garray, d_j.data().get() + cusp->coo_nd, noff * sizeof(PetscInt), hipMemcpyDeviceToHost));
  PetscCall(PetscLogGpuToCpu((noff + cusp->coo_no) * sizeof(PetscInt)));
  PetscCall(ISLocalToGlobalMappingCreate(PETSC_COMM_SELF, 1, noff, b->garray, PETSC_COPY_VALUES, &l2g));
  PetscCall(ISLocalToGlobalMappingSetType(l2g, ISLOCALTOGLOBALMAPPINGHASH));
  PetscCall(ISGlobalToLocalMappingApply(l2g, IS_GTOLM_DROP, cusp->coo_no, jj, &N, jj));
  PetscCheck(N == cusp->coo_no, PETSC_COMM_SELF, PETSC_ERR_PLIB, "Unexpected is size %" PetscInt_FMT " != %" PetscInt_FMT " coo size", N, cusp->coo_no);
  PetscCall(ISLocalToGlobalMappingDestroy(&l2g));

  PetscCall(MatCreate(PETSC_COMM_SELF, &b->A));
  PetscCall(MatSetSizes(b->A, B->rmap->n, B->cmap->n, B->rmap->n, B->cmap->n));
  PetscCall(MatSetType(b->A, MATSEQAIJCUSPARSE));
  PetscCall(MatCreate(PETSC_COMM_SELF, &b->B));
  PetscCall(MatSetSizes(b->B, B->rmap->n, noff, B->rmap->n, noff));
  PetscCall(MatSetType(b->B, MATSEQAIJCUSPARSE));

  /* GPU memory, cusparse specific call handles it internally */
  PetscCall(MatSetPreallocationCOO_SeqAIJCUSPARSE_Basic(b->A, cusp->coo_nd, d_i.data().get(), d_j.data().get()));
  PetscCall(MatSetPreallocationCOO_SeqAIJCUSPARSE_Basic(b->B, cusp->coo_no, d_i.data().get() + cusp->coo_nd, jj));
  PetscCall(PetscFree(jj));

  PetscCall(MatCUSPARSESetFormat(b->A, MAT_CUSPARSE_MULT, cusp->diagGPUMatFormat));
  PetscCall(MatCUSPARSESetFormat(b->B, MAT_CUSPARSE_MULT, cusp->offdiagGPUMatFormat));

  PetscCall(MatBindToCPU(b->A, B->boundtocpu));
  PetscCall(MatBindToCPU(b->B, B->boundtocpu));
  PetscCall(MatSetUpMultiply_MPIAIJ(B));
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatSetPreallocationCOO_MPIAIJCUSPARSE(Mat mat, PetscCount coo_n, PetscInt coo_i[], PetscInt coo_j[])
{
  Mat_MPIAIJ         *mpiaij = (Mat_MPIAIJ *)mat->data;
  Mat_MPIAIJCUSPARSE *mpidev;
  PetscBool           coo_basic = PETSC_TRUE;
  PetscMemType        mtype     = PETSC_MEMTYPE_DEVICE;
  PetscInt            rstart, rend;

  PetscFunctionBegin;
  PetscCall(PetscFree(mpiaij->garray));
  PetscCall(VecDestroy(&mpiaij->lvec));
#if defined(PETSC_USE_CTABLE)
  PetscCall(PetscHMapIDestroy(&mpiaij->colmap));
#else
  PetscCall(PetscFree(mpiaij->colmap));
#endif
  PetscCall(VecScatterDestroy(&mpiaij->Mvctx));
  mat->assembled     = PETSC_FALSE;
  mat->was_assembled = PETSC_FALSE;
  PetscCall(MatResetPreallocationCOO_MPIAIJ(mat));
  PetscCall(MatResetPreallocationCOO_MPIAIJCUSPARSE(mat));
  if (coo_i) {
    PetscCall(PetscLayoutGetRange(mat->rmap, &rstart, &rend));
    PetscCall(PetscGetMemType(coo_i, &mtype));
    if (PetscMemTypeHost(mtype)) {
      for (PetscCount k = 0; k < coo_n; k++) { /* Are there negative indices or remote entries? */
        if (coo_i[k] < 0 || coo_i[k] < rstart || coo_i[k] >= rend || coo_j[k] < 0) {
          coo_basic = PETSC_FALSE;
          break;
        }
      }
    }
  }
  /* All ranks must agree on the value of coo_basic */
  PetscCall(MPIU_Allreduce(MPI_IN_PLACE, &coo_basic, 1, MPIU_BOOL, MPI_LAND, PetscObjectComm((PetscObject)mat)));
  if (coo_basic) {
    PetscCall(MatSetPreallocationCOO_MPIAIJCUSPARSE_Basic(mat, coo_n, coo_i, coo_j));
  } else {
    PetscCall(MatSetPreallocationCOO_MPIAIJ(mat, coo_n, coo_i, coo_j));
    mat->offloadmask = PETSC_OFFLOAD_CPU;
    /* creates the GPU memory */
    PetscCall(MatSeqAIJCUSPARSECopyToGPU(mpiaij->A));
    PetscCall(MatSeqAIJCUSPARSECopyToGPU(mpiaij->B));
    mpidev                   = static_cast<Mat_MPIAIJCUSPARSE *>(mpiaij->spptr);
    mpidev->use_extended_coo = PETSC_TRUE;

    PetscCallCUDA(hipMalloc((void **)&mpidev->Ajmap1_d, (mpiaij->Annz + 1) * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Aperm1_d, mpiaij->Atot1 * sizeof(PetscCount)));

    PetscCallCUDA(hipMalloc((void **)&mpidev->Bjmap1_d, (mpiaij->Bnnz + 1) * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Bperm1_d, mpiaij->Btot1 * sizeof(PetscCount)));

    PetscCallCUDA(hipMalloc((void **)&mpidev->Aimap2_d, mpiaij->Annz2 * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Ajmap2_d, (mpiaij->Annz2 + 1) * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Aperm2_d, mpiaij->Atot2 * sizeof(PetscCount)));

    PetscCallCUDA(hipMalloc((void **)&mpidev->Bimap2_d, mpiaij->Bnnz2 * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Bjmap2_d, (mpiaij->Bnnz2 + 1) * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->Bperm2_d, mpiaij->Btot2 * sizeof(PetscCount)));

    PetscCallCUDA(hipMalloc((void **)&mpidev->Cperm1_d, mpiaij->sendlen * sizeof(PetscCount)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->sendbuf_d, mpiaij->sendlen * sizeof(PetscScalar)));
    PetscCallCUDA(hipMalloc((void **)&mpidev->recvbuf_d, mpiaij->recvlen * sizeof(PetscScalar)));

    PetscCallCUDA(hipMemcpy(mpidev->Ajmap1_d, mpiaij->Ajmap1, (mpiaij->Annz + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Aperm1_d, mpiaij->Aperm1, mpiaij->Atot1 * sizeof(PetscCount), hipMemcpyHostToDevice));

    PetscCallCUDA(hipMemcpy(mpidev->Bjmap1_d, mpiaij->Bjmap1, (mpiaij->Bnnz + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Bperm1_d, mpiaij->Bperm1, mpiaij->Btot1 * sizeof(PetscCount), hipMemcpyHostToDevice));

    PetscCallCUDA(hipMemcpy(mpidev->Aimap2_d, mpiaij->Aimap2, mpiaij->Annz2 * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Ajmap2_d, mpiaij->Ajmap2, (mpiaij->Annz2 + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Aperm2_d, mpiaij->Aperm2, mpiaij->Atot2 * sizeof(PetscCount), hipMemcpyHostToDevice));

    PetscCallCUDA(hipMemcpy(mpidev->Bimap2_d, mpiaij->Bimap2, mpiaij->Bnnz2 * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Bjmap2_d, mpiaij->Bjmap2, (mpiaij->Bnnz2 + 1) * sizeof(PetscCount), hipMemcpyHostToDevice));
    PetscCallCUDA(hipMemcpy(mpidev->Bperm2_d, mpiaij->Bperm2, mpiaij->Btot2 * sizeof(PetscCount), hipMemcpyHostToDevice));

    PetscCallCUDA(hipMemcpy(mpidev->Cperm1_d, mpiaij->Cperm1, mpiaij->sendlen * sizeof(PetscCount), hipMemcpyHostToDevice));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

__global__ static void MatPackCOOValues(const PetscScalar kv[], PetscCount nnz, const PetscCount perm[], PetscScalar buf[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < nnz; i += grid_size) buf[i] = kv[perm[i]];
}

__global__ static void MatAddLocalCOOValues(const PetscScalar kv[], InsertMode imode, PetscCount Annz, const PetscCount Ajmap1[], const PetscCount Aperm1[], PetscScalar Aa[], PetscCount Bnnz, const PetscCount Bjmap1[], const PetscCount Bperm1[], PetscScalar Ba[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < Annz + Bnnz; i += grid_size) {
    PetscScalar sum = 0.0;
    if (i < Annz) {
      for (PetscCount k = Ajmap1[i]; k < Ajmap1[i + 1]; k++) sum += kv[Aperm1[k]];
      Aa[i] = (imode == INSERT_VALUES ? 0.0 : Aa[i]) + sum;
    } else {
      i -= Annz;
      for (PetscCount k = Bjmap1[i]; k < Bjmap1[i + 1]; k++) sum += kv[Bperm1[k]];
      Ba[i] = (imode == INSERT_VALUES ? 0.0 : Ba[i]) + sum;
    }
  }
}

__global__ static void MatAddRemoteCOOValues(const PetscScalar kv[], PetscCount Annz2, const PetscCount Aimap2[], const PetscCount Ajmap2[], const PetscCount Aperm2[], PetscScalar Aa[], PetscCount Bnnz2, const PetscCount Bimap2[], const PetscCount Bjmap2[], const PetscCount Bperm2[], PetscScalar Ba[])
{
  PetscCount       i         = blockIdx.x * blockDim.x + threadIdx.x;
  const PetscCount grid_size = gridDim.x * blockDim.x;
  for (; i < Annz2 + Bnnz2; i += grid_size) {
    if (i < Annz2) {
      for (PetscCount k = Ajmap2[i]; k < Ajmap2[i + 1]; k++) Aa[Aimap2[i]] += kv[Aperm2[k]];
    } else {
      i -= Annz2;
      for (PetscCount k = Bjmap2[i]; k < Bjmap2[i + 1]; k++) Ba[Bimap2[i]] += kv[Bperm2[k]];
    }
  }
}

static PetscErrorCode MatSetValuesCOO_MPIAIJCUSPARSE(Mat mat, const PetscScalar v[], InsertMode imode)
{
  Mat_MPIAIJ         *mpiaij = static_cast<Mat_MPIAIJ *>(mat->data);
  Mat_MPIAIJCUSPARSE *mpidev = static_cast<Mat_MPIAIJCUSPARSE *>(mpiaij->spptr);
  Mat                 A = mpiaij->A, B = mpiaij->B;
  PetscCount          Annz = mpiaij->Annz, Annz2 = mpiaij->Annz2, Bnnz = mpiaij->Bnnz, Bnnz2 = mpiaij->Bnnz2;
  PetscScalar        *Aa, *Ba = NULL;
  PetscScalar        *vsend = mpidev->sendbuf_d, *v2 = mpidev->recvbuf_d;
  const PetscScalar  *v1     = v;
  const PetscCount   *Ajmap1 = mpidev->Ajmap1_d, *Ajmap2 = mpidev->Ajmap2_d, *Aimap2 = mpidev->Aimap2_d;
  const PetscCount   *Bjmap1 = mpidev->Bjmap1_d, *Bjmap2 = mpidev->Bjmap2_d, *Bimap2 = mpidev->Bimap2_d;
  const PetscCount   *Aperm1 = mpidev->Aperm1_d, *Aperm2 = mpidev->Aperm2_d, *Bperm1 = mpidev->Bperm1_d, *Bperm2 = mpidev->Bperm2_d;
  const PetscCount   *Cperm1 = mpidev->Cperm1_d;
  PetscMemType        memtype;

  PetscFunctionBegin;
  if (mpidev->use_extended_coo) {
    PetscMPIInt size;

    PetscCallMPI(MPI_Comm_size(PetscObjectComm((PetscObject)mat), &size));
    PetscCall(PetscGetMemType(v, &memtype));
    if (PetscMemTypeHost(memtype)) { /* If user gave v[] in host, we need to copy it to device */
      PetscCallCUDA(hipMalloc((void **)&v1, mpiaij->coo_n * sizeof(PetscScalar)));
      PetscCallCUDA(hipMemcpy((void *)v1, v, mpiaij->coo_n * sizeof(PetscScalar), hipMemcpyHostToDevice));
    }

    if (imode == INSERT_VALUES) {
      PetscCall(MatSeqAIJCUSPARSEGetArrayWrite(A, &Aa)); /* write matrix values */
      PetscCall(MatSeqAIJCUSPARSEGetArrayWrite(B, &Ba));
    } else {
      PetscCall(MatSeqAIJCUSPARSEGetArray(A, &Aa)); /* read & write matrix values */
      PetscCall(MatSeqAIJCUSPARSEGetArray(B, &Ba));
    }

    /* Pack entries to be sent to remote */
    if (mpiaij->sendlen) {
      MatPackCOOValues<<<(mpiaij->sendlen + 255) / 256, 256>>>(v1, mpiaij->sendlen, Cperm1, vsend);
      PetscCallCUDA(hipPeekAtLastError());
    }

    /* Send remote entries to their owner and overlap the communication with local computation */
    PetscCall(PetscSFReduceWithMemTypeBegin(mpiaij->coo_sf, MPIU_SCALAR, PETSC_MEMTYPE_CUDA, vsend, PETSC_MEMTYPE_CUDA, v2, MPI_REPLACE));
    /* Add local entries to A and B */
    if (Annz + Bnnz > 0) {
      MatAddLocalCOOValues<<<(Annz + Bnnz + 255) / 256, 256>>>(v1, imode, Annz, Ajmap1, Aperm1, Aa, Bnnz, Bjmap1, Bperm1, Ba);
      PetscCallCUDA(hipPeekAtLastError());
    }
    PetscCall(PetscSFReduceEnd(mpiaij->coo_sf, MPIU_SCALAR, vsend, v2, MPI_REPLACE));

    /* Add received remote entries to A and B */
    if (Annz2 + Bnnz2 > 0) {
      MatAddRemoteCOOValues<<<(Annz2 + Bnnz2 + 255) / 256, 256>>>(v2, Annz2, Aimap2, Ajmap2, Aperm2, Aa, Bnnz2, Bimap2, Bjmap2, Bperm2, Ba);
      PetscCallCUDA(hipPeekAtLastError());
    }

    if (imode == INSERT_VALUES) {
      PetscCall(MatSeqAIJCUSPARSERestoreArrayWrite(A, &Aa));
      PetscCall(MatSeqAIJCUSPARSERestoreArrayWrite(B, &Ba));
    } else {
      PetscCall(MatSeqAIJCUSPARSERestoreArray(A, &Aa));
      PetscCall(MatSeqAIJCUSPARSERestoreArray(B, &Ba));
    }
    if (PetscMemTypeHost(memtype)) PetscCallCUDA(hipFree((void *)v1));
  } else {
    PetscCall(MatSetValuesCOO_MPIAIJCUSPARSE_Basic(mat, v, imode));
  }
  mat->offloadmask = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(PETSC_SUCCESS);
}

static PetscErrorCode MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE(Mat A, MatReuse scall, IS *glob, Mat *A_loc)
{
  Mat             Ad, Ao;
  const PetscInt *cmap;

  PetscFunctionBegin;
  PetscCall(MatMPIAIJGetSeqAIJ(A, &Ad, &Ao, &cmap));
  PetscCall(MatSeqAIJCUSPARSEMergeMats(Ad, Ao, scall, A_loc));
  if (glob) {
    PetscInt cst, i, dn, on, *gidx;

    PetscCall(MatGetLocalSize(Ad, NULL, &dn));
    PetscCall(MatGetLocalSize(Ao, NULL, &on));
    PetscCall(MatGetOwnershipRangeColumn(A, &cst, NULL));
    PetscCall(PetscMalloc1(dn + on, &gidx));
    for (i = 0; i < dn; i++) gidx[i] = cst + i;
    for (i = 0; i < on; i++) gidx[i + dn] = cmap[i];
    PetscCall(ISCreateGeneral(PetscObjectComm((PetscObject)Ad), dn + on, gidx, PETSC_OWN_POINTER, glob));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatMPIAIJSetPreallocation_MPIAIJCUSPARSE(Mat B, PetscInt d_nz, const PetscInt d_nnz[], PetscInt o_nz, const PetscInt o_nnz[])
{
  Mat_MPIAIJ         *b              = (Mat_MPIAIJ *)B->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)b->spptr;
  PetscInt            i;

  PetscFunctionBegin;
  PetscCall(PetscLayoutSetUp(B->rmap));
  PetscCall(PetscLayoutSetUp(B->cmap));
  if (PetscDefined(USE_DEBUG) && d_nnz) {
    for (i = 0; i < B->rmap->n; i++) PetscCheck(d_nnz[i] >= 0, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "d_nnz cannot be less than 0: local row %" PetscInt_FMT " value %" PetscInt_FMT, i, d_nnz[i]);
  }
  if (PetscDefined(USE_DEBUG) && o_nnz) {
    for (i = 0; i < B->rmap->n; i++) PetscCheck(o_nnz[i] >= 0, PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "o_nnz cannot be less than 0: local row %" PetscInt_FMT " value %" PetscInt_FMT, i, o_nnz[i]);
  }
#if defined(PETSC_USE_CTABLE)
  PetscCall(PetscHMapIDestroy(&b->colmap));
#else
  PetscCall(PetscFree(b->colmap));
#endif
  PetscCall(PetscFree(b->garray));
  PetscCall(VecDestroy(&b->lvec));
  PetscCall(VecScatterDestroy(&b->Mvctx));
  /* Because the B will have been resized we simply destroy it and create a new one each time */
  PetscCall(MatDestroy(&b->B));
  if (!b->A) {
    PetscCall(MatCreate(PETSC_COMM_SELF, &b->A));
    PetscCall(MatSetSizes(b->A, B->rmap->n, B->cmap->n, B->rmap->n, B->cmap->n));
  }
  if (!b->B) {
    PetscMPIInt size;
    PetscCallMPI(MPI_Comm_size(PetscObjectComm((PetscObject)B), &size));
    PetscCall(MatCreate(PETSC_COMM_SELF, &b->B));
    PetscCall(MatSetSizes(b->B, B->rmap->n, size > 1 ? B->cmap->N : 0, B->rmap->n, size > 1 ? B->cmap->N : 0));
  }
  PetscCall(MatSetType(b->A, MATSEQAIJCUSPARSE));
  PetscCall(MatSetType(b->B, MATSEQAIJCUSPARSE));
  PetscCall(MatBindToCPU(b->A, B->boundtocpu));
  PetscCall(MatBindToCPU(b->B, B->boundtocpu));
  PetscCall(MatSeqAIJSetPreallocation(b->A, d_nz, d_nnz));
  PetscCall(MatSeqAIJSetPreallocation(b->B, o_nz, o_nnz));
  PetscCall(MatCUSPARSESetFormat(b->A, MAT_CUSPARSE_MULT, cusparseStruct->diagGPUMatFormat));
  PetscCall(MatCUSPARSESetFormat(b->B, MAT_CUSPARSE_MULT, cusparseStruct->offdiagGPUMatFormat));
  B->preallocated = PETSC_TRUE;
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatMult_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(VecScatterBegin(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->A->ops->mult)(a->A, xx, yy));
  PetscCall(VecScatterEnd(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->B->ops->multadd)(a->B, a->lvec, yy, yy));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatZeroEntries_MPIAIJCUSPARSE(Mat A)
{
  Mat_MPIAIJ *l = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(MatZeroEntries(l->A));
  PetscCall(MatZeroEntries(l->B));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatMultAdd_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy, Vec zz)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall(VecScatterBegin(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->A->ops->multadd)(a->A, xx, yy, zz));
  PetscCall(VecScatterEnd(a->Mvctx, xx, a->lvec, INSERT_VALUES, SCATTER_FORWARD));
  PetscCall((*a->B->ops->multadd)(a->B, a->lvec, zz, zz));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatMultTranspose_MPIAIJCUSPARSE(Mat A, Vec xx, Vec yy)
{
  Mat_MPIAIJ *a = (Mat_MPIAIJ *)A->data;

  PetscFunctionBegin;
  PetscCall((*a->B->ops->multtranspose)(a->B, xx, a->lvec));
  PetscCall((*a->A->ops->multtranspose)(a->A, xx, yy));
  PetscCall(VecScatterBegin(a->Mvctx, a->lvec, yy, ADD_VALUES, SCATTER_REVERSE));
  PetscCall(VecScatterEnd(a->Mvctx, a->lvec, yy, ADD_VALUES, SCATTER_REVERSE));
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatCUSPARSESetFormat_MPIAIJCUSPARSE(Mat A, MatCUSPARSEFormatOperation op, MatCUSPARSEStorageFormat format)
{
  Mat_MPIAIJ         *a              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)a->spptr;

  PetscFunctionBegin;
  switch (op) {
  case MAT_CUSPARSE_MULT_DIAG:
    cusparseStruct->diagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_MULT_OFFDIAG:
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseStruct->diagGPUMatFormat    = format;
    cusparseStruct->offdiagGPUMatFormat = format;
    break;
  default:
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_SUP, "unsupported operation %d for MatCUSPARSEFormatOperation. Only MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_DIAG, and MAT_CUSPARSE_MULT_ALL are currently supported.", op);
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatSetFromOptions_MPIAIJCUSPARSE(Mat A, PetscOptionItems *PetscOptionsObject)
{
  MatCUSPARSEStorageFormat format;
  PetscBool                flg;
  Mat_MPIAIJ              *a              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE      *cusparseStruct = (Mat_MPIAIJCUSPARSE *)a->spptr;

  PetscFunctionBegin;
  PetscOptionsHeadBegin(PetscOptionsObject, "MPIAIJCUSPARSE options");
  if (A->factortype == MAT_FACTOR_NONE) {
    PetscCall(PetscOptionsEnum("-mat_cusparse_mult_diag_storage_format", "sets storage format of the diagonal blocks of (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->diagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_MULT_DIAG, format));
    PetscCall(PetscOptionsEnum("-mat_cusparse_mult_offdiag_storage_format", "sets storage format of the off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->offdiagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_MULT_OFFDIAG, format));
    PetscCall(PetscOptionsEnum("-mat_cusparse_storage_format", "sets storage format of the diagonal and off-diagonal blocks (mpi)aijcusparse gpu matrices for SpMV", "MatCUSPARSESetFormat", MatCUSPARSEStorageFormats, (PetscEnum)cusparseStruct->diagGPUMatFormat, (PetscEnum *)&format, &flg));
    if (flg) PetscCall(MatCUSPARSESetFormat(A, MAT_CUSPARSE_ALL, format));
  }
  PetscOptionsHeadEnd();
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatAssemblyEnd_MPIAIJCUSPARSE(Mat A, MatAssemblyType mode)
{
  Mat_MPIAIJ         *mpiaij = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusp   = (Mat_MPIAIJCUSPARSE *)mpiaij->spptr;
  PetscObjectState    onnz   = A->nonzerostate;

  PetscFunctionBegin;
  PetscCall(MatAssemblyEnd_MPIAIJ(A, mode));
  if (mpiaij->lvec) PetscCall(VecSetType(mpiaij->lvec, VECSEQCUDA));
  if (onnz != A->nonzerostate && cusp->deviceMat) {
    PetscSplitCSRDataStructure d_mat = cusp->deviceMat, h_mat;

    PetscCall(PetscInfo(A, "Destroy device mat since nonzerostate changed\n"));
    PetscCall(PetscNew(&h_mat));
    PetscCallCUDA(hipMemcpy(h_mat, d_mat, sizeof(*d_mat), hipMemcpyDeviceToHost));
    PetscCallCUDA(hipFree(h_mat->colmap));
    if (h_mat->allocated_indices) {
      PetscCallCUDA(hipFree(h_mat->diag.i));
      PetscCallCUDA(hipFree(h_mat->diag.j));
      if (h_mat->offdiag.j) {
        PetscCallCUDA(hipFree(h_mat->offdiag.i));
        PetscCallCUDA(hipFree(h_mat->offdiag.j));
      }
    }
    PetscCallCUDA(hipFree(d_mat));
    PetscCall(PetscFree(h_mat));
    cusp->deviceMat = NULL;
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

PetscErrorCode MatDestroy_MPIAIJCUSPARSE(Mat A)
{
  Mat_MPIAIJ         *aij            = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)aij->spptr;

  PetscFunctionBegin;
  PetscCheck(cusparseStruct, PETSC_COMM_SELF, PETSC_ERR_COR, "Missing spptr");
  if (cusparseStruct->deviceMat) {
    PetscSplitCSRDataStructure d_mat = cusparseStruct->deviceMat, h_mat;

    PetscCall(PetscInfo(A, "Have device matrix\n"));
    PetscCall(PetscNew(&h_mat));
    PetscCallCUDA(hipMemcpy(h_mat, d_mat, sizeof(*d_mat), hipMemcpyDeviceToHost));
    PetscCallCUDA(hipFree(h_mat->colmap));
    if (h_mat->allocated_indices) {
      PetscCallCUDA(hipFree(h_mat->diag.i));
      PetscCallCUDA(hipFree(h_mat->diag.j));
      if (h_mat->offdiag.j) {
        PetscCallCUDA(hipFree(h_mat->offdiag.i));
        PetscCallCUDA(hipFree(h_mat->offdiag.j));
      }
    }
    PetscCallCUDA(hipFree(d_mat));
    PetscCall(PetscFree(h_mat));
  }
  /* Free COO */
  PetscCall(MatResetPreallocationCOO_MPIAIJCUSPARSE(A));
  PetscCallCXX(delete cusparseStruct);
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJSetPreallocation_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJGetLocalMatMerge_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetPreallocationCOO_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetValuesCOO_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatCUSPARSESetFormat_C", NULL));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatConvert_mpiaijcusparse_hypre_C", NULL));
  PetscCall(MatDestroy_MPIAIJ(A));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/* defines MatSetValues_MPICUSPARSE_Hash() */
#define TYPE AIJ
#define TYPE_AIJ
#define SUB_TYPE_CUSPARSE
#include "../src/mat/impls/aij/mpi/mpihashmat.h"
#undef TYPE
#undef TYPE_AIJ
#undef SUB_TYPE_CUSPARSE

static PetscErrorCode MatSetUp_MPI_HASH_CUSPARSE(Mat A)
{
  Mat_MPIAIJ         *b              = (Mat_MPIAIJ *)A->data;
  Mat_MPIAIJCUSPARSE *cusparseStruct = (Mat_MPIAIJCUSPARSE *)b->spptr;

  PetscFunctionBegin;
  PetscCall(MatSetUp_MPI_Hash(A));
  PetscCall(MatCUSPARSESetFormat(b->A, MAT_CUSPARSE_MULT, cusparseStruct->diagGPUMatFormat));
  PetscCall(MatCUSPARSESetFormat(b->B, MAT_CUSPARSE_MULT, cusparseStruct->offdiagGPUMatFormat));
  A->preallocated = PETSC_TRUE;
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_INTERN PetscErrorCode MatConvert_MPIAIJ_MPIAIJCUSPARSE(Mat B, MatType, MatReuse reuse, Mat *newmat)
{
  Mat_MPIAIJ *a;
  Mat         A;

  PetscFunctionBegin;
  PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  if (reuse == MAT_INITIAL_MATRIX) PetscCall(MatDuplicate(B, MAT_COPY_VALUES, newmat));
  else if (reuse == MAT_REUSE_MATRIX) PetscCall(MatCopy(B, *newmat, SAME_NONZERO_PATTERN));
  A             = *newmat;
  A->boundtocpu = PETSC_FALSE;
  PetscCall(PetscFree(A->defaultvectype));
  PetscCall(PetscStrallocpy(VECCUDA, &A->defaultvectype));

  a = (Mat_MPIAIJ *)A->data;
  if (a->A) PetscCall(MatSetType(a->A, MATSEQAIJCUSPARSE));
  if (a->B) PetscCall(MatSetType(a->B, MATSEQAIJCUSPARSE));
  if (a->lvec) PetscCall(VecSetType(a->lvec, VECSEQCUDA));

  if (reuse != MAT_REUSE_MATRIX && !a->spptr) PetscCallCXX(a->spptr = new Mat_MPIAIJCUSPARSE);

  A->ops->assemblyend           = MatAssemblyEnd_MPIAIJCUSPARSE;
  A->ops->mult                  = MatMult_MPIAIJCUSPARSE;
  A->ops->multadd               = MatMultAdd_MPIAIJCUSPARSE;
  A->ops->multtranspose         = MatMultTranspose_MPIAIJCUSPARSE;
  A->ops->setfromoptions        = MatSetFromOptions_MPIAIJCUSPARSE;
  A->ops->destroy               = MatDestroy_MPIAIJCUSPARSE;
  A->ops->zeroentries           = MatZeroEntries_MPIAIJCUSPARSE;
  A->ops->productsetfromoptions = MatProductSetFromOptions_MPIAIJBACKEND;
  A->ops->setup                 = MatSetUp_MPI_HASH_CUSPARSE;

  PetscCall(PetscObjectChangeTypeName((PetscObject)A, MATMPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJGetLocalMatMerge_C", MatMPIAIJGetLocalMatMerge_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatMPIAIJSetPreallocation_C", MatMPIAIJSetPreallocation_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatCUSPARSESetFormat_C", MatCUSPARSESetFormat_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetPreallocationCOO_C", MatSetPreallocationCOO_MPIAIJCUSPARSE));
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatSetValuesCOO_C", MatSetValuesCOO_MPIAIJCUSPARSE));
#if defined(PETSC_HAVE_HYPRE)
  PetscCall(PetscObjectComposeFunction((PetscObject)A, "MatConvert_mpiaijcusparse_hypre_C", MatConvert_AIJ_HYPRE));
#endif
  PetscFunctionReturn(PETSC_SUCCESS);
}

PETSC_EXTERN PetscErrorCode MatCreate_MPIAIJCUSPARSE(Mat A)
{
  PetscFunctionBegin;
  PetscCall(PetscDeviceInitialize(PETSC_DEVICE_CUDA));
  PetscCall(MatCreate_MPIAIJ(A));
  PetscCall(MatConvert_MPIAIJ_MPIAIJCUSPARSE(A, MATMPIAIJCUSPARSE, MAT_INPLACE_MATRIX, &A));
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*@
   MatCreateAIJCUSPARSE - Creates a sparse matrix in `MATAIJCUSPARSE` (compressed row) format
   (the default parallel PETSc format).  This matrix will ultimately pushed down
   to NVIDIA GPUs and use the CuSPARSE library for calculations.

   Collective

   Input Parameters:
+  comm - MPI communicator, set to `PETSC_COMM_SELF`
.  m - number of local rows (or `PETSC_DECIDE` to have calculated if `M` is given)
           This value should be the same as the local size used in creating the
           y vector for the matrix-vector product y = Ax.
.  n - This value should be the same as the local size used in creating the
       x vector for the matrix-vector product y = Ax. (or PETSC_DECIDE to have
       calculated if `N` is given) For square matrices `n` is almost always `m`.
.  M - number of global rows (or `PETSC_DETERMINE` to have calculated if `m` is given)
.  N - number of global columns (or `PETSC_DETERMINE` to have calculated if `n` is given)
.  d_nz  - number of nonzeros per row in DIAGONAL portion of local submatrix
           (same value is used for all local rows)
.  d_nnz - array containing the number of nonzeros in the various rows of the
           DIAGONAL portion of the local submatrix (possibly different for each row)
           or `NULL`, if `d_nz` is used to specify the nonzero structure.
           The size of this array is equal to the number of local rows, i.e `m`.
           For matrices you plan to factor you must leave room for the diagonal entry and
           put in the entry even if it is zero.
.  o_nz  - number of nonzeros per row in the OFF-DIAGONAL portion of local
           submatrix (same value is used for all local rows).
-  o_nnz - array containing the number of nonzeros in the various rows of the
           OFF-DIAGONAL portion of the local submatrix (possibly different for
           each row) or `NULL`, if `o_nz` is used to specify the nonzero
           structure. The size of this array is equal to the number
           of local rows, i.e `m`.

   Output Parameter:
.  A - the matrix

   Level: intermediate

   Notes:
   It is recommended that one use the `MatCreate()`, `MatSetType()` and/or `MatSetFromOptions()`,
   MatXXXXSetPreallocation() paradigm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, `MatSeqAIJSetPreallocation()`]

   The AIJ format, also called the
   compressed row storage), is fully compatible with standard Fortran
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.

.seealso: [](chapter_matrices), `Mat`, `MATAIJCUSPARSE`, `MatCreate()`, `MatCreateAIJ()`, `MatSetValues()`, `MatSeqAIJSetColumnIndices()`, `MatCreateSeqAIJWithArrays()`, `MatCreateAIJ()`, `MATMPIAIJCUSPARSE`, `MATAIJCUSPARSE`
@*/
PetscErrorCode MatCreateAIJCUSPARSE(MPI_Comm comm, PetscInt m, PetscInt n, PetscInt M, PetscInt N, PetscInt d_nz, const PetscInt d_nnz[], PetscInt o_nz, const PetscInt o_nnz[], Mat *A)
{
  PetscMPIInt size;

  PetscFunctionBegin;
  PetscCall(MatCreate(comm, A));
  PetscCall(MatSetSizes(*A, m, n, M, N));
  PetscCallMPI(MPI_Comm_size(comm, &size));
  if (size > 1) {
    PetscCall(MatSetType(*A, MATMPIAIJCUSPARSE));
    PetscCall(MatMPIAIJSetPreallocation(*A, d_nz, d_nnz, o_nz, o_nnz));
  } else {
    PetscCall(MatSetType(*A, MATSEQAIJCUSPARSE));
    PetscCall(MatSeqAIJSetPreallocation(*A, d_nz, d_nnz));
  }
  PetscFunctionReturn(PETSC_SUCCESS);
}

/*MC
   MATAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as `MATMPIAIJCUSPARSE`.

   A matrix type type whose data resides on NVIDIA GPUs. These matrices can be in either
   CSR, ELL, or Hybrid format. The ELL and HYB formats require CUDA 4.2 or later.
   All matrix calculations are performed on NVIDIA GPUs using the CuSPARSE library.

   This matrix type is identical to `MATSEQAIJCUSPARSE` when constructed with a single process communicator,
   and `MATMPIAIJCUSPARSE` otherwise.  As a result, for single process communicators,
   `MatSeqAIJSetPreallocation()` is supported, and similarly `MatMPIAIJSetPreallocation()` is supported
   for communicators controlling multiple processes.  It is recommended that you call both of
   the above preallocation routines for simplicity.

   Options Database Keys:
+  -mat_type mpiaijcusparse - sets the matrix type to `MATMPIAIJCUSPARSE`
.  -mat_cusparse_storage_format csr - sets the storage format of diagonal and off-diagonal matrices. Other options include ell (ellpack) or hyb (hybrid).
.  -mat_cusparse_mult_diag_storage_format csr - sets the storage format of diagonal matrix. Other options include ell (ellpack) or hyb (hybrid).
-  -mat_cusparse_mult_offdiag_storage_format csr - sets the storage format of off-diagonal matrix. Other options include ell (ellpack) or hyb (hybrid).

  Level: beginner

.seealso: [](chapter_matrices), `Mat`, `MatCreateAIJCUSPARSE()`, `MATSEQAIJCUSPARSE`, `MATMPIAIJCUSPARSE`, `MatCreateSeqAIJCUSPARSE()`, `MatCUSPARSESetFormat()`, `MatCUSPARSEStorageFormat`, `MatCUSPARSEFormatOperation`
M*/

/*MC
   MATMPIAIJCUSPARSE - A matrix type to be used for sparse matrices; it is as same as `MATAIJCUSPARSE`.

  Level: beginner

.seealso: [](chapter_matrices), `Mat`, `MATAIJCUSPARSE`, `MATSEQAIJCUSPARSE`
M*/

// get GPU pointers to stripped down Mat. For both seq and MPI Mat.
PetscErrorCode MatCUSPARSEGetDeviceMatWrite(Mat A, PetscSplitCSRDataStructure *B)
{
  PetscSplitCSRDataStructure d_mat;
  PetscMPIInt                size;
  int                       *ai = NULL, *bi = NULL, *aj = NULL, *bj = NULL;
  PetscScalar               *aa = NULL, *ba = NULL;
  Mat_SeqAIJ                *jaca = NULL, *jacb = NULL;
  Mat_SeqAIJCUSPARSE        *cusparsestructA = NULL;
  CsrMatrix                 *matrixA = NULL, *matrixB = NULL;

  PetscFunctionBegin;
  PetscCheck(A->assembled, PetscObjectComm((PetscObject)A), PETSC_ERR_SUP, "Need already assembled matrix");
  if (A->factortype != MAT_FACTOR_NONE) {
    *B = NULL;
    PetscFunctionReturn(PETSC_SUCCESS);
  }
  PetscCallMPI(MPI_Comm_size(PetscObjectComm((PetscObject)A), &size));
  // get jaca
  if (size == 1) {
    PetscBool isseqaij;

    PetscCall(PetscObjectBaseTypeCompare((PetscObject)A, MATSEQAIJ, &isseqaij));
    if (isseqaij) {
      jaca = (Mat_SeqAIJ *)A->data;
      PetscCheck(jaca->roworiented, PetscObjectComm((PetscObject)A), PETSC_ERR_SUP, "Device assembly does not currently support column oriented values insertion");
      cusparsestructA = (Mat_SeqAIJCUSPARSE *)A->spptr;
      d_mat           = cusparsestructA->deviceMat;
      PetscCall(MatSeqAIJCUSPARSECopyToGPU(A));
    } else {
      Mat_MPIAIJ *aij = (Mat_MPIAIJ *)A->data;
      PetscCheck(aij->roworiented, PetscObjectComm((PetscObject)A), PETSC_ERR_SUP, "Device assembly does not currently support column oriented values insertion");
      Mat_MPIAIJCUSPARSE *spptr = (Mat_MPIAIJCUSPARSE *)aij->spptr;
      jaca                      = (Mat_SeqAIJ *)aij->A->data;
      cusparsestructA           = (Mat_SeqAIJCUSPARSE *)aij->A->spptr;
      d_mat                     = spptr->deviceMat;
      PetscCall(MatSeqAIJCUSPARSECopyToGPU(aij->A));
    }
    if (cusparsestructA->format == MAT_CUSPARSE_CSR) {
      Mat_SeqAIJCUSPARSEMultStruct *matstruct = (Mat_SeqAIJCUSPARSEMultStruct *)cusparsestructA->mat;
      PetscCheck(matstruct, PETSC_COMM_SELF, PETSC_ERR_PLIB, "Missing Mat_SeqAIJCUSPARSEMultStruct for A");
      matrixA = (CsrMatrix *)matstruct->mat;
      bi      = NULL;
      bj      = NULL;
      ba      = NULL;
    } else SETERRQ(PETSC_COMM_SELF, PETSC_ERR_SUP, "Device Mat needs MAT_CUSPARSE_CSR");
  } else {
    Mat_MPIAIJ *aij = (Mat_MPIAIJ *)A->data;
    PetscCheck(aij->roworiented, PetscObjectComm((PetscObject)A), PETSC_ERR_SUP, "Device assembly does not currently support column oriented values insertion");
    jaca                      = (Mat_SeqAIJ *)aij->A->data;
    jacb                      = (Mat_SeqAIJ *)aij->B->data;
    Mat_MPIAIJCUSPARSE *spptr = (Mat_MPIAIJCUSPARSE *)aij->spptr;

    PetscCheck(A->nooffprocentries || aij->donotstash, PetscObjectComm((PetscObject)A), PETSC_ERR_SUP, "Device assembly does not currently support offproc values insertion. Use MatSetOption(A,MAT_NO_OFF_PROC_ENTRIES,PETSC_TRUE) or MatSetOption(A,MAT_IGNORE_OFF_PROC_ENTRIES,PETSC_TRUE)");
    cusparsestructA                     = (Mat_SeqAIJCUSPARSE *)aij->A->spptr;
    Mat_SeqAIJCUSPARSE *cusparsestructB = (Mat_SeqAIJCUSPARSE *)aij->B->spptr;
    PetscCheck(cusparsestructA->format == MAT_CUSPARSE_CSR, PETSC_COMM_SELF, PETSC_ERR_SUP, "Device Mat A needs MAT_CUSPARSE_CSR");
    if (cusparsestructB->format == MAT_CUSPARSE_CSR) {
      PetscCall(MatSeqAIJCUSPARSECopyToGPU(aij->A));
      PetscCall(MatSeqAIJCUSPARSECopyToGPU(aij->B));
      Mat_SeqAIJCUSPARSEMultStruct *matstructA = (Mat_SeqAIJCUSPARSEMultStruct *)cusparsestructA->mat;
      Mat_SeqAIJCUSPARSEMultStruct *matstructB = (Mat_SeqAIJCUSPARSEMultStruct *)cusparsestructB->mat;
      PetscCheck(matstructA, PETSC_COMM_SELF, PETSC_ERR_PLIB, "Missing Mat_SeqAIJCUSPARSEMultStruct for A");
      PetscCheck(matstructB, PETSC_COMM_SELF, PETSC_ERR_PLIB, "Missing Mat_SeqAIJCUSPARSEMultStruct for B");
      matrixA = (CsrMatrix *)matstructA->mat;
      matrixB = (CsrMatrix *)matstructB->mat;
      if (jacb->compressedrow.use) {
        if (!cusparsestructB->rowoffsets_gpu) {
          cusparsestructB->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n + 1);
          cusparsestructB->rowoffsets_gpu->assign(jacb->i, jacb->i + A->rmap->n + 1);
        }
        bi = thrust::raw_pointer_cast(cusparsestructB->rowoffsets_gpu->data());
      } else {
        bi = thrust::raw_pointer_cast(matrixB->row_offsets->data());
      }
      bj = thrust::raw_pointer_cast(matrixB->column_indices->data());
      ba = thrust::raw_pointer_cast(matrixB->values->data());
    } else SETERRQ(PETSC_COMM_SELF, PETSC_ERR_SUP, "Device Mat B needs MAT_CUSPARSE_CSR");
    d_mat = spptr->deviceMat;
  }
  if (jaca->compressedrow.use) {
    if (!cusparsestructA->rowoffsets_gpu) {
      cusparsestructA->rowoffsets_gpu = new THRUSTINTARRAY32(A->rmap->n + 1);
      cusparsestructA->rowoffsets_gpu->assign(jaca->i, jaca->i + A->rmap->n + 1);
    }
    ai = thrust::raw_pointer_cast(cusparsestructA->rowoffsets_gpu->data());
  } else {
    ai = thrust::raw_pointer_cast(matrixA->row_offsets->data());
  }
  aj = thrust::raw_pointer_cast(matrixA->column_indices->data());
  aa = thrust::raw_pointer_cast(matrixA->values->data());

  if (!d_mat) {
    PetscSplitCSRDataStructure h_mat;

    // create and populate strucy on host and copy on device
    PetscCall(PetscInfo(A, "Create device matrix\n"));
    PetscCall(PetscNew(&h_mat));
    PetscCallCUDA(hipMalloc((void **)&d_mat, sizeof(*d_mat)));
    if (size > 1) { /* need the colmap array */
      Mat_MPIAIJ *aij = (Mat_MPIAIJ *)A->data;
      PetscInt   *colmap;
      PetscInt    ii, n = aij->B->cmap->n, N = A->cmap->N;

      PetscCheck(!n || aij->garray, PETSC_COMM_SELF, PETSC_ERR_PLIB, "MPIAIJ Matrix was assembled but is missing garray");

      PetscCall(PetscCalloc1(N + 1, &colmap));
      for (ii = 0; ii < n; ii++) colmap[aij->garray[ii]] = (int)(ii + 1);
#if defined(PETSC_USE_64BIT_INDICES)
      { // have to make a long version of these
        int      *h_bi32, *h_bj32;
        PetscInt *h_bi64, *h_bj64, *d_bi64, *d_bj64;
        PetscCall(PetscCalloc4(A->rmap->n + 1, &h_bi32, jacb->nz, &h_bj32, A->rmap->n + 1, &h_bi64, jacb->nz, &h_bj64));
        PetscCallCUDA(hipMemcpy(h_bi32, bi, (A->rmap->n + 1) * sizeof(*h_bi32), hipMemcpyDeviceToHost));
        for (int i = 0; i < A->rmap->n + 1; i++) h_bi64[i] = h_bi32[i];
        PetscCallCUDA(hipMemcpy(h_bj32, bj, jacb->nz * sizeof(*h_bj32), hipMemcpyDeviceToHost));
        for (int i = 0; i < jacb->nz; i++) h_bj64[i] = h_bj32[i];

        PetscCallCUDA(hipMalloc((void **)&d_bi64, (A->rmap->n + 1) * sizeof(*d_bi64)));
        PetscCallCUDA(hipMemcpy(d_bi64, h_bi64, (A->rmap->n + 1) * sizeof(*d_bi64), hipMemcpyHostToDevice));
        PetscCallCUDA(hipMalloc((void **)&d_bj64, jacb->nz * sizeof(*d_bj64)));
        PetscCallCUDA(hipMemcpy(d_bj64, h_bj64, jacb->nz * sizeof(*d_bj64), hipMemcpyHostToDevice));

        h_mat->offdiag.i         = d_bi64;
        h_mat->offdiag.j         = d_bj64;
        h_mat->allocated_indices = PETSC_TRUE;

        PetscCall(PetscFree4(h_bi32, h_bj32, h_bi64, h_bj64));
      }
#else
      h_mat->offdiag.i         = (PetscInt *)bi;
      h_mat->offdiag.j         = (PetscInt *)bj;
      h_mat->allocated_indices = PETSC_FALSE;
#endif
      h_mat->offdiag.a = ba;
      h_mat->offdiag.n = A->rmap->n;

      PetscCallCUDA(hipMalloc((void **)&h_mat->colmap, (N + 1) * sizeof(*h_mat->colmap)));
      PetscCallCUDA(hipMemcpy(h_mat->colmap, colmap, (N + 1) * sizeof(*h_mat->colmap), hipMemcpyHostToDevice));
      PetscCall(PetscFree(colmap));
    }
    h_mat->rstart = A->rmap->rstart;
    h_mat->rend   = A->rmap->rend;
    h_mat->cstart = A->cmap->rstart;
    h_mat->cend   = A->cmap->rend;
    h_mat->M      = A->cmap->N;
#if defined(PETSC_USE_64BIT_INDICES)
    {
      int      *h_ai32, *h_aj32;
      PetscInt *h_ai64, *h_aj64, *d_ai64, *d_aj64;

      static_assert(sizeof(PetscInt) == 8, "");
      PetscCall(PetscCalloc4(A->rmap->n + 1, &h_ai32, jaca->nz, &h_aj32, A->rmap->n + 1, &h_ai64, jaca->nz, &h_aj64));
      PetscCallCUDA(hipMemcpy(h_ai32, ai, (A->rmap->n + 1) * sizeof(*h_ai32), hipMemcpyDeviceToHost));
      for (int i = 0; i < A->rmap->n + 1; i++) h_ai64[i] = h_ai32[i];
      PetscCallCUDA(hipMemcpy(h_aj32, aj, jaca->nz * sizeof(*h_aj32), hipMemcpyDeviceToHost));
      for (int i = 0; i < jaca->nz; i++) h_aj64[i] = h_aj32[i];

      PetscCallCUDA(hipMalloc((void **)&d_ai64, (A->rmap->n + 1) * sizeof(*d_ai64)));
      PetscCallCUDA(hipMemcpy(d_ai64, h_ai64, (A->rmap->n + 1) * sizeof(*d_ai64), hipMemcpyHostToDevice));
      PetscCallCUDA(hipMalloc((void **)&d_aj64, jaca->nz * sizeof(*d_aj64)));
      PetscCallCUDA(hipMemcpy(d_aj64, h_aj64, jaca->nz * sizeof(*d_aj64), hipMemcpyHostToDevice));

      h_mat->diag.i            = d_ai64;
      h_mat->diag.j            = d_aj64;
      h_mat->allocated_indices = PETSC_TRUE;

      PetscCall(PetscFree4(h_ai32, h_aj32, h_ai64, h_aj64));
    }
#else
    h_mat->diag.i            = (PetscInt *)ai;
    h_mat->diag.j            = (PetscInt *)aj;
    h_mat->allocated_indices = PETSC_FALSE;
#endif
    h_mat->diag.a = aa;
    h_mat->diag.n = A->rmap->n;
    h_mat->rank   = PetscGlobalRank;
    // copy pointers and metadata to device
    PetscCallCUDA(hipMemcpy(d_mat, h_mat, sizeof(*d_mat), hipMemcpyHostToDevice));
    PetscCall(PetscFree(h_mat));
  } else {
    PetscCall(PetscInfo(A, "Reusing device matrix\n"));
  }
  *B             = d_mat;
  A->offloadmask = PETSC_OFFLOAD_GPU;
  PetscFunctionReturn(PETSC_SUCCESS);
}
